#include "hip/hip_runtime.h"
#include "Main.cuh"
#include "Theo.cuh"

extern int NRAD;
extern float *Rmed, *Rinf, CAVITYRATIO, CAVITYRADIUS, SIGMASLOPE, SIGMA0, *QplusMed, *SigmaMed, *SigmaInf, ADIABATICINDEX;
extern float *EnergyMed, ScalingFactor, *CoolingTimeMed, *QplusMed, R, MU, ASPECTRATIO, FLARINGINDEX, COOLINGTIME0;

__host__ void FillSigma ()
{
    for (int i = 0; i < NRAD; i++) {
      SigmaMed[i] = Sigma(Rmed[i]);
      SigmaInf[i] = Sigma(Rinf[i]);
    }
}

__host__ void FillEnergy ()
{
  for (int i = 0; i < NRAD; i++) {
    EnergyMed[i] = Energy(Rmed[i]);
  }
}

/* Thermal energy */
__host__ float Energy(float r)
{
  float energy0;
  if (ADIABATICINDEX == 1.0) {
    fprintf(stderr, "The adiabatic index must differ from unity to initialized \
    the gas internal energy. I must exit.\n");
    exit(1);
  }
  else
    energy0 = R/MU/(ADIABATICINDEX-1.0)*SIGMA0*powf(ASPECTRATIO,2.0)*powf(r,-SIGMASLOPE-1.0+2.0*FLARINGINDEX);
  return energy0;
}

/* Surface density */
__host__ float Sigma(float r)
{
  float cavity = 1.0;
  if (r < CAVITYRADIUS) cavity = 1.0/CAVITYRATIO;
  /* This is *not* a steady state */
  /* profile, if a cavity is defined. It first needs */
  /* to relax towards steady state, on a viscous time scale */

  return cavity*ScalingFactor*SIGMA0*powf(r,-SIGMASLOPE);
}

__host__ float CoolingTime(float r)
{
  float ct0;
  ct0 = COOLINGTIME0*powf(r,2.0+2.0*FLARINGINDEX);
  return ct0;
}

__host__ void FillCoolingTime()
{
  for (int i = 0; i < NRAD; i++) {
    CoolingTimeMed[i] = CoolingTime(Rmed[i]);
  }
}

__host__ void FillQplus()
{
  for (int i = 0; i < NRAD; i++) {
    QplusMed[i] = Qplusinit(Rmed[i]);
  }
}

__host__ float Qplusinit(float r)
{
  float qp0, viscosity;
  viscosity = FViscosity(r);
  qp0 = 2.25*viscosity*SIGMA0*powf(r,-SIGMASLOPE-3.0);
  return qp0;
}
