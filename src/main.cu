#include "hip/hip_runtime.h"
// nvcc hello-world.cu -L /usr/local/cuda/lib -lcudart -o hello-world

#include "main.cuh"
#include "kernels.cuh"
#include "fondam.cuh"
#include "SourceEuler.cuh"
#include "Psys.cuh"
#include "Pframeforce.cuh"
#include "Init.cuh"
#include "hip/hip_runtime.h"
#include "Output.cuh"
#include "Force.cuh"
#include "SideEuler.cuh"

using namespace std;

extern int NRAD, NSEC;
float *Rinf, *Rmed, *Rsup, *Surf, *invRinf, *invSurf, *invdiffSurf;
float *invdiffRsup, *invdiffRmed, *invRmed, *Radii;
float *SigmaMed, *SigmaInf, *dens, *EnergyMed, *energy;
extern int SelfGravity, Corotating, FREQUENCY;
extern float OMEGAFRAME, OmegaFrame1;
float *cosns, *sinns;
int nrad2pot, nsec2pot;
extern float *press, *CellAbscissa, *CellOrdinate, HillRadius, PhysicalTimeInitial, PhysicalTime, CVNR;
int blocksize = 32;
int size_grid;
bool ZMPlus = false, verbose = false, Restart = false;
extern int Adiabaticc;
static int StillWriteOneOutput;
int NbRestart = 0;
float mdcp;
float exces_mdcp = 0.0, ScalingFactor = 1.0;
int dimfxy=11, TimeStep = 0;
int static InnerOutputCounter=0;
bool TimeToWrite;

float *vradint, *pot, *vrad, *vthetaint, *vtheta, *powRmed;
float *temperatureint, *densint, *vradnew, *vthetanew, *energyint;
float *fieldsrc, *vt_int;

__host__ int main(int argc, char *argv[])
{

  bool disable = false, TimeInfo = false, Profiling = false;
  bool Stockholm = false;
  char ParameterFile[256];

  PlanetarySystem *sys;
  Force *force;

  float *gas_v_rad, *gas_v_theta, *gas_label;
  float *rho, xpl, ypl;

  if (argc == 1) PrintUsage (argv[0]);

  strcpy (ParameterFile, "");
  for (int i = 1; i < argc; i++) {
    if (*(argv[i]) == '-') {
      if (strspn (argv[i], "-secndovtpfamzib0123456789") != strlen (argv[i]))
	PrintUsage (argv[0]);
      if (strchr (argv[i], 'n'))
	disable = true;
      if (strchr (argv[i], 'e'))
	Stockholm = true;
      if (strchr (argv[i], 'v'))
	verbose = true;
      if (strchr (argv[i], 't'))
	TimeInfo = true;
      if (strchr (argv[i], 'c'))
	SloppyCFL = true;
      if (strchr (argv[i], 'p'))
	Profiling = true;
      if (strchr (argv[i], 'd'))
	debug = true;
      if (strchr (argv[i], 'b'))
	CentrifugalBalance = true;
      if (strchr (argv[i], 'm'))
	Merge = true;
      if (strchr (argv[i], 'a'))
	MonitorIntegral = true;
      if (strchr (argv[i], 'z'))
	FakeSequential = true;
      if (strchr (argv[i], 'i')) {
	StoreSigma = true;
	if (Adiabaticc)
	  StoreEnergy = true;
      }
      if (strchr (argv[i], '0'))
	OnlyInit = true;
      if ((argv[i][1] >= '1') && (argv[i][1] <= '9')) {
	GotoNextOutput = true;
	StillWriteOneOutput = (int)(argv[i][1]-'0');
      }
      if (strchr (argv[i], 's')) {
	Restart = true;
	i++;
	NbRestart = atoi(argv[i]);
	if ((NbRestart < 0)) {
	  printf ("Incorrect restart number\n");
	  PrintUsage (argv[0]);
	}
      }
      if (strchr (argv[i], 'o')) {
	OverridesOutputdir = true;
	i++;
	sprintf (NewOutputdir, "%s", argv[i]);
      } else {
	if (strchr (argv[i], 'f')) {
	  i++;
	  ScalingFactor = atof(argv[i]);
	  printf ("Scaling factor = %g\n", ScalingFactor);
	  if ((ScalingFactor <= 0)) {
	    printf ("Incorrect scaling factor\n");
	    PrintUsage (argv[0]);
	  }
	}
      }
    }
    else strcpy (ParameterFile, argv[i]);
  }

  if ( (StoreSigma || StoreEnergy) && !(Restart)) {
    printf ("You cannot use tabulated surface density\n");
    printf ("or surface internal energy in a non-restart run.\n");
    printf ("Aborted\n");
    exit (0);
  }
  if (ParameterFile[0] == 0) PrintUsage (argv[0]);


  ReadFile(ParameterFile);
  size_grid = (NRAD+1)*NSEC;

  if(!isPow2(NRAD)) nrad2pot = NearestPowerOf2(NRAD);
  if(!isPow2(NSEC)) nsec2pot = NearestPowerOf2(NSEC);

  dim3 dimGrid( nsec2pot/blocksize, nrad2pot/blocksize );
  dim3 dimBlock( blocksize, blocksize );

  if (verbose == YES) TellEverything();
  if (disable == YES) exit(0);
  printf("Allocating arrays...\n");
  fflush(stdout);

  dens = (float *) malloc(sizeof(float)*(size_grid));
  energy = (float *) malloc(sizeof(float)*size_grid);
  gas_v_rad = (float *) malloc(sizeof(float)*size_grid);
  gas_v_theta = (float *) malloc(sizeof(float)*size_grid);
  gas_label = (float *) malloc(sizeof(float)*size_grid);
  EnergyMed = (float *) malloc(sizeof(float)*NRAD);
  SigmaMed = (float *) malloc(sizeof(float)*NRAD);
  SigmaInf = (float *) malloc(sizeof(float)*NRAD);
  vt_int = (float *) malloc(sizeof(float)*NRAD);
  printf("done.\n");

  FillPolar1DArray();

  force = AllocateForce (dimfxy);

  char configplanet[100];
  strncpy(configplanet, PLANETCONFIG.c_str(), sizeof(configplanet));
  configplanet[sizeof(configplanet)-1]=0;

  /* Here planets are initialized feeling star potential but they do
     not feel disk potential  */

  sys = InitPlanetarySystem(configplanet);

  /* Gas density initialization */
  InitGasDensity ();

  /* If energy equation is taken into account, we initialize the gas
     thermal energy  */
  if ( Adiabaticc ) {
      InitGasEnergy ();
  }

  if ( SelfGravity ) {
    /* If SelfGravity = YES or Z, planets are initialized feeling disk
       potential. Only the surface density is required to calculate
       the radial self-gravity acceleration. The disk radial and
       azimutal velocities are not updated */

    //compute_selfgravity (gas_density, gas_v_rad, gas_v_theta, foostep, updatevelocities);
    //init_planetarysys_withSG (sys);
  }
  ListPlanets (sys);
  OmegaFrame1 = OMEGAFRAME;

  if (Corotating) OmegaFrame1 = GetPsysInfo (sys, FREQUENCY);
  /* Only gas velocities remain to be initialized */

  Initialization (dens, gas_v_rad, gas_v_theta, energy, gas_label, sys);

  xpl = sys->x[0];
  ypl = sys->y[0];

  mdcp = CircumPlanetaryMasshost(xpl, ypl);

  float dt = 0.999;

  EmptyPlanetSystemFile (sys);
  PhysicalTimeInitial = PhysicalTime;
  MultiplyPolarGridbyConstanthost(dens);

  for (int i = 0; i <= NTOT; i++) {
    InnerOutputCounter++;

    if (InnerOutputCounter == 1) {
      InnerOutputCounter = 0;
      WriteBigPlanetSystemFile (sys, TimeStep);
      UpdateLog(force, sys, dens, energy, TimeStep, PhysicalTime, dimfxy);
    }
    if (NINTERM * (TimeStep = (i / NINTERM)) == i)
    {
      /* Outputs are done here */
      TimeToWrite = YES;
      SendOutput (TimeStep, dens, vrad, vtheta, energy, gas_label);
      //WritePlanetSystemFile (sys, TimeStep);
      //printf("%d\n", NINTERM*TimeStep);
    }
    else TimeToWrite = NO;

  }

  vradint = (float *) malloc(sizeof(float)*size_grid);
  pot = (float *) malloc(sizeof(float)*size_grid);
  vrad = (float *) malloc(sizeof(float)*size_grid);
  vthetaint = (float *) malloc(sizeof(float)*size_grid);
  vtheta = (float *) malloc(sizeof(float)*size_grid);
  powRmed = (float *) malloc(sizeof(float)*NRAD);
  temperatureint = (float *)malloc(sizeof(float)*size_grid);
  densint = (float *)malloc(sizeof(float)*size_grid);
  vradnew = (float *)malloc(sizeof(float)*size_grid);
  vthetanew = (float *)malloc(sizeof(float)*size_grid);
  energyint = (float *)malloc(sizeof(float)*size_grid);

  for (int i  = 0; i < size_grid; i++) {
    pot[i] = 0.00001*i;
    densint[i] = 0.1;
  }

 for (int i = 0; i < NRAD; i++) {
   powRmed[i] = powf(Rmed[i],-2.5+SIGMASLOPE);
 }


  substep1host(vrad, vtheta, dens, dt);
  substep2host(dens, energy, dt);
  ActualiseGasVrad(vrad, vradnew);
  ActualiseGasVtheta(vtheta, vthetanew);

  ApplyBoundaryCondition (vrad, vtheta, dens, energy, dt);

/* esto es parte de substep1

  if (SelfGravity){
    selfgravityupdate = YES;
    compute_selfgravity(Rho, VradInt, VthetaInt, dt, selfgravityupdate);
  }
  ComputeViscousTerms (VradInt, VthetaInt, Rho);
  UpdateVelocitiesWithViscosity(vradint, vthetaint, rho, dt);

  if (!Evanescent) ApplySubKeplerianBoundary(VthetaInt);
*/

  FILE *f;
  f = fopen("datos.txt","w");

  for (int i = 0; i < size_grid; i++)
  {
    fprintf(f, "%f\n",vthetaint[i] );
  }

  fclose(f);
	return EXIT_SUCCESS;
}

__host__ float CircumPlanetaryMasshost(float xpl, float ypl)
{

  dim3 dimGrid( nsec2pot/blocksize, nrad2pot/blocksize );
  dim3 dimBlock( blocksize, blocksize );
  float *mdcp0, *mdcp0_d, *CellAbscissa_d, *CellOrdinate_d, *Surf_d, *dens_d;

  mdcp0 = (float *)malloc(size_grid*sizeof(float));

  gpuErrchk(hipMalloc((void**)&Surf_d,NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dens_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&CellAbscissa_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&CellOrdinate_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mdcp0_d,size_grid*sizeof(float)));

  // gpuErrchk(hipMalloc(&fieldsrc_d, ((NRAD+1)*NSEC)*sizeof(float)));
  gpuErrchk(hipMemcpy(Surf_d, Surf, NRAD*sizeof(float), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(dens_d, dens, size_grid*sizeof(float), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(CellAbscissa_d, CellAbscissa, size_grid*sizeof(float), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(CellOrdinate_d, CellOrdinate, size_grid*sizeof(float), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(mdcp0_d, mdcp0, size_grid*sizeof(float), hipMemcpyHostToDevice));

  CircumPlanetaryMass<<<dimGrid, dimBlock>>> (dens_d, Surf_d, CellAbscissa_d, CellOrdinate_d, xpl, ypl, NRAD, NSEC, HillRadius, mdcp0_d);
  gpuErrchk(hipDeviceSynchronize());

  hipFree(Surf_d );
  hipFree(CellAbscissa_d);
  hipFree(CellOrdinate_d);
  hipFree(dens_d);

  // reduction mdcp
  mdcp = deviceReduce(mdcp0_d, size_grid);
  hipFree(mdcp0_d);
  return mdcp;
}

__host__ void MultiplyPolarGridbyConstanthost(float *dens)
{
  dim3 dimGrid( nsec2pot/blocksize, nrad2pot/blocksize );
  dim3 dimBlock( blocksize, blocksize );

  float *dens_d;

  gpuErrchk(hipMalloc((void**)&dens_d,size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(dens_d, dens, size_grid*sizeof(float), hipMemcpyHostToDevice ));

  MultiplyPolarGridbyConstant<<<dimGrid, dimBlock>>>(dens_d, NRAD, NSEC, ScalingFactor);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(dens, dens_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  hipFree(dens_d);

}

__host__ void substep1host(float *vrad, float *vtheta, float *dens, float dt)
{
  dim3 dimGrid( nsec2pot/blocksize, nrad2pot/blocksize );
  dim3 dimBlock( blocksize, blocksize );

  float *press_d, *dens_d, *vradint_d, *pot_d, *vrad_d, *vthetaint_d, *vtheta_d, *invdiffRmed_d;
  float *invRinf_d, *Rinf_d, *Rmed_d, *powRmed_d;

  gpuErrchk(hipMalloc((void**)&press_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dens_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vradint_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&pot_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vrad_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vthetaint_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vtheta_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&invdiffRmed_d, NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&invRinf_d,NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Rinf_d,NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Rmed_d,NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&powRmed_d,NRAD*sizeof(float)));

  gpuErrchk(hipMemcpy(press_d, press, size_grid*sizeof(float), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(dens_d, dens, size_grid*sizeof(float), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(vradint_d, vradint, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(pot_d, pot, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vrad_d, vrad, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vthetaint_d, vthetaint, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vtheta_d, vtheta, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(invdiffRmed_d, invdiffRmed, NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(invRinf_d, invRinf, NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Rinf_d, Rinf, NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Rmed_d, Rmed, NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(powRmed_d, powRmed, NRAD*sizeof(float), hipMemcpyHostToDevice));

  substep1<<<dimGrid, dimBlock>>>(press_d, dens_d, vradint_d, invdiffRmed_d,pot_d,Rinf_d,
    invRinf_d, vrad_d, vthetaint_d, vtheta_d, Rmed_d,  dt, NRAD, NSEC, OmegaFrame1, ZMPlus,
    IMPOSEDDISKDRIFT, SIGMASLOPE, powRmed_d);

  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(vradint, vradint_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(vthetaint, vthetaint_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));


  hipFree(press_d );
  hipFree(dens_d);
  hipFree(vradint_d);
  hipFree(pot_d);
  hipFree(vrad_d);
  hipFree(vthetaint_d);
  hipFree(vtheta_d);
  hipFree(invdiffRmed_d);
  hipFree(invRinf_d);
  hipFree(Rinf_d);
  hipFree(Rmed_d);
  hipFree(powRmed_d);

}

__host__ void substep2host(float *dens, float *energy, float dt)
{
  dim3 dimGrid( nsec2pot/blocksize, nrad2pot/blocksize );
  dim3 dimBlock( blocksize, blocksize );

  float *temperatureint_d, *densint_d, *vradnew_d, *vthetanew_d, *energy_d, *energyint_d, *invdiffRsup_d;
  float *vradint_d, *vthetaint_d, *invdiffRmed_d, *Rmed_d, *dens_d;

  gpuErrchk(hipMalloc((void**)&densint_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vradint_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vthetaint_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&temperatureint_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&invdiffRmed_d, NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&invdiffRsup_d, NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&densint_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Rmed_d, NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vradnew_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vthetanew_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&energy_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&energyint_d, size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(densint_d, densint, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vradint_d, vradint, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vthetaint_d, vthetaint, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(temperatureint_d, temperatureint, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(invdiffRmed_d, invdiffRmed, NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(invdiffRsup_d, invdiffRsup, NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(densint_d, densint, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Rmed_d, Rmed, NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vradnew_d, vradnew, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vthetanew_d, vthetanew, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(energy_d, energy, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(energyint_d, energyint, size_grid*sizeof(float), hipMemcpyHostToDevice));


  substep2<<<dimGrid, dimBlock>>>(dens_d, vradint_d, vthetaint_d, temperatureint_d, NRAD, NSEC, CVNR, invdiffRmed_d,
  invdiffRsup_d, densint_d, Adiabaticc, Rmed_d, dt, vradnew_d, vthetanew_d, energy_d, energyint_d);

  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(densint, densint_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(temperatureint, temperatureint_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(vthetanew, vthetanew_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(vradnew, vradnew_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(energyint, energyint_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));

  hipFree(dens_d);
  hipFree(vradint_d);
  hipFree(vthetaint_d);
  hipFree(temperatureint_d);
  hipFree(invdiffRmed_d);
  hipFree(invdiffRsup_d);
  hipFree(densint_d);
  hipFree(Rmed_d);
  hipFree(vradnew_d);
  hipFree(vthetanew_d);
  hipFree(energy_d);
  hipFree(energyint_d);

}

__host__ void ActualiseGasVtheta(float *vtheta, float *vthetanew)
{
  float *vthetanew_d, *vtheta_d;

  gpuErrchk(hipMalloc((void**)&vthetanew_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vtheta_d,size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(vthetanew_d, vthetanew, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vtheta_d, vtheta, size_grid*sizeof(float), hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(vtheta_d, vthetanew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));

  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(vtheta_d, vthetanew_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));

  hipFree(vthetanew_d);
  hipFree(vtheta_d);
}

__host__ void ActualiseGasVrad(float *vrad, float *vradnew)
{
  float *vradnew_d, *vrad_d;

  gpuErrchk(hipMalloc((void**)&vradnew_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vrad_d,size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(vradnew_d, vradnew, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vrad_d, vrad, size_grid*sizeof(float), hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(vrad_d, vradnew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));

  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(vrad, vrad_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));

  hipFree(vrad_d);
  hipFree(vradnew_d);
}
