// nvcc hello-world.cu -L /usr/local/cuda/lib -lcudart -o hello-world

#include "main.cuh"
#include "kernels.cuh"

int blocksize = 32;
int nrad = 128;
int nsec = 384;
int size_grid = nrad*nsec;
float OmegaFrame = 0.12871;
bool ZMPlus = 0;
float IMPOSEDDISKDRIFT = 0.0;
float SIGMASLOPE = 0.0;


__host__ long NearestPowerOf2(long n)
{
  if(!n) return n; //(0 ==2^0)

  int x=1;
  while (x < n)
  {
    x<<=1;
  }
  return x;
}

__host__ bool isPow2(unsigned int x)
{
  return ((x&(x-1)==0));
}


__host__ int main(int argc, char *argv[])
{
  float *press, *rho, *vradint, *invdiffRmed, *pot, *invRinf, *Rinf, *vrad, *vthetaint, *vtheta, *Rmed;
  float *press_d,*rho_d,*vradint_d,*invdiffRmed_d,*pot_d, *invRinf_d, *Rinf_d, *vrad_d, *vthetaint_d, *vtheta_d, *Rmed_d;
  int nrad2pot, nsec2pot;

  printf("%s\n",argv[0]);
  
  ReadFile();

  float dt = 0.999;
  press = (float *) malloc(sizeof(float)*size_grid);
  rho = (float *) malloc(sizeof(float)*size_grid );
  vradint = (float *) malloc(sizeof(float)*size_grid);
  pot = (float *) malloc(sizeof(float)*size_grid);
  vrad = (float *) malloc(sizeof(float)*size_grid);
  vthetaint = (float *) malloc(sizeof(float)*size_grid);
  vtheta = (float *) malloc(sizeof(float)*size_grid);
  invdiffRmed = (float *) malloc(sizeof(float)*nrad);
  invRinf = (float *) malloc(sizeof(float)*nrad);
  Rinf = (float *) malloc(sizeof(float)*nrad);
  Rmed = (float *) malloc(sizeof(float)*nrad);

  for (int i  = 0; i < size_grid; i++) {
    press[i] = i;
    rho[i] = 2*i;
    pot[i] = 0.001*i;
    vrad[i] = 0.212*i;
    vtheta[i] = 0.1;
    if (i < nrad) {
      invdiffRmed[i]= 0.002;
      Rinf[i] = 0.001;
      invRinf[i] = 1/Rinf[i];
      Rmed[i] = 1/invdiffRmed[i];
    }
  }

  if(!isPow2(nrad)) nrad2pot = NearestPowerOf2(nrad);
  if(!isPow2(nsec)) nsec2pot = NearestPowerOf2(nsec);

	hipMalloc((void**)&press_d, size_grid*sizeof(float));
	hipMalloc((void**)&rho_d, size_grid*sizeof(float) );
  hipMalloc((void**)&vradint_d, size_grid*sizeof(float));
  hipMalloc((void**)&pot_d, size_grid*sizeof(float));
  hipMalloc((void**)&vrad_d, size_grid*sizeof(float));
  hipMalloc((void**)&vthetaint_d, size_grid*sizeof(float));
  hipMalloc((void**)&vtheta_d, size_grid*sizeof(float));
  hipMalloc((void**)&invdiffRmed_d, nrad*sizeof(float));
  hipMalloc((void**)&invRinf_d,nrad*sizeof(float));
  hipMalloc((void**)&Rinf_d,nrad*sizeof(float));
  hipMalloc((void**)&Rmed_d,nrad*sizeof(float));


	hipMemcpy(press_d, press, size_grid*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy(rho_d, rho, size_grid*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy(vradint_d, vradint, size_grid*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(pot_d, pot, size_grid*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vrad_d, vrad, size_grid*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vthetaint_d, vthetaint, size_grid*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vtheta_d, vtheta, size_grid*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(invdiffRmed_d, invdiffRmed, nrad*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(invRinf_d, invRinf, nrad*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Rinf_d, Rinf, nrad*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Rmed_d, Rmed, nrad*sizeof(float), hipMemcpyHostToDevice);


	dim3 dimGrid( nsec2pot/blocksize, nrad2pot/blocksize );
	dim3 dimBlock( blocksize, blocksize );

	substep1<<<dimGrid, dimBlock>>>(press_d, rho_d, vradint_d, invdiffRmed_d,pot_d,Rinf_d,
    invRinf_d, vrad_d, vthetaint_d, vtheta_d, Rmed_d,  dt, nrad, nsec, OmegaFrame, ZMPlus,
    IMPOSEDDISKDRIFT, SIGMASLOPE);

	hipMemcpy(vradint, vradint_d, size_grid*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(vthetaint, vthetaint_d, size_grid*sizeof(float), hipMemcpyDeviceToHost);


	hipFree(press_d );
	hipFree(rho_d );
  hipFree(vradint_d);
  hipFree(pot_d);
  hipFree(invdiffRmed_d);
  hipFree(invRinf_d);
  hipFree(Rinf_d);
  hipFree(vrad_d);
  hipFree(vthetaint_d);
  hipFree(vtheta_d);
  hipFree(Rmed_d);

  /*if (SelfGravity){
    selfgravityupdate = YES;
    compute_selfgravity(Rho, VradInt, VthetaInt, dt, selfgravityupdate);
  }
  ComputeViscousTerms (VradInt, VthetaInt, Rho);*/
  printf("%d\n", nsec);
  UpdateVelocitiesWithViscosity(vradint, vthetaint, rho, dt);
/*
  if (!Evanescent) ApplySubKeplerianBoundary(VthetaInt);
*/
  FILE *f;
  f = fopen("datos.txt","w");

  for (int i = 0; i < size_grid; i++)
  {
    fprintf(f, "%f\n",vthetaint[i] );
  }

  fclose(f);
	return EXIT_SUCCESS;
}
