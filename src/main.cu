// nvcc hello-world.cu -L /usr/local/cuda/lib -lcudart -o hello-world

#include "main.cuh"
#include "kernels.cuh"
#include "readfiles.cuh"

using namespace std;

int blocksize = 32;
int nrad = 128;
int nsec = 384;
int size_grid = nrad*nsec;
float OmegaFrame = 0.12871;
bool ZMPlus = false, verbose = false, Adiabatic = false, Restart = false;
static int StillWriteOneOutput;
int NbRestart = 0;

float ScalingFactor = 1.0;

__host__ long NearestPowerOf2(long n)
{
  if(!n) return n; //(0 ==2^0)

  int x=1;
  while (x < n)
  {
    x<<=1;
  }
  return x;
}

__host__ bool isPow2(unsigned int x)
{
  return ((x&(x-1)==0));
}


__host__ int main(int argc, char *argv[])
{

  bool disable = false, TimeInfo = false, Profiling = false;
  bool Stockholm = false;
  char ParameterFile[256];

  float *press, *rho, *vradint, *invdiffRmed, *pot, *invRinf, *Rinf, *vrad, *vthetaint, *vtheta, *Rmed;
  float *press_d,*rho_d,*vradint_d,*invdiffRmed_d,*pot_d, *invRinf_d, *Rinf_d, *vrad_d, *vthetaint_d, *vtheta_d, *Rmed_d;
  int nrad2pot, nsec2pot;


  if (argc == 1) PrintUsage (argv[0]);

  strcpy (ParameterFile, "");
  for (int i = 1; i < argc; i++) {
    if (*(argv[i]) == '-') {
      if (strspn (argv[i], "-secndovtpfamzib0123456789") != strlen (argv[i]))
	PrintUsage (argv[0]);
      if (strchr (argv[i], 'n'))
	disable = true;
      if (strchr (argv[i], 'e'))
	Stockholm = true;
      if (strchr (argv[i], 'v'))
	verbose = true;
      if (strchr (argv[i], 't'))
	TimeInfo = true;
      if (strchr (argv[i], 'c'))
	SloppyCFL = true;
      if (strchr (argv[i], 'p'))
	Profiling = true;
      if (strchr (argv[i], 'd'))
	debug = true;
      if (strchr (argv[i], 'b'))
	CentrifugalBalance = true;
      if (strchr (argv[i], 'm'))
	Merge = true;
      if (strchr (argv[i], 'a'))
	MonitorIntegral = true;
      if (strchr (argv[i], 'z'))
	FakeSequential = true;
      if (strchr (argv[i], 'i')) {
	StoreSigma = true;
	if (Adiabatic)
	  StoreEnergy = true;
      }
      if (strchr (argv[i], '0'))
	OnlyInit = true;
      if ((argv[i][1] >= '1') && (argv[i][1] <= '9')) {
	GotoNextOutput = true;
	StillWriteOneOutput = (int)(argv[i][1]-'0');
      }
      if (strchr (argv[i], 's')) {
	Restart = true;
	i++;
	NbRestart = atoi(argv[i]);
	if ((NbRestart < 0)) {
	  printf ("Incorrect restart number\n");
	  PrintUsage (argv[0]);
	}
      }
      if (strchr (argv[i], 'o')) {
	OverridesOutputdir = true;
	i++;
	sprintf (NewOutputdir, "%s", argv[i]);
      } else {
	if (strchr (argv[i], 'f')) {
	  i++;
	  ScalingFactor = atof(argv[i]);
	  printf ("Scaling factor = %g\n", ScalingFactor);
	  if ((ScalingFactor <= 0)) {
	    printf ("Incorrect scaling factor\n");
	    PrintUsage (argv[0]);
	  }
	}
      }
    }
    else strcpy (ParameterFile, argv[i]);
  }

  if ( (StoreSigma || StoreEnergy) && !(Restart)) {
    printf ("You cannot use tabulated surface density\n");
    printf ("or surface internal energy in a non-restart run.\n");
    printf ("Aborted\n");
    exit (0);
  }
  if (ParameterFile[0] == 0) PrintUsage (argv[0]);


  ReadFile(ParameterFile);

  float dt = 0.999;
  press = (float *) malloc(sizeof(float)*size_grid);
  rho = (float *) malloc(sizeof(float)*size_grid );
  vradint = (float *) malloc(sizeof(float)*size_grid);
  pot = (float *) malloc(sizeof(float)*size_grid);
  vrad = (float *) malloc(sizeof(float)*size_grid);
  vthetaint = (float *) malloc(sizeof(float)*size_grid);
  vtheta = (float *) malloc(sizeof(float)*size_grid);
  invdiffRmed = (float *) malloc(sizeof(float)*nrad);
  invRinf = (float *) malloc(sizeof(float)*nrad);
  Rinf = (float *) malloc(sizeof(float)*nrad);
  Rmed = (float *) malloc(sizeof(float)*nrad);

  for (int i  = 0; i < size_grid; i++) {
    press[i] = i;
    rho[i] = 2*i;
    pot[i] = 0.001*i;
    vrad[i] = 0.212*i;
    vtheta[i] = 0.1;
    if (i < nrad) {
      invdiffRmed[i]= 0.002;
      Rinf[i] = 0.001;
      invRinf[i] = 1/Rinf[i];
      Rmed[i] = 1/invdiffRmed[i];
    }
  }

  if(!isPow2(nrad)) nrad2pot = NearestPowerOf2(nrad);
  if(!isPow2(nsec)) nsec2pot = NearestPowerOf2(nsec);

	hipMalloc((void**)&press_d, size_grid*sizeof(float));
	hipMalloc((void**)&rho_d, size_grid*sizeof(float) );
  hipMalloc((void**)&vradint_d, size_grid*sizeof(float));
  hipMalloc((void**)&pot_d, size_grid*sizeof(float));
  hipMalloc((void**)&vrad_d, size_grid*sizeof(float));
  hipMalloc((void**)&vthetaint_d, size_grid*sizeof(float));
  hipMalloc((void**)&vtheta_d, size_grid*sizeof(float));
  hipMalloc((void**)&invdiffRmed_d, nrad*sizeof(float));
  hipMalloc((void**)&invRinf_d,nrad*sizeof(float));
  hipMalloc((void**)&Rinf_d,nrad*sizeof(float));
  hipMalloc((void**)&Rmed_d,nrad*sizeof(float));


	hipMemcpy(press_d, press, size_grid*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy(rho_d, rho, size_grid*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy(vradint_d, vradint, size_grid*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(pot_d, pot, size_grid*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vrad_d, vrad, size_grid*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vthetaint_d, vthetaint, size_grid*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vtheta_d, vtheta, size_grid*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(invdiffRmed_d, invdiffRmed, nrad*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(invRinf_d, invRinf, nrad*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Rinf_d, Rinf, nrad*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Rmed_d, Rmed, nrad*sizeof(float), hipMemcpyHostToDevice);


	dim3 dimGrid( nsec2pot/blocksize, nrad2pot/blocksize );
	dim3 dimBlock( blocksize, blocksize );

	substep1<<<dimGrid, dimBlock>>>(press_d, rho_d, vradint_d, invdiffRmed_d,pot_d,Rinf_d,
    invRinf_d, vrad_d, vthetaint_d, vtheta_d, Rmed_d,  dt, nrad, nsec, OmegaFrame, ZMPlus,
    IMPOSEDDISKDRIFT, SIGMASLOPE);

	hipMemcpy(vradint, vradint_d, size_grid*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(vthetaint, vthetaint_d, size_grid*sizeof(float), hipMemcpyDeviceToHost);


	hipFree(press_d );
	hipFree(rho_d );
  hipFree(vradint_d);
  hipFree(pot_d);
  hipFree(invdiffRmed_d);
  hipFree(invRinf_d);
  hipFree(Rinf_d);
  hipFree(vrad_d);
  hipFree(vthetaint_d);
  hipFree(vtheta_d);
  hipFree(Rmed_d);

  /*if (SelfGravity){
    selfgravityupdate = YES;
    compute_selfgravity(Rho, VradInt, VthetaInt, dt, selfgravityupdate);
  }
  ComputeViscousTerms (VradInt, VthetaInt, Rho);*/
  UpdateVelocitiesWithViscosity(vradint, vthetaint, rho, dt);
/*
  if (!Evanescent) ApplySubKeplerianBoundary(VthetaInt);
*/
  FILE *f;
  f = fopen("datos.txt","w");

  for (int i = 0; i < size_grid; i++)
  {
    fprintf(f, "%f\n",vthetaint[i] );
  }

  fclose(f);
	return EXIT_SUCCESS;
}
