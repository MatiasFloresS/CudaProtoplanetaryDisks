#include "hip/hip_runtime.h"
#include "Main.cuh"

extern int blocksize2, size_grid, NRAD, NSEC;

extern float *GLOBAL_bufarray;
extern float *gridfield_d, *GLOBAL_bufarray_d, *axifield_d, *SG_Accr_d, *GLOBAL_AxiSGAccr_d;

extern float ASPECTRATIO, TRANSITIONWIDTH, TRANSITIONRATIO, TRANSITIONRADIUS;
extern float PhysicalTime, PhysicalTimeInitial, LAMBDADOUBLING;

extern dim3 dimGrid, dimBlock, dimGrid4;

__global__ void Substep1Kernel (float *Pressure, float *Dens, float *VradInt, float *invdiffRmed, float *Potential,
   float *Rinf, float *invRinf, float *Vrad, float *VthetaInt, float *Vtheta, float *Rmed, float dt,
   int nrad, int nsec, float OmegaFrame, int ZMPlus, float IMPOSEDDISKDRIFT, float SIGMASLOPE,
   float *powRmed)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;
  float gradp, gradphi, vt2;

  // i=1->nrad , j=0->nsec
  if (i > 0 && i<nrad && j<nsec){
    gradp = 2.0*(Pressure[i*nsec + j] - Pressure[(i-1)*nsec + j])/(Dens[i*nsec + j] + Dens[(i-1)*nsec + j])*invdiffRmed[i];
    gradphi = (Potential[i*nsec + j]-Potential[(i-1)*nsec + j])*invdiffRmed[i];
    vt2 = Vtheta[i*nsec + j] + Vtheta[(i-1)*nsec + j] + Vtheta[i*nsec + (j+1)%nsec] + Vtheta[(i-1)*nsec + (j+1)%nsec];
    vt2 = vt2/4.0+Rinf[i]*OmegaFrame;
    vt2 = vt2*vt2;
    VradInt[i*nsec + j] = dt*(-gradp - gradphi + vt2*invRinf[i]);
    if ( i == 5 && j == 5)
    {
      printf("dt %g\n", dt);
      printf("%g\n", gradp);
      printf("%g\n", gradphi);
      printf("%g\n", vt2);
      printf("%g\n", invRinf[i]);
      printf("%g\n", VradInt[i*nsec + j]);
    }
  }


  // i=0->nrad ,   j=0->nsec
  if (i<nrad && j<nsec){
    gradp = 2.0*(Pressure[i*nsec + j] - Pressure[i*nsec + ((j-1)+nsec)%nsec])/(Dens[i*nsec +j] +Dens[i*nsec + ((j-1)+nsec)%nsec]) \
    *1.0/(2.0*M_PI/nsec*Rmed[i]);

    if (ZMPlus) gradp *= 1; //gradp *= SG_aniso_coeff;  Definir mas adelante SG_aniso_coeff

    gradphi = (Potential[i*nsec+ j] - Potential[i*nsec + ((j-1)+nsec)%nsec])*1.0/(2.0*M_PI/nsec*Rmed[i]);
    VthetaInt[i*nsec + j] = Vtheta[i*nsec + j]- dt*(gradp+gradphi);
    VthetaInt[i*nsec + j] += dt*IMPOSEDDISKDRIFT*0.5*powRmed[i];
  }
}

__global__ void Substep3Kernel (float *Dens, float *Qplus, float *viscosity_array, float *TAURR, float *TAURP,float *TAUPP,
  float *DivergenceVelocity, int nrad, int nsec, float *Rmed, int Cooling, float *EnergyNew, float dt, float *EnergyMed,
  float *SigmaMed, float *CoolingTimeMed, float *Energy, float ADIABATICINDEX, float *QplusMed)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;
  float den, num;

  i+= 1;

  if (i<nrad && j<nsec){
    if (viscosity_array[i] != 0.0){
      Qplus[i*nsec + j] = 0.5/viscosity_array[i]/Dens[i*nsec + j]*(TAURR[i*nsec + j]*TAURR[i*nsec + j] + \
        TAURP[i*nsec + j]* TAURP[i*nsec + j] + TAUPP[i*nsec + j]*TAUPP[i*nsec + j]);

      Qplus[i*nsec + j] += (2.0/9.0)*viscosity_array[i]*Dens[i*nsec + j]*DivergenceVelocity[i*nsec + j]* \
        DivergenceVelocity[i*nsec + j];
    }
    else Qplus[i*nsec + j] = 0.0;

  }
  __syncthreads();
  i-= 1;
  if (i==0 && j<nsec){
    /* We calculate the heating source term Qplus for i=0 */

    if (viscosity_array[i] != 0) {
      /* power-law extrapolation */
      Qplus[i*nsec + j] = Qplus[(i+1)*nsec + j]*expf(logf(Qplus[(i+1)*nsec + j]/Qplus[(i+2)*nsec + j]) * \
        logf(Rmed[i]/Rmed[i+1]) / logf(Rmed[i+1]/Rmed[i+2]));
    }
    else Qplus[i*nsec + j] = 0.0;
  }

  __syncthreads();

  if (i<nrad && j<nsec){
    if (!Cooling){
      num = dt*Qplus[i*nsec + j] + Energy[i*nsec + j];
      den = 1.0+(ADIABATICINDEX-1.0)*dt*DivergenceVelocity[i*nsec + j];
      EnergyNew[i*nsec + j] = num/den;
    }
    else{
      num = EnergyMed[i]*dt*Dens[i*nsec + j]/SigmaMed[i] + CoolingTimeMed[i]*Energy[i*nsec + j] + \
        dt*CoolingTimeMed[i]*(Qplus[i*nsec + j]-QplusMed[i]*Dens[i*nsec + j]/SigmaMed[i]);

      den = dt + CoolingTimeMed[i] + (ADIABATICINDEX-1.0)*dt*CoolingTimeMed[i]*DivergenceVelocity[i*nsec + j];
      EnergyNew[i*nsec + j] = num/den;
    }
  }
}

__global__ void UpdateVelocitiesKernel (float *VthetaInt, float *VradInt, float *invRmed, float *Rmed, float *Rsup,
  float *Rinf, float *invdiffRmed, float *invdiffRsup, float *Dens, float *invRinf, float *TAURR, float *TAURP,
  float *TAUPP, float DeltaT, int nrad, int nsec)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  i+=1;
  /* Now we can update velocities
     with the viscous source term
     of Navier-Stokes equation */

  /* vtheta first */
  if (i<nrad-1 && j<nsec){
    VthetaInt[i*nsec +j] += DeltaT*invRmed[i]*((Rsup[i]*TAURP[(i+1)*nsec+ j]-Rinf[i]*TAURP[i*nsec +j])*invdiffRsup[i] + \
    (TAUPP[i*nsec +j]-TAUPP[i*nsec + ((j-1)+nsec)%nsec])*1.0/(2.0*M_PI/nsec) + 0.5*(TAURP[i*nsec + j] + TAURP[(i+1)*nsec +j]))/ \
    (0.5*(Dens[i*nsec +j]+Dens[i*nsec + ((j-1)+nsec)%nsec]));
  }

  /* now vrad */
  if (i<nrad && j<nsec){
    VradInt[i*nsec +j] += DeltaT*invRinf[i]*((Rmed[i]*TAURR[i*nsec +j]- Rmed[i-1]*TAURR[(i-1)*nsec + j])*invdiffRmed[i] + \
    (TAURP[i*nsec + (j+1)%nsec] -TAURP[i*nsec + j])*1.0/(2.0*M_PI/nsec) - 0.5*(TAURP[i*nsec +j] + TAURP[(i-1)*nsec + j]))/ \
    (0.5*(Dens[i*nsec +j] + Dens[(i-1)*nsec + j]));

  }
}

__global__ void InitComputeAccelKernel (float *CellAbscissa, float *CellOrdinate, float *Rmed, int nsec, int nrad)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad && j<nsec){
    CellAbscissa[i*nsec+j] = Rmed[i] * cosf((2.0*M_PI*(float)j)/(float)nsec);
    CellOrdinate[i*nsec+j] = Rmed[i] * sinf((2.0*M_PI*(float)j)/(float)nsec);
  }
}

__global__ void ComputeSoundSpeedKernel (float *SoundSpeed, float *Dens, float *Rmed, float *Energy, int nsec, int nrad,
  int Adiabatic, float ADIABATICINDEX, float FLARINGINDEX, float ASPECTRATIO, float TRANSITIONWIDTH,
  float TRANSITIONRADIUS, float TRANSITIONRATIO, float PhysicalTime, float PhysicalTimeInitial, float LAMBDADOUBLING)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float AspectRatio;
  if (i<nrad && j<nsec){
    if (!Adiabatic){
      AspectRatio = AspectRatioDevice(Rmed[i], ASPECTRATIO, TRANSITIONWIDTH, TRANSITIONRADIUS, TRANSITIONRATIO,
        PhysicalTime, PhysicalTimeInitial, LAMBDADOUBLING);
      SoundSpeed[i*nsec + j] = AspectRatio*sqrtf(G*1.0/Rmed[i])*powf(Rmed[i], FLARINGINDEX);
    }
    else SoundSpeed[i*nsec + j] = sqrtf(ADIABATICINDEX*(ADIABATICINDEX-1.0)*Energy[i*nsec + j]/Dens[i*nsec + j]);
  }
}

__global__ void ComputePressureFieldKernel (float *SoundSpeed, float *Dens, float *Pressure, int Adiabatic, int nrad,
  int nsec, float ADIABATICINDEX, float *Energy) /* LISTO */
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad && j<nsec){
    if (!Adiabatic) Pressure[i*nsec + j] = Dens[i*nsec + j]*SoundSpeed[i*nsec + j]*SoundSpeed[i*nsec + j];
    /* Since SoundSpeed is not update from initialization, cs remains axisymmetric*/
    else Pressure[i*nsec + j] = (ADIABATICINDEX-1.0)*Energy[i*nsec + j];
  }
}


__global__ void ComputeTemperatureFieldKernel (float *Dens, float *Temperature, float *Pressure, float *Energy,
  float ADIABATICINDEX, int Adiabatic, int nsec, int nrad) /* LISTO */
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad && j<nsec){
    if (!Adiabatic) Temperature[i*nsec + j] = MU/R*Pressure[i*nsec + j]/Dens[i*nsec + j];
    else Temperature[i*nsec + j] = MU/R*(ADIABATICINDEX-1.0)*Energy[i*nsec + j]/Dens[i*nsec + j];
  }
}


/* LISTO */
__global__ void InitLabelKernel (float *Label, float xp, float yp, float rhill, float *Rmed, int nrad, int nsec)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad && j<nsec){
    float distance, angle, x, y;
    angle = (float)j / (float)nsec*2.0*M_PI;
    x = Rmed[i] * cosf(angle);
    y = Rmed[i] * sinf(angle);
    distance = sqrtf((x - xp) * (x - xp) + (y - yp)*(y -yp));

    if (distance < rhill) Label[i*nsec + j] = 1.0;
    else Label[i*nsec + j] = 0.0;

  }
}

__global__ void CircumPlanetaryMassKernel (float *Dens, float *Surf, float *CellAbscissa, float *CellOrdinate,
  float xpl, float ypl, int nrad, int nsec, float HillRadius, float *mdcp0) /* LISTA */
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float dist;

  if (i<nrad && j<nsec){
    dist = sqrtf((CellAbscissa[i*nsec + j]-xpl)*(CellAbscissa[i*nsec + j]-xpl) + (CellOrdinate[i*nsec + j]-ypl) * \
    (CellOrdinate[i*nsec + j]-ypl));
    if (dist < HillRadius) mdcp0[i*nsec + j] =  Surf[i]* Dens[i*nsec + j];
    else mdcp0[i*nsec + j] = 0.0;
  }
}

template <bool nIsPow2>
__global__ void DeviceReduceKernel (float *g_idata, float *g_odata, unsigned int n)
{
  extern __shared__ float sdata[];

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int blockSize = blockDim.x;
  unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  float mySum = 0.f;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (i < n){
    mySum += g_idata[i];

    // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
    if (nIsPow2 || i + blockSize < n)
      mySum += g_idata[i+blockSize];

    i += gridSize;
  }
  // each thread puts its local sum into shared memory
  sdata[tid] = mySum;
  __syncthreads();

  // do reduction in shared mem
  if ((blockSize >= 512) && (tid < 256)){
    sdata[tid] = mySum = mySum + sdata[tid + 256];
  }
  __syncthreads();

  if ((blockSize >= 256) &&(tid < 128)){
    sdata[tid] = mySum = mySum + sdata[tid + 128];
  }
   __syncthreads();

  if ((blockSize >= 128) && (tid <  64)){
     sdata[tid] = mySum = mySum + sdata[tid +  64];
  }
  __syncthreads();

  #if (__CUDA_ARCH__ >= 300 )
    if (tid < 32){
      // Fetch final intermediate sum from 2nd warp
      if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2){
          mySum += __shfl_down(mySum, offset);
        }
      }
  #else
    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32)){
      sdata[tid] = mySum = mySum + sdata[tid + 32];
    }
    __syncthreads();

    if ((blockSize >=  32) && (tid < 16)){
      sdata[tid] = mySum = mySum + sdata[tid + 16];
    }
    __syncthreads();

    if ((blockSize >=  16) && (tid <  8)){
      sdata[tid] = mySum = mySum + sdata[tid +  8];
    }
    __syncthreads();

    if ((blockSize >=   8) && (tid <  4)){
      sdata[tid] = mySum = mySum + sdata[tid +  4];
    }
    __syncthreads();

    if ((blockSize >=   4) && (tid <  2)){
      sdata[tid] = mySum = mySum + sdata[tid +  2];
    }
    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1)){
      sdata[tid] = mySum = mySum + sdata[tid +  1];
    }
    __syncthreads();
  #endif

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = mySum;
}

__host__ long NearestPowerOf2 (long n)
{
  if(!n) return n; //(0 ==2^0)

  int x=1;
  while (x < n){
    x<<=1;
  }
  return x;
}

__host__ bool IsPow2 (unsigned int x)
{
  return ((x&(x-1)==0));
}

__host__ float DeviceReduce (float *in, int N)
{
  float *device_out;
  gpuErrchk(hipMalloc(&device_out, sizeof(float)*1024));
  gpuErrchk(hipMemset(device_out, 0, sizeof(float)*1024));

  int threads = 32;
  int blocks = min((int(NearestPowerOf2(N)) + threads - 1) / threads, 1024);
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

  bool isPower2 = IsPow2(N);
  if(isPower2){
    DeviceReduceKernel<true><<<blocks, threads, smemSize>>>(in, device_out, N);
    gpuErrchk(hipDeviceSynchronize());
  }
  else{
    DeviceReduceKernel<false><<<blocks, threads, smemSize>>>(in, device_out, N);
    gpuErrchk(hipDeviceSynchronize());
  }

  float *h_odata = (float *) malloc(blocks*sizeof(float));
  float sum = 0.0;

  gpuErrchk(hipMemcpy(h_odata, device_out, blocks * sizeof(float),hipMemcpyDeviceToHost));
  for (int i=0; i<blocks; i++){
    sum += h_odata[i];
  }
  hipFree(device_out);
  free(h_odata);
	return sum;
}


/* LISTA */
__global__ void MultiplyPolarGridbyConstantKernel (float *Dens, int nrad, int nsec, float ScalingFactor)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad+1 && j<nsec)
    Dens[i*nsec + j] *= ScalingFactor;
}

__global__ void Substep2Kernel (float *Dens, float *VradInt, float *VthetaInt, float *TemperInt, int nrad,
  int nsec, float CVNR, float *invdiffRmed, float *invdiffRsup, float *DensInt, int Adiabatic, float *Rmed,
  float dt, float *VradNew, float *VthetaNew, float *Energy, float *EnergyInt)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float dv;
  if (i<nrad && j<nsec){
    dv = VradInt[(i+1)*nsec + j] - VradInt[i*nsec + j];

    if (dv < 0.0) DensInt[i*nsec + j] = CVNR*CVNR*Dens[i*nsec + j]*dv*dv;
    else DensInt[i*nsec + j] = 0.0;

    dv = VthetaInt[i*nsec + (j+1)%nsec] - VthetaInt[i*nsec + j];
    if (dv < 0.0) TemperInt[i*nsec + j] = CVNR*CVNR*Dens[i*nsec + j]*dv*dv;
    else TemperInt[i*nsec + j] = 0.0;
  }
  i+=1;

  if (i<nrad && j<nsec){
    VradNew[i*nsec + j] = VradInt[i*nsec + j] - dt*2.0/(Dens[i*nsec + j] + Dens[(i-1)*nsec + j])*(DensInt[i*nsec + j] - DensInt[(i-1)*nsec + j]) *\
      invdiffRmed[i];
  }
  i-=1;

  if (i<nrad && j<nsec){
    VthetaNew[i*nsec + j] = VthetaInt[i*nsec + j] - dt*2.0/(Dens[i*nsec + j] + Dens[i*nsec + ((j-1)+nsec)%nsec])*(DensInt[i*nsec + j] - \
      DensInt[(i-1)*nsec + j])* 1.0/(2.0*M_PI*Rmed[i]/nsec);
  }
  if (Adiabatic){
    i+=1;

    if (i<nrad && j<nsec){
      EnergyInt[i*nsec + j] = Energy[i*nsec + j] - dt*DensInt[i*nsec + j]*(VradInt[i*nsec + j+1] - VradInt[i*nsec + j])*invdiffRsup[i] - \
      dt*TemperInt[i*nsec + j]*(VthetaInt[i*nsec + (j+1)%nsec] - VthetaInt[i*nsec + j])* 1.0/(2.0*M_PI*Rmed[i]/nsec);
    }
  }
}

__global__ void OpenBoundaryKernel (float *Vrad, float *Dens, float *Energy, int nsec, float *SigmaMed)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = 1;

  if(j < nsec){
    Dens[(i-1)*nsec + j] = Dens[i*nsec + j]; // copy first ring into ghost ring
    Energy[(i-1)*nsec + j] = Energy[i*nsec + j];
    if (Vrad[(i+1)*nsec + j] > 0.0 || (Dens[i*nsec + j] < SigmaMed[0]))
      Vrad[i*nsec + j] = 0.0; // we just allow outflow [inwards]
    else
      Vrad[i*nsec +j] = Vrad[(i+1)*nsec + j];
  }
}

__global__ void ReduceCsKernel (float *SoundSpeed, float *cs0, float *cs1, float *csnrm1, float *csnrm2, int nsec, int nrad)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i=0;

  if(j<nsec){
    cs0[j] = SoundSpeed[i*nsec +j];
    cs1[j] = SoundSpeed[(i+1)*nsec +j];
  }
  i = nrad-1;
  if(j<nsec){
    csnrm2[j] = SoundSpeed[(i-1)*nsec +j];
    csnrm1[j] = SoundSpeed[i*nsec +j];
  }
}

__global__ void ReduceMeanKernel (float *Dens, float *Energy, int nsec, float *mean_dens, float *mean_energy,
  float *mean_dens2, float *mean_energy2, int nrad)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = 0;

  if(j<nsec){
    mean_dens[j] = Dens[i*nsec+ j];
    mean_energy[j] = Energy[i*nsec +j];
  }
  i = nrad-1;
  if(j<nsec){
    mean_dens2[j] = Dens[i*nsec + j];
    mean_energy2[j] = Energy[i*nsec + j];
  }
}

__global__ void NonReflectingBoundaryKernel (float *Dens, float *Energy, int i_angle, int nsec, float *Vrad, float *SoundSpeed,
  float SigmaMed, int nrad, float SigmaMed2, int i_angle2)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = 1;
  float Vrad_med;

  if (j<nsec){

    if(j+i_angle >= nsec){
      Dens[j+i_angle - nsec] = Dens[i*nsec + j];
      Energy[j+i_angle - nsec] = Energy[i*nsec + j];
    }

    else if(j+i_angle < 0){
      Dens[j+i_angle + nsec] = Dens[i*nsec + j];
      Energy[j+i_angle + nsec] = Energy[i*nsec + j];
    }
    else{
      Dens[j+i_angle] = Dens[i*nsec + j];
      Energy[j+i_angle] = Energy[i*nsec + j];
    }
    __syncthreads();

    Vrad_med = -SoundSpeed[i*nsec + j]*(Dens[i*nsec + j]-SigmaMed)/SigmaMed;
    Vrad[i*nsec + j] = 2.*Vrad_med-Vrad[(i+1)*nsec + j];
  }

  i = nrad-1;
  if (j<nsec){
    if (j-i_angle2 >= nsec){
      Dens[i*nsec + j] = Dens[j-i_angle2 + (i-2)*nsec ];
      Energy[i*nsec + j] = Energy[j-i_angle2 + (i-2)*nsec ];
    }
    else if (j-i_angle2 < 0){
      Dens[i*nsec + j] = Dens[j-i_angle2 + i*nsec];
      Energy[i*nsec + j] = Energy[j-i_angle2 + i*nsec];
    }
    else{
      Dens[i*nsec + j] = Dens[j-i_angle2 + (i-1)*nsec];
      Energy[i*nsec + j] = Energy[j-i_angle2 + (i-1)*nsec];
    }
    __syncthreads();

    Vrad_med = SoundSpeed[i*nsec + j]*(Dens[(i-1)*nsec + j]-SigmaMed2)/SigmaMed2;
    Vrad[i*nsec + j] = 2.*Vrad_med - Vrad[(i-1)*nsec + j];
  }
}

__global__ void MinusMeanKernel (float *Dens, float *Energy, float SigmaMed, float mean_dens_r, float mean_dens_r2,
  float mean_energy_r,float mean_energy_r2, float EnergyMed, int nsec, int nrad, float SigmaMed2, float EnergyMed2)
{
    int j = threadIdx.x + blockDim.x*blockIdx.x;
    int i = 0;
    if (j< nsec){
      Dens[i*nsec + j] += SigmaMed - mean_dens_r;
      Energy[i*nsec + j] += EnergyMed - mean_energy_r;
    }

    i = nrad-1;
    if (j < nsec){
      Dens[i*nsec + j] += SigmaMed2 - mean_dens_r2;
      Energy[i*nsec + j] += EnergyMed2 - mean_energy_r2;
    }
  }

__global__ void Make1DprofileKernel (float *gridfield, float *axifield, int nsec, int nrad)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j;

  if (i < nrad){
    float sum = 0.0;

    for (j = 0; j < nsec; j++)
      sum += gridfield[i*nsec + j];

    axifield[i] = sum/(float)nsec;
  }
}


__host__ void Make1Dprofile (int option)
{

  /* GLOBAL AxiSGAccr option */
  if (option == 1){
    gpuErrchk(hipMemcpy(gridfield_d, SG_Accr_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
    //gpuErrchk(hipMemcpy(GLOBAL_AxiSGAccr_d, axifield_d, NRAD*sizeof(float), hipMemcpyDeviceToHost));

  }
  /* GLOBAL_bufarray option */
  if (option == 2){
    //gpuErrchk(hipMemcpy(gridfield_d, SG_Accr_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
    //gpuErrchk(hipMemcpy(GLOBAL_AxiSGAccr_d, axifield_d, NRAD*sizeof(float), hipMemcpyDeviceToHost));
  }

  Make1DprofileKernel<<<dimGrid4, dimBlock>>>(gridfield_d, axifield_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());

}

/* LISTO */
__global__ void InitGasVelocitiesKernel (float *viscosity_array, int nsec, int nrad, int SelfGravity, float *Rmed,
  float ASPECTRATIO, float FLARINGINDEX, float SIGMASLOPE, int CentrifugalBalance, float *Vrad, float *Vtheta,
  float ViscosityAlpha, float IMPOSEDDISKDRIFT, float SIGMA0, float *SigmaInf, float OmegaFrame, float *Rinf, float *vt_cent)
{
    int j = threadIdx.x + blockDim.x*blockIdx.x;
    int i = threadIdx.y + blockDim.y*blockIdx.y;

    float omega, r, ri;

    if (i < nrad+1 && j < nsec){
      if (i == nrad){
        r = Rmed[nrad - 1];
        ri = Rinf[nrad - 1];
      }
      else{
        r = Rmed[i];
        ri = Rinf[i];
      }

      if (!SelfGravity){
        omega = sqrtf(G*1.0/r/r/r);
        Vtheta[i*nsec + j] = omega*r*sqrtf(1.0-powf(ASPECTRATIO,2.0)*powf(r,2.0*FLARINGINDEX)* \
        (1.+SIGMASLOPE-2.0*FLARINGINDEX));
      }
      Vtheta[i*nsec + j] -= OmegaFrame*r;

      if (CentrifugalBalance) Vtheta[i*nsec + j] = vt_cent[i];

      if (i == nrad) Vrad[i*nsec + j] = 0.0;
      else Vrad[i*nsec + j] = IMPOSEDDISKDRIFT*SIGMA0/SigmaInf[i]/ri;

      if (ViscosityAlpha) Vrad[i*nsec + j] -= 3.0*viscosity_array[i]/r*(-SIGMASLOPE+2.0*FLARINGINDEX+1.0);
      else Vrad[i*nsec + j] -= 3.0*viscosity_array[i]/r*(-SIGMASLOPE+.5);

      __syncthreads();

      if (j < nsec) Vrad[j] = Vrad[nrad*nsec + j] = 0.0;
    }
  }



__global__ void ComputeForceKernel (float *CellAbscissa, float *CellOrdinate, float *Surf, float *Dens, float x,
  float y, float rsmoothing, float *forcesxi, float *forcesyi, float *forcesxo, float *forcesyo, int nsec,
  int nrad, float a, float *Rmed, int dimfxy, float rh)
{

    int j = threadIdx.x + blockDim.x*blockIdx.x;
    int i = threadIdx.y + blockDim.y*blockIdx.y;
    int k;
    float cellmass, dx, dy, d2, InvDist3, dist2, distance, resultxi, resultyi;
    float resultxo, resultyo, hillcutfactor, hill_cut;

    if (i<nrad && j<nsec){
      cellmass = Surf[i] * Dens[i*nsec + j];
      dx = CellAbscissa[i*nsec + j] - x;
      dy = CellOrdinate[i*nsec + j] - y;
      d2 = dx*dx + dy*dy;
      dist2 = d2 + rsmoothing*rsmoothing;
      distance = sqrtf(dist2);
      InvDist3 = 1.0/dist2/distance;

      for (k = 0; k < dimfxy; k++){
        hillcutfactor =  (float) k / (float)(dimfxy-1);
        if (k != 0){
          rh *= hillcutfactor;
          hill_cut = 1.-expf(-d2/(rh*rh));
        }
        else hill_cut = 1.;

        if (Rmed[i] < a){
          resultxi = G * cellmass * dx * InvDist3 * hill_cut;
          resultyi = G * cellmass * dy * InvDist3 * hill_cut;
          atomicAdd(&(forcesxi[k]), resultxi);
          atomicAdd(&(forcesyi[k]), resultyi);
        }
        else{
          resultxo = G * cellmass * dx * InvDist3 * hill_cut;
          resultyo = G * cellmass * dy * InvDist3 * hill_cut;
          atomicAdd(&(forcesxo[k]), resultxo);
          atomicAdd(&(forcesyo[k]), resultyo);
        }
      }
    }
  }

__global__ void ViscousTermsKernel (float *Vradial, float *Vazimutal , float *DRR, float *DPP, float *DivergenceVelocity,
  float *DRP, float *invdiffRsup, float *invRmed, float *Rsup, float *Rinf, float *invdiffRmed, int nrad,
  int nsec, float *TAURR, float *TAUPP, float *dens, float *viscosity_array, float *TAURP, float *invRinf)
{
   int j = threadIdx.x + blockDim.x*blockIdx.x;
   int i = threadIdx.y + blockDim.y*blockIdx.y;

   float dphi, invdphi, onethird;
   if (i<nrad && j<nsec){ /* Drr, Dpp and divV computation */
     dphi = 2.0*M_PI/(float)nsec;
     invdphi = 1.0/dphi;
     onethird = 1.0/3.0;

     DRR[i*nsec + j] = (Vradial[(i+1)*nsec + j] - Vradial[i*nsec + j])*invdiffRsup[i];
     DPP[i*nsec + j] = (Vazimutal[i*nsec + (j+1)%nsec] - Vazimutal[i*nsec + j])*invdphi*invRmed[i]+0.5* \
       (Vradial[(i+1)*nsec + j]+Vradial[i*nsec + j])*invRmed[i];
     DivergenceVelocity[i*nsec + j] = (Vradial[i*nsec + (j+1)]*Rsup[i]-Vradial[i*nsec + j]*Rinf[i])*invdiffRsup[i] * \
       invRmed[i];
     DivergenceVelocity[i*nsec + j] += (Vazimutal[i*nsec + (j+1)%nsec]-Vazimutal[i*nsec + j])*invdphi*invRmed[i];

     if (i > 0) DRP[i*nsec + j] = 0.5*(Rinf[i]*(Vazimutal[i*nsec + j]*invRmed[i]-Vazimutal[(i-1)*nsec + j])*invRmed[i-1])* \
          invdiffRmed[i] + (Vradial[i*nsec + j]-Vradial[i*nsec + (j-1)%nsec])*invdphi*invRinf[i];
   }

   if (i<nrad && j<nsec){ /* TAUrr and TAUpp computation */
     TAURR[i*nsec + j] = 2.0*dens[i*nsec + j]*viscosity_array[i]*(DRR[i*nsec + j]-onethird*DivergenceVelocity[i*nsec + j]);
     TAUPP[i*nsec + j] = 2.0*dens[i*nsec + j]*viscosity_array[i]*(DPP[i*nsec + j]-onethird*DivergenceVelocity[i*nsec + j]);

     if (i > 0) TAURP[i*nsec + j] = 2.0*0.25*(dens[i*nsec + j] + dens[(i-1)*nsec + j] + dens[(i-1)*nsec + ((j-1)+nsec)%nsec])* \
         viscosity_array[i]*DRP[i*nsec + j];
   }
 }

__global__ void LRMomentaKernel (float *RadMomP, float *RadMomM, float *ThetaMomP, float *ThetaMomM, float *Dens,
  float *Vrad, float *Vtheta, int nrad, int nsec, float *Rmed, float OmegaFrame)
{
   int j = threadIdx.x + blockDim.x*blockIdx.x;
   int i = threadIdx.y + blockDim.y*blockIdx.y;

   if (i<nrad && j<nsec){
     RadMomP[i*nsec + j] = Dens[i*nsec + j] * Vrad[(i)*nsec + j]; // (i+1)*nsec
     RadMomM[i*nsec + j] = Dens[i*nsec + j] * Vrad[i*nsec + j];
     /* it is the angular momentum -> ThetaMomP */
     ThetaMomP[i*nsec + j] = Dens[i*nsec + j] * (Vtheta[i*nsec + (j+1)%nsec]+Rmed[i]*OmegaFrame)*Rmed[i];
     ThetaMomM[i*nsec + j] = Dens[i*nsec + j] * (Vtheta[i*nsec + j]+Rmed[i]*OmegaFrame)*Rmed[i];
   }
 }

__global__ void ExtQtyKernel (float *ExtLabel, float *Dens, float *Label, int nsec, int nrad)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad && j<nsec)
    ExtLabel[i*nsec + j] = Dens[i*nsec + j]*Label[i*nsec + j];
}

__global__ void StarRadKernel (float *Qbase, float *Vrad, float *QStar, float dt, int nrad, int nsec,
  float *invdiffRmed, float *Rmed, float *dq)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float dqm, dqp;

  if (i<nrad && j<nsec){
    if (i == 0 || i == nrad-1) dq[j*nrad + i] = 0.0;
    else {
      dqm = (Qbase[i*nsec + j] - Qbase[(i-1)*nsec + j])*invdiffRmed[i];
      dqp = (Qbase[(i+1)*nsec + j] - Qbase[i*nsec + j])*invdiffRmed[i+1];

      if (dqp * dqm > 0.0) dq[j*nrad + i] = 2.0*dqp*dqm/(dqp+dqm);
      else dq[j*nrad + i] = 0.0;
    }
    __syncthreads();

    if (Vrad[i*nsec + j] > 0.0) // aca asumen vrad = 0 para i=0?
      QStar[i*nsec + j] = Qbase[(i-1)*nsec + j] + (Rmed[i]-Rmed[i-1]-Vrad[i*nsec + j]*dt)*0.5*dq[j*nrad + i-1];
    else
      QStar[i*nsec + j] = Qbase[i*nsec + j]-(Rmed[i+1]-Rmed[i]+Vrad[i*nsec + j]*dt)*0.5*dq[j*nrad + i];
  }
  if (j<nsec)
    QStar[j] = QStar[j+nsec*nrad] = 0.0;
}

__global__ void ComputeFFTKernel (float *Radii, hipfftComplex *SGP_Kr, hipfftComplex *SGP_Kt, float SGP_eps, int nrad, int nsec,
hipfftComplex *SGP_Sr, hipfftComplex *SGP_St, float *Dens, float *Rmed, float *Kr_aux, float *Kt_aux)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;
  float u, cosj, sinj, coshu, expu, den_SGP_K, theta, base;
  float a, var, radii;

  if (i<2*nrad && j<nsec){
    SGP_Kr[i*nsec + j].x = Kr_aux[i*nsec + j];
    SGP_Kr[i*nsec + j].y = 0.;

    SGP_Kt[i*nsec + j].x = Kt_aux[i*nsec + j];
    SGP_Kt[i*nsec + j].y = 0.;

    SGP_Sr[i*nsec + j].y = 0.;
    SGP_St[i*nsec + j].y = 0.;

    if (i<nrad){
      var = Dens[i*nsec + j] * sqrtf(Rmed[i]/Rmed[0]);
      SGP_Sr[i*nsec + j].x = var;
      SGP_St[i*nsec + j].x = var*(Rmed[i]/Rmed[0]);
    }
    else{
      SGP_Sr[i*nsec + j].x = 0.;
      SGP_St[i*nsec + j].x = 0.;
    }
  }
}

__global__ void ComputeConvolutionKernel (hipfftComplex *Gr, hipfftComplex *Gphi, hipfftComplex *SGP_Kr, hipfftComplex *SGP_Kt,
  hipfftComplex *SGP_Sr, hipfftComplex *SGP_St, int nsec, int nrad)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<2*nrad && j<nsec){
    Gphi[i*nsec + j].x = -G*(SGP_Kt[i*nsec + j].x * SGP_St[i*nsec + j].x - \
      SGP_Kt[i*nsec + j].y * SGP_St[i*nsec + j].y);

    Gphi[i*nsec + j].y = -G*(SGP_Kt[i*nsec + j].x * SGP_St[i*nsec + j].y + \
      SGP_Kt[i*nsec + j].y * SGP_St[i*nsec + j].x);

    Gr[i*nsec + j].x = -G*(SGP_Kr[i*nsec + j].x * SGP_Sr[i*nsec + j].x - \
      SGP_Kr[i*nsec + j].y * SGP_Sr[i*nsec + j].y);

    Gr[i*nsec + j].y = -G*(SGP_Kr[i*nsec + j].x * SGP_Sr[i*nsec + j].y + \
      SGP_Kr[i*nsec + j].y *SGP_Sr[i*nsec + j].x);
  }
}

__global__ void ComputeSgAccKernel (float *SG_Accr, float *SG_Acct, float *Dens , float SGP_rstep, float SGP_tstep,
  float SGP_eps, int nrad, int nsec, float *Rmed, hipfftComplex *Gr, hipfftComplex *Gphi)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float normaccr, normacct, divRmed;

  if (i<nrad && j<nsec){
    divRmed = Rmed[i]/Rmed[0];
    normaccr = SGP_rstep * SGP_tstep / ((float)(2*nrad) * (float)nsec);
    normacct = normaccr;
    normaccr /= sqrtf(divRmed);
    normacct /= (divRmed * sqrtf(divRmed));
    SG_Acct[i*nsec + j] = Gphi[i*nsec + j].x * normaccr;

    SG_Accr[i*nsec + j] = Gr[i*nsec + j].x * normaccr;
    SG_Accr[i*nsec + j] += G*Dens[i*nsec + j]*SGP_rstep*SGP_tstep / SGP_eps;
  }
}

__global__ void Update_sgvelocityKernel (float *Vradial, float *Vazimutal, float *SG_Accr, float *SG_Acct, float *Rinf,
  float *Rmed, float *invdiffRmed, float dt, int nrad, int nsec)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  int jm1, lm1;

  /* Here we update velocity fields to take into acount self-gravity */
  if (i<nrad && j<nsec){
    /* We compute VRAD - half-centered in azimuth - from centered-in-cell radial sg acceleration. */
    if (i > 0) Vradial[i*nsec + j] +=  dt*((Rinf[i] - Rmed[i-1]) * SG_Accr[i*nsec + j] + \
    (Rmed[i] - Rinf[i]) * SG_Accr[(i-1)*nsec + j]) *invdiffRmed[i]; // caso !SGZeroMode

    /* We compute VTHETA - half-centered in radius - from centered-in-cell azimutal sg acceleration. */
    Vazimutal[i*nsec + j] = 0.5 * dt * (SG_Acct[i*nsec + j] + SG_Acct[i*nsec + (j-1)%nsec]);
  }
}

__global__ void Azimutalvelocity_withSGKernel (float *Vtheta, float *Rmed, float FLARINGINDEX, float SIGMASLOPE,
  float ASPECTRATIO, float *GLOBAL_bufarray, int nrad, int nsec)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float omegakep, omega, invr;
  if (i<nrad && j<nsec){
    invr = 1./Rmed[i];
    omegakep = sqrtf(G*1.0*invr*invr*invr);
    omega = sqrtf(omegakep*omegakep* (1.0 - (1.+SIGMASLOPE-2.0*FLARINGINDEX)*powf(ASPECTRATIO,2.0)* \
      powf(Rmed[i],2.0*FLARINGINDEX)) - invr*GLOBAL_bufarray[i]);

    Vtheta[i*nsec + j] = Rmed[i]*omega;
  }
}

__global__ void CrashKernel (float *array, int nrad, int nsec, int Crash)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad && j<nsec){
    if (array[i*nsec + j] < 0.0)
      array[i*nsec + j] = 1.0;
    else
      array[i*nsec + j] = 0.0;
  }
}

__global__ void EvanescentBoundaryKernel(float *Rmed, float *Vrad, float *Vtheta, float *Energy, float *Dens,
  float *viscosity_array, float DRMIN, float DRMAX, int nrad, int nsec, float Tin,
  float Tout, float step, float SIGMASLOPE, float FLARINGINDEX, float *GLOBAL_bufarray, float OmegaFrame,
  float *SigmaMed, float *EnergyMed, int Adiabatic, int SelfGravity, float ASPECTRATIO, float TRANSITIONWIDTH,
  float TRANSITIONRADIUS, float TRANSITIONRATIO, float PhysicalTime, float PhysicalTimeInitial, float LAMBDADOUBLING)
{
    int j = threadIdx.x + blockDim.x*blockIdx.x;
    int i = threadIdx.y + blockDim.y*blockIdx.y;

    float damping, lambda, vtheta0, vrad0, energy0, dens0, AspectRatio;
    if (i<nrad && j<nsec){
      if ((Rmed[i] < DRMIN) || (Rmed[i] > DRMAX)){
        /* Damping operates only inside the wave killing zones */
        if(Rmed[i] < DRMIN){
          damping = (Rmed[i]-DRMIN)/(Rmed[0]-DRMIN);
          lambda = damping*damping*10.0*step/Tin;
        }
        if (Rmed[i] > DRMAX){
          damping = (Rmed[i]-DRMAX)/(Rmed[nrad-1]-DRMAX);
          lambda = damping*damping*10.0*step/Tout;
        }
        if(!SelfGravity){
          AspectRatio = AspectRatioDevice(Rmed[i], ASPECTRATIO, TRANSITIONWIDTH, TRANSITIONRADIUS, TRANSITIONRATIO,
            PhysicalTime, PhysicalTimeInitial, LAMBDADOUBLING);
          vtheta0 = sqrtf(G*1.0/Rmed[i] * (1.0 - (1.0+SIGMASLOPE-2.0*FLARINGINDEX)*powf(AspectRatio,2.0) * \
          powf(Rmed[i],2.0*FLARINGINDEX)));
        }
        if (SelfGravity){
          AspectRatio = AspectRatioDevice(Rmed[i], ASPECTRATIO, TRANSITIONWIDTH, TRANSITIONRADIUS, TRANSITIONRATIO,
            PhysicalTime, PhysicalTimeInitial, LAMBDADOUBLING);
          vtheta0 = sqrtf(G*1.0/Rmed[i] * (1.0 - (1.0+SIGMASLOPE-2.0*FLARINGINDEX)*powf(AspectRatio,2.0) * \
          powf(Rmed[i],2.0*FLARINGINDEX)) - Rmed[i]*GLOBAL_bufarray[i]);
        }
        /* this could be refined if CentrifugalBalance is used... */
        vtheta0 -= Rmed[i]*OmegaFrame;
        vrad0 = -3.0*viscosity_array[i]/Rmed[i]*(-SIGMASLOPE+.5);
        dens0 = SigmaMed[i];
        energy0 = EnergyMed[i];

        Vrad[i*nsec + j] = (Vrad[i*nsec + j] + lambda*vrad0)/(1.0+lambda);
        Vtheta[i*nsec + j] = (Vtheta[i*nsec + j] + lambda*vtheta0)/(1.0+lambda);
        Dens[i*nsec + j] = (Dens[i*nsec + j] + lambda*dens0)/(1.0+lambda);
        if (Adiabatic)
          Energy[i*nsec + j] = (Energy[i*nsec + j] + lambda*energy0)/(1.0+lambda);
      }
    }
}

__global__ void DivisePolarGridKernel (float *res, float *num, float *denom, int nrad, int nsec)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad && j<nsec) // i <= nrad
    res[i*nsec + j] = num[i*nsec + j]/(denom[i*nsec + j] + 1e-20);
}

__global__ void VanLeerRadialKernel (float *Rinf, float *Rsup, float *QRStar, float *DensStar, float *Vrad,
  float *LostByDisk, int nsec, int nrad, float dt, int OpenInner)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float varq, dtheta;

  if (i<nrad && j<nsec){
    dtheta = 2.0*M_PI/float(nsec);
    varq = dt*dtheta*Rinf[i]*QRStar[i*nsec + j]* DensStar[i*nsec + j]*Vrad[i*nsec + j];
    varq -= dt*dtheta*Rsup[i]*QRStar[(i+1)*nsec + j]* DensStar[(i+1)*nsec + j]*Vrad[(i+1)*nsec + j];
    if (i==0 && OpenInner)
      LostByDisk[j] = varq;
  }
}

__global__ void VanLeerThetaKernel (float *Rsup, float *Rinf, float *Surf, float dt, int nrad, int nsec,
  int UniformTransport, int *NoSplitAdvection, float *QRStar, float *DensStar, float *Vtheta, float *Qbase)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float dxrad, invsurf, varq;

  if (i<nrad && j<nsec){
    if ((UniformTransport == NO) || (NoSplitAdvection[i] == NO)){
      dxrad = (Rsup[i]-Rinf[i])*dt;
      invsurf = 1.0/Surf[i];
      varq = dxrad*QRStar[i*nsec + j]*DensStar[i*nsec + j]*Vtheta[i*nsec + j];
      varq -= dxrad*QRStar[i*nsec + (j+1)%nsec]*DensStar[i*nsec + (j+1)%nsec]*Vtheta[i*nsec + (j+1)%nsec];
      Qbase[i*nsec + j] += varq*invsurf;
    }
  }
}

__global__ void ComputeAverageThetaVelocitiesKernel(float *Vtheta, float *VMed, int nsec, int nrad)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;

  float moy = 0.0;
  if (i<nrad){
    for (int j = 0; j < nsec; j++)
      moy += Vtheta[i*nsec + j];

    VMed[i] = moy/(float)nsec;
  }
}

__global__ void ComputeResidualsKernel (float *VthetaRes, float *VMed, int nsec, int nrad)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad && j<nsec)
    VthetaRes[i*nsec + j] = -VMed[i];
}

__global__ void ComputeConstantResidualKernel (float *VMed, float *invRmed, int *Nshift, int *NoSplitAdvection,
  int nsec, int nrad, float dt, float *Vtheta, float *VthetaRes, float *Rmed, int FastTransport)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float maxfrac, Ntilde, Nround, invdt;
  int nitemp;

  if (i<nrad && j<nsec){
    if (FastTransport) maxfrac = 1.0;
    else maxfrac = 0.0;

    invdt = 1.0/dt;
    Ntilde = VMed[i]*invRmed[i]*dt*(float)nsec/2.0/M_PI;
    Nround = floor(Ntilde+0.5);
    nitemp = (int)Nround;
    Nshift[i] = (int)nitemp;

    Vtheta[i*nsec + j] = (Ntilde-Nround)*Rmed[i]*invdt*2.0*M_PI/(float)nsec;

    if (maxfrac < 0.5){
      NoSplitAdvection[i] = YES;
      VthetaRes[i*nsec + j] += Vtheta[i*nsec + j];
      Vtheta[i*nsec + j] = 0.0;
    }
    else{
      NoSplitAdvection[i] = NO;
    }
  }
}

__global__ void StarThetaKernel (float *Qbase, float *Rmed, float *Vtheta, float *QStar, int nrad, int nsec,
  float *dq, float dt)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float dxtheta, ksi, invdxtheta, dqp, dqm;
  if (i<nrad && j<nsec){
    if (i<nrad){
      dxtheta = 2.0*M_PI/(float)nsec*Rmed[i];
      invdxtheta = 1.0/dxtheta;
    }
    dqm = (Qbase[i*nsec + j] - Qbase[i*nsec + ((j-1)+nsec)%nsec]);
    dqp = (Qbase[i*nsec + (j+1)%nsec] - Qbase[i*nsec + j]);

    if (dqp * dqm > 0.0)
      dq[i*nsec + j] = dqp*dqm/(dqp+dqm)*invdxtheta;
    else
      dq[i*nsec + j] = 0.0;
    __syncthreads();

    ksi = Vtheta[i*nsec + j]*dt;

    if (ksi > 0.0)
      QStar[i*nsec + j] = Qbase[i*nsec + ((j-1)+nsec)%nsec]+(dxtheta-ksi)*dq[i*nsec + ((j-1)+nsec)%nsec];
    else
      QStar[i*nsec + j] = Qbase[i*nsec + j]-(dxtheta+ksi)*dq[i*nsec + j];
   }
}

__global__ void AdvectSHIFTKernel (float *array, float *TempShift, int nsec, int nrad, int *Nshift)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  int ji, modji;

  if (i<nrad && j<nsec){
    ji = j-Nshift[i];
    modji = ji%nsec;
    if (ji < 0) ji += nsec;

    TempShift[i*nsec + j] = array[i*nsec + modji];
    __syncthreads();

    array[i*nsec + j] = TempShift[i*nsec + j];
  }
}

__global__ void ComputeVelocitiesKernel (float *Vrad, float *Vtheta, float *Dens, float *Rmed, float *ThetaMomP,
  float *ThetaMomM, float *RadMomP, float *RadMomM, int nrad, int nsec, float OmegaFrame)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad && j<nsec){
    if (i == 0) Vrad[i*nsec + j] = 0.0;
    else {
      Vrad[i*nsec + j] = (RadMomP[(i-1)*nsec + j] + RadMomM[i*nsec + j])/(Dens[i*nsec + j] +
        Dens[(i-1)*nsec + j] + 1e-20);
      Vtheta[i*nsec + j] = (ThetaMomP[i*nsec + ((j-1)+nsec)%nsec] + ThetaMomM[i*nsec + j])/(Dens[i*nsec + j] +
        Dens[i*nsec + ((j-1)+nsec)%nsec] + 1e-15)/Rmed[i] - Rmed[i]*OmegaFrame;
      /* It was the angular momentum */
    }
  }
}

__global__ void ComputeSpeQtyKernel (float *Label, float *Dens, float *ExtLabel, int nrad, int nsec)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  if (i<nrad && j<nsec){
    Label[i*nsec + j] = ExtLabel[i*nsec + j]/Dens[i*nsec + j];
    /* Compressive flow if line commentarized
    Label[i*nsec + j] = ExtLabel[i*nsec + j] */
  }
}

__global__ void FillForcesArraysKernel (float *Rmed, int nsec, int nrad, float xplanet, float yplanet, float smooth,
  float mplanet, int Indirect_Term, float InvPlanetDistance3, float *Potential, Pair IndirectTerm)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float InvDistance, angle, x, y, distance, distancesmooth, pot;

  if (i<nrad && j<nsec){
    InvDistance = 1.0/Rmed[i];
    angle = (float)j/(float)nsec*2.0*M_PI;
    x = Rmed[i]*cosf(angle);
    y = Rmed[i]*sinf(angle);
    distance = (x-xplanet)*(x-xplanet)+(y-yplanet)*(y-yplanet);
    distancesmooth = sqrtf(distance+smooth);
    pot = -G*mplanet/distancesmooth; /* Direct term from planet */
    if (Indirect_Term)
      pot += G*mplanet*InvPlanetDistance3*(x*xplanet+y*yplanet); /* Indirect term from planet */
    Potential[i*nsec + j] += pot;

    /* -- Gravitational potential from star on gas -- */
    pot = -G*1.0*InvDistance; /* Direct term from star */
    pot -= IndirectTerm.x*x + IndirectTerm.y*y; /* Indirect term from star */
    Potential[i*nsec + j] += pot;
  }
}

__global__ void CorrectVthetaKernel (float *Vtheta, float domega, float *Rmed, int nrad, int nsec)
{
    int j = threadIdx.x + blockDim.x*blockIdx.x;
    int i = threadIdx.y + blockDim.y*blockIdx.y;

    if (i<nrad && j<nsec)
      Vtheta[i*nsec + j] -= domega*Rmed[i];
}

__global__ void ConditionCFLKernel1D (float *Rsup, float *Rinf, float *Rmed, int nrad, int nsec,
  float *Vtheta, float *Vmoy)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j;
  float dxrad, dxtheta;

  if (i<nrad){
    dxrad = Rsup[i]-Rinf[i];
    dxtheta = Rmed[i]*2.0*M_PI/(float)nsec;
    Vmoy[i] = 0.0;

    for (j = 0; j < nsec; j++)
      Vmoy[i] += Vtheta[i*nsec + j];

    Vmoy[i] /= (float)nsec;
  }
}

__global__ void ConditionCFLKernel2D1 (float *Rsup, float *Rinf, float *Rmed, int nsec, int nrad,
  float *Vresidual, float *Vtheta, float *Vmoy, int FastTransport, float *SoundSpeed, float *Vrad,
  float CVNR, float *DT2D, float CFLSECURITY)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  int i = threadIdx.y + blockDim.y*blockIdx.y;

  float dxrad, dxtheta, invdt1, invdt2, invdt3, invdt4, dvr, dvt, dt;

  if (i > 0 && i<nrad && j<nsec){
    dxrad = Rsup[i]-Rinf[i];
    dxtheta = Rmed[i]*2.0*M_PI/(float)nsec;
    if (FastTransport) Vresidual[i*nsec + j] = Vtheta[i*nsec + j]-Vmoy[i]; /* Fargo algorithm */
    else Vresidual[i*nsec + j] = Vtheta[i*nsec + j];                       /* Standard algorithm */
    //Vresidual[i*nsec + nsec] = Vresidual[i*nsec];
    invdt1 = SoundSpeed[i*nsec + j]/(min2(dxrad,dxtheta));
    invdt2 = fabs(Vrad[i*nsec + j])/dxrad;
    invdt3 = fabs(Vresidual[i*nsec + j])/dxtheta;
    dvr = Vrad[(i+1)*nsec + j]-Vrad[i*nsec + j];
    dvt = Vtheta[i*nsec + (j+1)%nsec]-Vtheta[i*nsec + j];
    if (dvr >= 0.0) dvr = 1e-10;
    else dvr = -dvr;
    if (dvr >= 0.0) dvt = 1e-10;
    else dvt = -dvt;
    invdt4 = max2(dvr/dxrad, dvt/dxtheta);
    invdt4*= 4.0*CVNR*CVNR;
    dt = CFLSECURITY/sqrtf(invdt1*invdt1+invdt2*invdt2+invdt3*invdt3+invdt4*invdt4);
    DT2D[i*nsec + j] = dt; // array nrad*nsec size dt
  }
}



__global__ void ConditionCFLKernel2D2 (float *newDT, float *DT2D, float *DT1D, float *Vmoy, float *invRmed,
  int *CFL, int nsec, int nrad, float CFLSECURITY, float DeltaT)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int k;
  float dt;
  float newdt = 1e30;

  if (i>0 && i<nrad){
    newDT[i] = newdt;
    for (k = 0; k < nsec; k++)
      if (DT2D[i*nsec + k] < newDT[i]) newDT[i] = DT2D[i*nsec + k]; // for each dt in nrad

  }
  if (i<nrad-1){
    dt = 2.0*M_PI*CFLSECURITY/(float)nsec/fabs(Vmoy[i]*invRmed[i]-Vmoy[i+1]*invRmed[i+1]);
    DT1D[i] = dt; // array nrad size dt
    __syncthreads();

    if (i == 1){ // one thread
      newdt = newDT[1];
      for (k = 2; k < nrad; k++)
        if (newDT[k] < newdt) newdt = newDT[k]; // min dt

      for (k = 0; k < nrad-1; k++) {
        if (DT1D[k] < newdt)
          newdt = DT1D[k];
      }

      if (DeltaT < newdt)
        newdt = DeltaT;

      CFL[0] = (int)(ceil(DeltaT/newdt));
    }
  }
}

__device__ float max2(float a, float b)
{
  if (a > b) return a;
  return b;
}

__device__ float min2(float a, float b)
{
  if (a < b) return a;
  return b;
}

__device__ float AspectRatioDevice(float r, float ASPECTRATIO, float TRANSITIONWIDTH, float TRANSITIONRADIUS,
  float TRANSITIONRATIO, float PhysicalTime, float PhysicalTimeInitial, float LAMBDADOUBLING)
{
  float aspectratio, rmin, rmax, scale;
  aspectratio = ASPECTRATIO;
  rmin = TRANSITIONRADIUS-TRANSITIONWIDTH*ASPECTRATIO;
  rmax = TRANSITIONRADIUS+TRANSITIONWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) aspectratio *= TRANSITIONRATIO;
  if ((r >= rmin) && (r <= rmax)){
    aspectratio *= expf((rmax-r)/(rmax-rmin)*logf(TRANSITIONRATIO));
  }
  return aspectratio;
}
