#include "hip/hip_runtime.h"
#include "Main.cuh"
#include "Param.cuh"

/* extern float device arrays */
extern float *SigmaInf_d, *CellAbscissa_d, *CellOrdinate_d;
extern float *Pressure_d, *SoundSpeed_d;
extern float *viscosity_array_d, *QStar_d, *ExtLabel_d, *dq_d, *DRP_d, *vt_cent_d;
extern float *QRStar_d;
extern float *DivergenceVelocity_d, *DRR_d, *DPP_d, *TAURR_d, *TAURP_d, *TAUPP_d;
extern float *DensStar_d, *LostByDisk_d, *VMed_d;
extern float *Vmoy_d, *newDT_d, *DT1D_d, *DT2D_d;

extern float *RadMomP_d, *RadMomM_d, *ThetaMomP_d, *ThetaMomM_d, *TempShift_d;
extern float *Vrad_d, *Energy_d, *Vtheta_d, *VthetaRes_d, *VradInt_d, *Vresidual_d, *Vazimutal_d, *Vradial_d;
extern float *VradInt, *VthetaInt, *VradNew, *VthetaNew, *VthetaRes, *EnergyInt, *EnergyNew, *DensInt, *TemperInt_d;
extern float *Temperature, *TemperInt, *Temperature_d, *RadMomP, *RadMomM, *ThetaMomP, *ThetaMomM, *Work_d, *Work;
extern float *TempShift, *Pressure, *SoundSpeed;

/* extern float values */
extern float OMEGAFRAME;
extern float PhysicalTimeInitial, PhysicalTime;
extern float THICKNESSSMOOTHING;

/* extern float host arrays */
extern float *CellAbscissa, *CellOrdinate, *vt_cent;
extern float *Kr_aux, *Kt_aux;
extern float *QRStar, *ExtLabel, *dq, *DivergenceVelocity;
extern float *DRP, *DRR, *DPP, *TAURR, *TAURP, *TAUPP, *Radii;
extern float *Surf, *invSurf, *powRmed;
extern float *DensStar;
extern float *Potential;

float *example;

/* float host arrays */
float *SigmaMed, *SigmaInf, *EnergyMed;
float *fieldsrc, *vt_int, *GLOBAL_bufarray, *CoolingTimeMed, *QplusMed , *viscosity_array;
float *cs1, *Qplus, *QStar, *Qbase, *cs0, *csnrm1, *csnrm2, *mean_dens, *mean_dens2;
float *mean_energy, *mean_energy2, *array, *mdcp0;
float *SG_Accr, *SG_Acct, *GLOBAL_AxiSGAccr;

/* float device arrays */
float *Surf_d, *powRmed_d, *invSurf_d;
float *Qplus_d;
float *EnergyMed_d, *SigmaMed_d, *CoolingTimeMed_d, *QplusMed_d, *gridfield_d, *GLOBAL_bufarray_d;
float *Label_d, *QStar_d, *Qbase_d, *Qbase2_d, *cs0_d, *cs1_d, *csnrm1_d, *csnrm2_d, *mean_dens_d;
float *mean_dens_d2, *mean_energy_d, *mean_energy_d2;
float *SGP_Kr, *SGP_Kt, *Radii_d, *SGP_St, *SGP_Sr, *fxi_d, *fxo_d, *fyi_d, *fyo_d;
float *Kr_aux_d, *Kt_aux_d, *SG_Acct_d, *SG_Accr_d, *array_d, *mdcp0_d, *axifield_d, *GLOBAL_AxiSGAccr_d;


float *Dens_d, *VthetaInt_d, *VthetaNew_d, *VradNew_d, *EnergyInt_d, *EnergyNew_d, *DensInt_d, *Potential_d;

float mdcp, SGP_tstep, SGP_eps, SGP_rstep;

extern float HillRadius;
extern float *invdiffRmed, *Rinf_d, *Rinf, *invRinf, *Rmed, *invRmed, *Rsup, *invdiffRsup;

double OmegaFrame;
float *invdiffRmed_d, *invRinf_d, *invRmed_d, *Rmed_d, *Rsup_d, *invdiffRsup_d;

float *q0, *PlanetMasses, *q1;

extern int NRAD, NSEC, Cooling;
extern int *NoSplitAdvection_d;
extern int *Nshift_d;

int nrad2pot, nsec2pot, size_grid, nrad2potSG, nsec2potplus, *CFL_d, *CFL;
int blocksize2D = 16;
int blocksize1D = 256;

int         TimeToWrite, Restart = NO; // OpenInner = NO;
int             TimeStep = 0, NbRestart = 0, verbose = NO;
int             dimfxy = 11;
static int      InnerOutputCounter = 0, StillWriteOneOutput;
extern int  Corotating;
extern int  SelfGravity, SGZeroMode, Adiabatic;
float           ScalingFactor = 1.0;

dim3 dimGrid2, dimBlock2, dimGrid, dimBlock, dimGrid3, dimGrid4;

hipfftHandle planf, planb;

hipfftDoubleComplex *SGP_Kt_dc, *SGP_Kr_dc, *SGP_St_dc, *SGP_Sr_dc, *Gr_dc, *Gphi_dc, *Gr_d, *Gphi_d, *SGP_Kt_d,       \
*SGP_Kr_d, *SGP_Sr_d, *SGP_St_d;



__host__ int main (int argc, char *argv[])
{
  //int device;
  //printf("enter gpu device: ");
  //scanf("%d", &device);


  //hipSetDevice(1); Using gpu nvidia m4000 8 gb
  hipSetDevice(0); // Using gpu nvidia m4000 8gb

  float     *Dens;
  float     *Vrad;
  float     *Vtheta;
  float     *Energy;
  float     *Label;
  int       i;
  float     foostep = 0.;
  int   disable = NO, TimeInfo = NO, Profiling = NO;
  int   Stockholm = NO, SGUpdate = NO;
  char      ParameterFile[256];
  char      configplanet[100];
  PlanetarySystem *sys;
  Force *force;

  if (argc == 1) PrintUsage (argv[0]);
  strcpy (ParameterFile, "");
  for (i = 1; i < argc; i++){
    if (*(argv[i]) == '-'){
      if (strspn (argv[i], "-secndovtpfamzib0123456789") != strlen (argv[i]))
        PrintUsage (argv[0]);
      if (strchr (argv[i], 'n'))
        disable = YES;
      if (strchr (argv[i], 'e'))
        Stockholm = YES;
      if (strchr (argv[i], 'v'))
        verbose = YES;
      if (strchr (argv[i], 't'))
        TimeInfo = YES;
      if (strchr (argv[i], 'c'))
        SloppyCFL = YES;
      if (strchr (argv[i], 'p'))
        Profiling = YES;
      if (strchr (argv[i], 'd'))
        debug = YES;
      if (strchr (argv[i], 'b'))
        CentrifugalBalance = YES;
      if (strchr (argv[i], 'm'))
        Merge = YES;
      if (strchr (argv[i], 'a'))
        MonitorIntegral = YES;
      if (strchr (argv[i], 'z'))
        FakeSequential = YES;
      if (strchr (argv[i], 'i')){
        StoreSigma = YES;
	      if (Adiabatic)
          StoreEnergy = YES;
      }
      if (strchr (argv[i], '0'))
        OnlyInit = YES;
      if ((argv[i][1] >= '1') && (argv[i][1] <= '9')){
	       GotoNextOutput = YES;
	       StillWriteOneOutput = (int)(argv[i][1]-'0');
      }
      if (strchr (argv[i], 's')){
        Restart = YES;
	      i++;
	      NbRestart = atoi(argv[i]);
	      if ((NbRestart < 0)){
          printf ("Incorrect restart number\n");
	        PrintUsage (argv[0]);
        }
      }
      if (strchr (argv[i], 'o')){
        OverridesOutputdir = YES;
	      i++;
	      sprintf (NewOutputdir, "%s", argv[i]);
      }
      else {
        if (strchr (argv[i], 'f')){
	        i++;
	        ScalingFactor = atof(argv[i]);
	        printf ("Scaling factor = %g\n", ScalingFactor);
	        if ((ScalingFactor <= 0)){
	          printf ("Incorrect scaling factor\n");
	          PrintUsage (argv[0]);
          }
        }
      }
    }
    else strcpy (ParameterFile, argv[i]);
  }
  if ((StoreSigma || StoreEnergy) && !(Restart)){
    printf ("You cannot use tabulated surface density\n");
    printf ("or surface internal energy in a non-restart run.\n");
    printf ("Aborted\n");
    exit (0);
  }
  if (ParameterFile[0] == 0) PrintUsage (argv[0]);
  ReadVariables(ParameterFile);

  /* Si elige la opcion SelfGravity, se crean los planes 2D de la cufft */
  if (SelfGravity){
    if ((hipfftPlan2d(&planf, 2*NRAD, NSEC, HIPFFT_Z2Z)) != HIPFFT_SUCCESS){
      printf("cufft plan error\n");
      exit(-1);
    }

    if ((hipfftPlan2d(&planb, 2*NRAD, NSEC , HIPFFT_Z2Z)) != HIPFFT_SUCCESS){
      printf("cufft plan error\n");
      exit(-1);
    }
  }

  /* size grid */
  size_grid = (NRAD+1)*NSEC;

  /* aca falta ordenar --------------------------------------->*/
  if(!IsPow2(NRAD+1)) nrad2pot = NearestPowerOf2(NRAD+1);
  if(!IsPow2(NSEC)) nsec2pot = NearestPowerOf2(NSEC);
  if(!IsPow2(2*(NRAD+1))) nrad2potSG = NearestPowerOf2(2*(NRAD+1));

  /* dim gridsize and blocksize of */

  dim3 dimG( nsec2pot/blocksize1D, 1);
  dim3 dimB( blocksize1D, 1);
  dimGrid = dimG;
  dimBlock = dimB;

  dim3 dimG2( nsec2pot/blocksize2D, nrad2pot/blocksize2D);
  dim3 dimB2( blocksize2D, blocksize2D );
  dimGrid2 = dimG2;
  dimBlock2 = dimB2;


  dim3 dimG3 (nsec2pot/blocksize2D, nrad2potSG/blocksize2D);
  dimGrid3 = dimG3;

  dim3 dimG4 (nrad2pot/blocksize1D, 1);
  dimGrid4 = dimG4;

  if (verbose == YES)
    TellEverything();
  if (disable == YES)
    exit(0);
  printf("Allocating arrays...\n");
  /* local arrays */
  Dens   = (float *)malloc(size_grid*sizeof(float));
  Vrad   = (float *)malloc(size_grid*sizeof(float));
  Vtheta = (float *)malloc(size_grid*sizeof(float));
  Energy = (float *)malloc(size_grid*sizeof(float));
  Label  = (float *)malloc(size_grid*sizeof(float));

  /* global arrays */
  CreateArrays();

  printf("done.\n");
  FillPolar1DArrays ();
  force = AllocateForce (dimfxy);

  /* string to char configplanet */
  strncpy(configplanet, PLANETCONFIG.c_str(), sizeof(configplanet));
  configplanet[sizeof(configplanet)-1]=0;

  /* Here planets are initialized feeling star potential but they do
     not feel disk potential  */
  sys = InitPlanetarySystem (configplanet);

  /* Gas density initialization */

  InitGasDensity (Dens);


  /* If energy equation is taken into account, we initialize the gas
     thermal energy  */
  if (Adiabatic)
    InitGasEnergy (Energy);

  Cudamalloc(Label, Dens, Vrad, Vtheta);

  if (SelfGravity){
    SGP_eps = THICKNESSSMOOTHING * ASPECTRATIO;
    SGP_rstep = logf(Radii[NRAD]/Radii[0])/(float)NRAD;
    SGP_tstep = 2.0*PI/(float)NSEC;

    /* If SelfGravity = YES or Z, planets are initialized feeling disk
       potential. Only the surface density is required to calculate
       the radial self-gravity acceleration. The disk radial and
       azimutal velocities are not updated */

    /* Here we copy Vrad to Vradial -> device to device */
    gpuErrchk(hipMemcpy(Vradial_d, Vrad_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
    /* Here we copy Vtheta to Vazimutal -> device to device */
    gpuErrchk(hipMemcpy(Vazimutal_d, Vtheta_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));

    compute_selfgravity(Dens, foostep, SGUpdate, 1);

    /* Here we copy Vradial to Vrad -> device to device */
    gpuErrchk(hipMemcpy(Vrad_d, Vradial_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
    /* Here we copy Vazimutal to Vtheta -> device to device */
    gpuErrchk(hipMemcpy(Vtheta_d, Vazimutal_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
    Init_planetarysys_withSG (sys);
  }

  ListPlanets (sys);
  OmegaFrame = OMEGAFRAME;
  if (Corotating) OmegaFrame = GetPsysInfo (sys, FREQUENCY);

  /* Only gas velocities remain to be initialized */
  Initialization (Dens, Vrad, Vtheta, Energy, Label, sys);

  /* Initial gas_density is used to compute the circumplanetary mass with initial
     density field */

  //mdcp = CircumPlanetaryMass (Dens, sys);

  EmptyPlanetSystemFile (sys);
  PhysicalTimeInitial = PhysicalTime;

  //MultiplyPolarGridbyConstant(Dens);
  double start = omp_get_wtime();
  for (int i = 0; i <= NTOT; i++){
    InnerOutputCounter++;

    if (InnerOutputCounter == 1){
      InnerOutputCounter = 0;
      //WriteBigPlanetSystemFile (sys, TimeStep);
      //UpdateLog(force, sys, Dens, Energy, TimeStep, PhysicalTime, dimfxy);
    }

    if (NINTERM * (TimeStep = (i / NINTERM)) == i){
      /* Outputs are done here */
      //printf("%d\n", i);
      TimeToWrite = YES;
      DeviceToHostcudaMemcpy(Dens, Energy, Label, Temperature, Vrad, Vtheta); // Traigo los valores desde la GPU
      SendOutput (TimeStep, Dens, Vrad, Vtheta, Energy, Label);
      //WritePlanetSystemFile (sys, TimeStep);
    }
    else TimeToWrite = NO;
    /* Algorithm loop begins here *
    /***********************/
    /* Hydrodynamical Part */
    /***********************/

    AlgoGas(force, Dens, Vrad, Vtheta, Energy, Label, sys, i);
  }

  double finish = omp_get_wtime();

  printf("%g\n", finish-start);
  //DeviceToHostcudaMemcpy(Dens, Energy, Label, Temperature, Vrad, Vtheta); // Traigo los valores desde la GPU

  //gpuErrchk(hipMemcpy(Pressure, Pressure_d,           size_grid*sizeof(float), hipMemcpyDeviceToHost));

  /*binFile(Vrad, NRAD*NSEC, "gvrad");
  binFile(Vtheta, NRAD*NSEC, "gvtheta");
  binFile(Dens, NRAD*NSEC, "gdens");
  binFile(Energy, NRAD*NSEC, "genergy");
  binFile(Temperature, NRAD*NSEC, "gtemp");
  binFile(Pressure, NRAD*NSEC, "gpress");
*/
  FreePlanetary (sys);
  FreeForce (force);



 FreeCuda();
 FreeArrays(Dens, Vrad, Vtheta, Energy, Label);

 if (SelfGravity){
      hipfftDestroy(planf);
      hipfftDestroy(planb);
  }

	return EXIT_SUCCESS;
}



__host__ void FreeCuda ()
{
  /* hipFree FillPolar1DArrays */
  hipFree(Radii_d);
  hipFree(Rinf_d);
  hipFree(Rmed_d);
  hipFree(Rsup_d);
  hipFree(Surf_d);
  hipFree(invRinf_d);
  hipFree(invRmed_d);
  hipFree(invdiffRsup_d);
  hipFree(invdiffRmed_d);
  hipFree(powRmed_d);
  hipFree(vt_cent_d);
  hipFree(Kr_aux_d);
  hipFree(Kt_aux_d);

  /* hipFree InitComputeAccel */
  hipFree(CellAbscissa_d);
  hipFree(CellOrdinate_d);

  /* hipFree polar grid */
  hipFree(Dens_d);
  hipFree(Vrad_d);
  hipFree(Vtheta_d);
  hipFree(Energy_d);
  hipFree(Label_d);

  /* hipFree InitEuler */
  hipFree(SoundSpeed_d);
  hipFree(Pressure_d);
  hipFree(Temperature_d);
  hipFree(TemperInt_d);
  hipFree(DensStar_d);
  hipFree(VradInt_d);
  hipFree(VthetaInt_d);
  hipFree(Potential_d);
  hipFree(DensInt_d);
  hipFree(VradNew_d);
  hipFree(VthetaNew_d);
  hipFree(EnergyInt_d);

  /* hipFree ReduceCS and ReduceMean*/
  hipFree(cs0_d);
  hipFree(cs1_d);
  hipFree(csnrm1_d);
  hipFree(csnrm2_d);
  hipFree(mean_dens_d);
  hipFree(mean_energy_d);
  hipFree(mean_dens_d2);
  hipFree(mean_energy_d2);

  /* hipFree ComputeForce */
  hipFree(fxi_d);
  hipFree(fxo_d);
  hipFree(fyi_d);
  hipFree(fyo_d);

  hipFree(Qplus_d);
  hipFree(EnergyNew_d);
  hipFree(EnergyMed_d);
  hipFree(SigmaMed_d);
  hipFree(CoolingTimeMed_d);
  hipFree(QplusMed_d);

  hipFree(viscosity_array_d);
  hipFree(SigmaInf_d);
  hipFree(mdcp0_d);

  hipFree(QStar_d);
  hipFree(Qbase_d);
  hipFree(Qbase2_d);

  /* hipFree Make1Dprofile */
  hipFree(gridfield_d);
  hipFree(GLOBAL_bufarray_d);

  /* hipFree Crashed */
  hipFree(array_d);

  /* hipFree SelfGravity */
  hipFree(SGP_Kt_d);
  hipFree(SGP_Kr_d);
  hipFree(SGP_St_d);
  hipFree(SGP_Sr_d);
  hipFree(SGP_Kt_dc);
  hipFree(SGP_Kr_dc);
  hipFree(SGP_St_dc);
  hipFree(SGP_Sr_dc);
  hipFree(Gr_dc);
  hipFree(Gphi_dc);
  hipFree(Gr_d);
  hipFree(Gphi_d);
  hipFree(SG_Accr_d);
  hipFree(SG_Acct_d);
  hipFree(axifield_d);
  hipFree(GLOBAL_AxiSGAccr_d);

  /* hipFree InitTransport */
  hipFree(RadMomP_d);
  hipFree(RadMomM_d);
  hipFree(ThetaMomP_d);
  hipFree(ThetaMomM_d);
  hipFree(Work_d);
  hipFree(QRStar_d);
  hipFree(ExtLabel_d);
  hipFree(dq_d);

  /* hipFree InitViscosity */
  hipFree(DivergenceVelocity_d);
  hipFree(DRP_d);
  hipFree(DRR_d);
  hipFree(DPP_d);
  hipFree(TAURR_d);
  hipFree(TAURP_d);
  hipFree(TAUPP_d);

  hipFree(LostByDisk_d);
  hipFree(VthetaRes_d);
  hipFree(VMed_d);
  hipFree(Nshift_d);
  hipFree(NoSplitAdvection_d);
  hipFree(TempShift_d);
  hipFree(Vmoy_d);
  hipFree(DT1D_d);
  hipFree(DT2D_d);
  hipFree(newDT_d);
  hipFree(Vresidual_d);

  hipFree(CFL_d);
}


__host__ void FreeArrays (float *Dens, float *Vrad, float *Vtheta, float *Energy, float *Label)
{
  /* free FillPolar1DArrays */
  free(Radii);
  free(Rinf);
  free(Rmed);
  free(Rsup);
  free(Surf);
  free(invRinf);
  free(invSurf);
  free(invRmed);
  free(invdiffRsup);
  free(invdiffRmed);
  free(powRmed);
  free(vt_cent);
  free(Kr_aux);
  free(Kt_aux);

  /* free InitComputeAccel */
  free(CellAbscissa);
  free(CellOrdinate);

  /* free polar grid */
  free(Dens);
  free(Vrad);
  free(Vtheta);
  free(Energy);
  free(Label);

  /* free ComputeForce */


  /* free InitEuler */
  free(TemperInt);
  free(Temperature);
  free(Pressure);
  free(SoundSpeed);
  free(DensStar);
  free(VradInt);
  free(VthetaInt);
  free(DensInt);
  free(VradNew);
  free(VthetaNew);
  free(EnergyInt);
  free(EnergyNew);
  free(Potential);

  free(VthetaRes);
  free(TempShift);
  free(q0);
  free(q1);
  free(PlanetMasses);

  free(SG_Accr);
  free(SG_Acct);

  free(array);
  free(mdcp0);
  free(EnergyMed);
  free(SigmaMed);
  free(SigmaInf);
  free(vt_int);
  free(GLOBAL_bufarray);
  free(QplusMed);
  free(CoolingTimeMed);
  free(viscosity_array);
  free(Qplus);
  free(QStar);
  free(Qbase);
  free(GLOBAL_AxiSGAccr);

  /* free ReduceCS and ReduceMean*/
  free(cs0);
  free(cs1);
  free(csnrm1);
  free(csnrm2);
  free(mean_dens);
  free(mean_dens2);
  free(mean_energy);
  free(mean_energy2);

  /* free InitTransport */
  free(RadMomP);
  free(RadMomM);
  free(ThetaMomP);
  free(ThetaMomM);
  free(Work);
  free(QRStar);
  free(ExtLabel);
  free(dq);

  /* free InitViscosity */
  free(DivergenceVelocity);
  free(DRP);
  free(DRR);
  free(DPP);
  free(TAURR);
  free(TAURP);
  free(TAUPP);

  free(CFL);

}


__host__ void DeviceToHostcudaMemcpy (float *Dens, float *Energy, float *Label, float *Temperature, float *Vrad, float *Vtheta)
{
  gpuErrchk(hipMemcpy(Vrad, Vrad_d,               size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(Dens, Dens_d,               size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(Temperature, Temperature_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(Label, Label_d,             size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(Vtheta, Vtheta_d,           size_grid*sizeof(float), hipMemcpyDeviceToHost));
  if (Adiabatic)
    gpuErrchk(hipMemcpy(Energy, Energy_d,           size_grid*sizeof(float), hipMemcpyDeviceToHost));
  if (Cooling)
    gpuErrchk(hipMemcpy(Qplus, Qplus_d,           size_grid*sizeof(float), hipMemcpyDeviceToHost));
}


__host__ void CreateArrays () // ordenar
{
  CFL   = (int *)malloc(sizeof(int));

  EnergyMed         = (float *)malloc((NRAD+1)*sizeof(float));
  SigmaMed          = (float *)malloc((NRAD+1)*sizeof(float));
  SigmaInf          = (float *)malloc((NRAD+1)*sizeof(float));
  vt_int            = (float *)malloc((NRAD+1)*sizeof(float));
  GLOBAL_bufarray   = (float *)malloc((NRAD+1)*sizeof(float));
  GLOBAL_AxiSGAccr  = (float *)malloc((NRAD+1)*sizeof(float));
  QplusMed          = (float *)malloc((NRAD+1)*sizeof(float));
  CoolingTimeMed    = (float *)malloc((NRAD+1)*sizeof(float));
  viscosity_array = (float *)malloc((NRAD+1)*sizeof(float));

  cs0             = (float *)malloc(NSEC*sizeof(float));
  cs1             = (float *)malloc(NSEC*sizeof(float));
  csnrm1          = (float *)malloc(NSEC*sizeof(float));
  csnrm2          = (float *)malloc(NSEC*sizeof(float));
  mean_dens       = (float *)malloc(NSEC*sizeof(float));
  mean_dens2      = (float *)malloc(NSEC*sizeof(float));
  mean_energy     = (float *)malloc(NSEC*sizeof(float));
  mean_energy2    = (float *)malloc(NSEC*sizeof(float));


  if (SelfGravity){
    SG_Accr         = (float *)malloc(size_grid*sizeof(float));
    SG_Acct         = (float *)malloc(size_grid*sizeof(float));
  }

  example           = (float *)malloc(NRAD*NSEC*sizeof(float));
  QStar           = (float *)malloc(size_grid*sizeof(float));
  Qbase           = (float *)malloc(size_grid*sizeof(float));
  array           = (float *)malloc(size_grid*sizeof(float));
  mdcp0           = (float *)malloc(size_grid*sizeof(float));
  q0              = (float *)malloc(400*sizeof(float));
  q1              = (float *)malloc(400*sizeof(float));
  PlanetMasses    = (float *)malloc(100*sizeof(float));

}


__host__ void Cudamalloc (float *Label, float *Dens, float *Vrad, float *Vtheta) // arreglar
{
  /* hipMalloc SelfGravity*/

  if (SelfGravity){
    gpuErrchk(hipMalloc((void**)&SGP_Kt_d,  2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&SGP_Kr_d,  2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&SGP_St_d,  2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&SGP_Sr_d,  2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&SGP_Kt_dc, 2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&SGP_Kr_dc, 2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&SGP_St_dc, 2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&SGP_Sr_dc, 2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&Gr_dc,     2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&Gphi_dc,   2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&Gr_d,      2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&Gphi_d,    2*size_grid*sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc((void**)&Kr_aux_d,  2*size_grid*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&Kt_aux_d,  2*size_grid*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&SG_Accr_d, size_grid*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&SG_Acct_d, size_grid*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&axifield_d, NRAD*sizeof(float)));
  }

  gpuErrchk(hipMalloc((void**)&Vradial_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Vazimutal_d, size_grid*sizeof(float)));

  gpuErrchk(hipMemset(Vradial_d, 0,    size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Vazimutal_d, 0,  size_grid*sizeof(float)));

  /* hipMalloc ComputeForce */

  gpuErrchk(hipMalloc(&fxi_d, NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMalloc(&fxo_d, NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMalloc(&fyi_d, NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMalloc(&fyo_d, NRAD*NSEC*sizeof(float)));

  gpuErrchk(hipMemset(fxi_d, 0, NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMemset(fxo_d, 0, NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMemset(fyi_d, 0, NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMemset(fyo_d, 0, NRAD*NSEC*sizeof(float)));

  /* hipMalloc FillPolar1DArrays */
  gpuErrchk(hipMalloc((void**)&Radii_d,           (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Rmed_d,            (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Rinf_d,            (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&invRmed_d,         (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Rsup_d,            (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&invdiffRmed_d,     (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&invRinf_d,         (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&powRmed_d,         (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&invdiffRsup_d,     (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Surf_d,            (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&invSurf_d,         (NRAD+1)*sizeof(float)));

  gpuErrchk(hipMemset(Radii_d, 0,        (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(Rmed_d, 0,         (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(Rinf_d, 0,         (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(invRmed_d, 0,      (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(Rsup_d, 0,         (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(invdiffRmed_d, 0,  (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(invRinf_d, 0,      (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(powRmed_d, 0,      (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(invdiffRsup_d, 0,  (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(Surf_d, 0,         (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(invSurf_d, 0,      (NRAD+1)*sizeof(float)));


  /* hipMalloc med */
  gpuErrchk(hipMalloc((void**)&EnergyMed_d,       (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&SigmaMed_d,        (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&CoolingTimeMed_d,  (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&QplusMed_d,        (NRAD+1)*sizeof(float)));

  gpuErrchk(hipMemset(EnergyMed_d, 0,      (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(SigmaMed_d, 0,       (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(CoolingTimeMed_d, 0, (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemset(QplusMed_d, 0,       (NRAD+1)*sizeof(float)));


  /* hipMalloc ReduceCs */
  gpuErrchk(hipMalloc((void**)&cs0_d,          NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&cs1_d,          NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&csnrm1_d,       NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&csnrm2_d,       NSEC*sizeof(float)));

  gpuErrchk(hipMemset(cs0_d, 0,    NSEC*sizeof(float)));
  gpuErrchk(hipMemset(cs1_d, 0,    NSEC*sizeof(float)));
  gpuErrchk(hipMemset(csnrm1_d, 0, NSEC*sizeof(float)));
  gpuErrchk(hipMemset(csnrm2_d, 0, NSEC*sizeof(float)));

  /* hipMalloc ReduceMean */
  gpuErrchk(hipMalloc((void**)&mean_dens_d,    NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mean_dens_d2,   NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mean_energy_d,  NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mean_energy_d2, NSEC*sizeof(float)));

  gpuErrchk(hipMemset(mean_dens_d, 0,    NSEC*sizeof(float)));
  gpuErrchk(hipMemset(mean_dens_d2, 0,   NSEC*sizeof(float)));
  gpuErrchk(hipMemset(mean_energy_d, 0,  NSEC*sizeof(float)));
  gpuErrchk(hipMemset(mean_energy_d2, 0, NSEC*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&Qplus_d,            size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&EnergyNew_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&GLOBAL_bufarray_d,  (NRAD+1)*sizeof(float)));

  gpuErrchk(hipMemset(Qplus_d, 0,            size_grid*sizeof(float)));
  gpuErrchk(hipMemset(EnergyNew_d, 0,        size_grid*sizeof(float)));
  gpuErrchk(hipMemset(GLOBAL_bufarray_d, 0,  (NRAD+1)*sizeof(float)));


  /* hipMalloc polar grid */
  gpuErrchk(hipMalloc((void**)&Vrad_d,           size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Vtheta_d,         size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Dens_d,           size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Label_d,          size_grid*sizeof(float)));

  gpuErrchk(hipMemset(Vrad_d, 0,   size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Vtheta_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Dens_d, 0,   size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Label_d, 0,  size_grid*sizeof(float)));


  gpuErrchk(hipMalloc((void**)&Vresidual_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&newDT_d,          NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DT1D_d,           NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&gridfield_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Qbase_d,          size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Qbase2_d,         size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&QStar_d,          size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&array_d,          size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mdcp0_d,          NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Vmoy_d,           size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DT2D_d,           NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&CFL_d,            sizeof(int)));

  gpuErrchk(hipMemset(Vresidual_d, 0,    size_grid*sizeof(float)));
  gpuErrchk(hipMemset(newDT_d, 0,        NRAD*sizeof(float)));
  gpuErrchk(hipMemset(DT1D_d, 0,         NRAD*sizeof(float)));
  gpuErrchk(hipMemset(gridfield_d, 0,    size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Qbase_d, 0,        size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Qbase2_d, 0,       size_grid*sizeof(float)));
  gpuErrchk(hipMemset(QStar_d, 0,        size_grid*sizeof(float)));
  gpuErrchk(hipMemset(array_d, 0,        size_grid*sizeof(float)));
  gpuErrchk(hipMemset(mdcp0_d, 0,        NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMemset(Vmoy_d, 0,         size_grid*sizeof(float)));
  gpuErrchk(hipMemset(DT2D_d, 0,         NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMemset(CFL_d, 0,          sizeof(int)));

  /* hipMemcpy Host to Device */
  gpuErrchk(hipMemcpy(Radii_d, Radii,             (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Rmed_d, Rmed,               (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Rinf_d, Rinf,               (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(invRmed_d, invRmed,         (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Rsup_d, Rsup,               (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(invdiffRmed_d, invdiffRmed, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(invRinf_d, invRinf,         (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(powRmed_d, powRmed,         (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(invdiffRsup_d, invdiffRsup, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Surf_d, Surf,               (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(invSurf_d, invSurf,         (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Dens_d, Dens,               size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(SigmaMed_d, SigmaMed,             (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  if (Adiabatic) gpuErrchk(hipMemcpy(EnergyMed_d, EnergyMed,           (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

}

__host__ void binFile(float *array, int sizeArray, char *name)
{
  FILE *f;
  char filename[100];
  sprintf(filename, "../output/%s.raw",name);
  f = fopen(filename, "w");
  fwrite(array, sizeof(float), sizeArray, f);
  fclose(f);
}
