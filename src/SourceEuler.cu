#include "hip/hip_runtime.h"
#include "Main.cuh"
#include "Kernels.cuh"
#include "SourceEuler.cuh"
#include "Theo.cuh"
using namespace std;

extern int blocksize, nsec2pot, nrad2pot, NRAD, NSEC, YES, LogGrid, size_grid, SelfGravity, ViscosityAlpha, \
Adiabaticc, Cooling, size_grid2;

extern bool CentrifugalBalance;
extern string OUTPUTDIR;

extern float RMAX, RMIN, PI, MU, R,*invRmed, *invRinf, *invSurf, *invdiffRmed, *invdiffRsup, *Radii, \
*GLOBAL_bufarray, *invdiffSurf, *Rinf, *Rmed, *Rsup, *Surf, *cosns, *sinns, ADIABATICINDEX, \
FLARINGINDEX, *vt_int, OmegaFrame1, *SigmaInf, G, ASPECTRATIO, SIGMA0, SIGMASLOPE, IMPOSEDDISKDRIFT, \
*CoolingTimeMed, *QplusMed , *viscosity_array, *dens_d, *Rmed_d;

float *press, *CellAbscissa, *CellOrdinate, *AspectRatioRmed, *SoundSpeed, *temperature, *vtheta_d, \
*CellAbscissa_d, *CellOrdinate_d, *sinns_d, *cosns_d, *vt_cent, *Rinf_d, *SigmaInf_d, *vrad_d, *SoundSpeed_d, \
*energy_d, *AspectRatioRmed_d, *press_d, *temperature_d, *viscosity_array_d, *Kr_aux, *Kt_aux;

extern dim3 dimGrid2, dimBlock2;

double *Radii2;

__host__ void FillPolar1DArray()
{
  FILE *input, *output;
  int i,ii;
  float drrsep;
  float temporary;
  string InputName, OutputName;
  drrsep = (RMAX-RMIN)/NRAD;
  InputName = OUTPUTDIR +"radii.dat";
  OutputName = OUTPUTDIR +"used_rad.dat";

  Radii = (float *) malloc(sizeof(float)*(NRAD+1));
  Radii2 = (double *) malloc(sizeof(double)*(NRAD+1));
  vt_cent = (float *) malloc(sizeof(float)*NRAD);
  Rinf = (float *) malloc(sizeof(float)*(NRAD));
  Rmed = (float *) malloc(sizeof(float)*(NRAD));
  Rsup = (float *) malloc(sizeof(float)*(NRAD));
  Surf = (float *) malloc(sizeof(float)*(NRAD));
  invRinf = (float *) malloc(sizeof(float)*(NRAD));
  invSurf = (float *) malloc(sizeof(float)*(NRAD));
  invRmed = (float *) malloc(sizeof(float)*(NRAD));
  invdiffSurf = (float *) malloc(sizeof(float)*(NRAD));
  invdiffRsup = (float *) malloc(sizeof(float)*(NRAD));
  invdiffRmed = (float *) malloc(sizeof(float)*(NRAD));

  char inputcharname[100];
  strncpy(inputcharname, InputName.c_str(), sizeof(inputcharname));
  inputcharname[sizeof(inputcharname)-1]=0;

  double u, theta, base,algo, den_SGP_K, algo2;

  Kr_aux = (float *)malloc(sizeof(float)*2*size_grid);
  Kt_aux = (float *)malloc(sizeof(float)*2*size_grid);

  input = fopen (inputcharname, "r");
  if (input == NULL)
  {
    printf("Warning : no `radii.dat' file found. Using default.\n");
    if (LogGrid == YES)
    {
      for (i = 0; i <= NRAD; i++)
      {
        Radii2[i] = RMIN*exp((double)i/(double)NRAD*log(RMAX / RMIN));
        Radii[i] = (float) Radii2[i];
      }


      for (i = 0; i < 2*NRAD; i++)
      {
        if(i<NRAD) u = log(Radii2[i]/Radii2[0]);
        else u = -log(Radii2[2*NRAD-i]/Radii2[0]);



        for (int j = 0; j < NSEC; j++) {
          theta = 2.0*M_PI*(double)j  / (double)NSEC;
          base = 0.03*0.03 * exp(u) + 2.0* (cosh(u) - cos(theta));
          den_SGP_K = pow(base , -1.5);

          algo = 1.0 + 0.03*0.03 - cos(theta) * exp(-u);
          algo *= den_SGP_K;

          algo2 = sin(theta) * den_SGP_K;
          if ( i==10 && j == 10) printf("%g\n",algo2 );
          Kr_aux[i*NSEC+j] = (float) algo;
          Kt_aux[i*NSEC+j] = (float) algo2;
        }

      }


      // FILE *f;
      // f = fopen("Kr.raw", "w");
      //
      // for (int i = 0; i < NRAD; i++) {
      //   for (int j = 0; j < NSEC; j++) {
      //     fwrite((void *) &Kr_aux[i*NSEC+j], 1, sizeof(float), f);
      //   }
      // }
      // fclose(f);
    }
    else {
      for (i = 0; i <= NRAD; i++) Radii[i] = RMIN+drrsep*i;
    }
  }
  else
  {
    printf("Reading 'radii.dat' file.\n");
    for (i = 0; i <= NRAD; i++)
    {
      fscanf (input, "%f", &temporary);
      Radii[i] = (float)temporary;
    }
  }

  for (i = 0; i < NRAD; i++)
  {
    //Rmed[i] = 2.0/3.0*(Radii[i+1]*Radii[i+1]*Radii[i+1]-Radii[i]*Radii[i]*Radii[i]);
    //Rmed[i] = Rmed[i] / (Radii[i+1]*Radii[i+1]-Radii[i]*Radii[i]);
  }

  for (i = 0; i < NRAD; i++)
  {
    Rinf[i] = Radii[i];
    Rsup[i] = Radii[i+1];
    Rmed[i] = 2.0/3.0*(Rsup[i]*Rsup[i]*Rsup[i]-Rinf[i]*Rinf[i]*Rinf[i]);
    Rmed[i] = Rmed[i] / (Rsup[i]*Rsup[i]-Rinf[i]*Rinf[i]);
    Surf[i] = M_PI*(Rsup[i]*Rsup[i]-Rinf[i]*Rinf[i])/(float)NSEC;
    invRmed[i] = 1.0/Rmed[i];
    invSurf[i] = 1.0/Surf[i];
    invdiffRsup[i] = 1.0/(Rsup[i]-Rinf[i]);
    invRinf[i] = 1.0/Rinf[i];
  }

  Rinf[NRAD]=Radii[NRAD];
  for (i = 1; i < NRAD; i++) invdiffRmed[i] = 1.0/(Rmed[i]-Rmed[i-1]);

  char outputcharname[100];
  strncpy(outputcharname, OutputName.c_str(), sizeof(outputcharname));
  outputcharname[sizeof(outputcharname)-1]=0;
  output = fopen (outputcharname, "w");
  if (output == NULL)
  {
    printf ("Can't write %s.\nProgram stopped.\n", outputcharname);
    exit (1);
  }
  for (i = 0; i <= NRAD; i++)fprintf (output, "%f\n", Radii[i]);

  fclose (output);
  if (input != NULL) fclose (input);
}

__host__ void InitEuler (float *dens, float *energy, float *vrad, float *vtheta)
{

  CellAbscissa = (float *)malloc(sizeof(float)*size_grid);
  CellOrdinate = (float *)malloc(sizeof(float)*size_grid);
  cosns = (float *)malloc(sizeof(float)*NSEC);
  sinns = (float *)malloc(sizeof(float)*NSEC);
  AspectRatioRmed = (float *)malloc(sizeof(float)*NRAD);
  press = (float *)malloc(sizeof(float)*size_grid);
  SoundSpeed = (float *)malloc(sizeof(float)*size_grid);
  temperature = (float *)malloc(sizeof(float)*size_grid);

  for (int i = 0; i < NSEC; i++)
  {
      cosns[i] = cos((2.0*HIP_PI_F*i)/NSEC);
      sinns[i] = sin((2.0*HIP_PI_F*i)/NSEC);
  }


  for (int i = 0; i < NRAD; i++) AspectRatioRmed[i] = AspectRatio(Rmed[i]);

  Computecudamalloc(dens, energy, vrad, vtheta);

  InitComputeAccel<<<dimGrid2, dimBlock2>>>(CellAbscissa_d, CellOrdinate_d, Rmed_d, cosns_d,
    sinns_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());

  ComputeSoundSpeed<<<dimGrid2, dimBlock2>>>(SoundSpeed_d, dens_d, Rmed_d, energy_d, NSEC, NRAD,
    Adiabaticc, ADIABATICINDEX, FLARINGINDEX, AspectRatioRmed_d);
  gpuErrchk(hipDeviceSynchronize());

  ComputePressureField<<<dimGrid2, dimBlock2>>>(SoundSpeed_d, dens_d, press_d, Adiabaticc, NRAD,
    NSEC, ADIABATICINDEX, energy_d);
  gpuErrchk(hipDeviceSynchronize());

  ComputeTemperatureField<<<dimGrid2, dimBlock2>>>(dens_d, temperature_d, press_d, energy_d, MU, R,
    ADIABATICINDEX, Adiabaticc, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());

  InitGasVelocitieshost(vrad, vtheta);

}


__host__ void InitGasVelocitieshost(float *vrad, float *vtheta)
{
  float t1, r, ri, vt_cent_d;
  int nrad2potlocal;

  CoolingTimeMed = (float *)malloc(sizeof(float)*size_grid);
  QplusMed = (float *)malloc(sizeof(float)*size_grid);
  viscosity_array = (float *)malloc(sizeof(float)*NRAD+1);

  /* Pressure is already initialized: cf initeuler in SourceEuler.c ...
    Initialization of azimutal velocity with exact centrifugal balance */

  if (CentrifugalBalance)
  {
    /* vt_int \equiv Romega = grad(P)/sigma + \partial(phi)/\partial(r) - acc_sg_radial
    ./bin/fargoGPU  -b in/template.par */

    gpuErrchk(hipMemcpy(press, press_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
    Make1Dprofilehost(press);

    /* global axisymmetric pressure field */
    for (int i = 1; i < NRAD; i++)
    {
      vt_int[i] = ( GLOBAL_bufarray[i] - GLOBAL_bufarray[i-1]) / \
      (.5*(Sigma(Rmed[i]) + Sigma(Rmed[i-1])))/(Rmed[i]-Rmed[i-1]);
    }
    /* Case of a disk with self-gravity */
    // if ( SelfGravity )

    for (int i = 1; i < NRAD; i++)
      vt_int[i] = sqrtf(vt_int[i]*Radii[i]) - Radii[i]*OmegaFrame1;

    t1 = vt_cent[0] = vt_int[1]+.75*(vt_int[1]-vt_int[2]);
    //r1 = ConstructSequence (vt_cent, vt_int, NRAD);
  }

  if (!CentrifugalBalance && SelfGravity) // init_azimutalvelocity_withSG (vtheta);

  if (ViscosityAlpha)
  {
    gpuErrchk(hipMemcpy(SoundSpeed, SoundSpeed_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
    Make1Dprofilehost(SoundSpeed);
  }


  if (Cooling)
  {
    FillCoolingTime();
    FillQplus();
  }

  for (int i = 0; i <= NRAD; i++)
  {
    if (i == NRAD) viscosity_array[i] = FViscosity(Rmed[NRAD-1]);
    else viscosity_array[i] = FViscosity(Rmed[i]);
  }

  if(!IsPow2(NRAD+1)) nrad2potlocal = NearestPowerOf2(NRAD+1);

  gpuErrchk(hipMalloc((void**)&viscosity_array_d, (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  InitGasVelocities<<<dimGrid2, dimBlock2>>>(viscosity_array_d, NSEC, NRAD, SelfGravity, Rmed_d,
  G, ASPECTRATIO, FLARINGINDEX, SIGMASLOPE, CentrifugalBalance, vrad_d, vtheta_d, ViscosityAlpha,
  IMPOSEDDISKDRIFT, SIGMA0, SigmaInf_d, OmegaFrame1, Rinf_d); // falta vt_cent_d

  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(vrad, vrad_d, size_grid2*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(vtheta, vtheta_d, size_grid2*sizeof(float), hipMemcpyDeviceToHost));

  for (int i = 0; i < NSEC; i++) vrad[i] = vrad[i+NSEC*NRAD] = 0.0;

  gpuErrchk(hipMemcpy(vrad_d, vrad, size_grid2*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vtheta_d, vtheta, size_grid2*sizeof(float), hipMemcpyHostToDevice));

}

__host__ void Computecudamalloc(float *dens, float *energy, float *vrad, float *vtheta)
{
  gpuErrchk(hipMalloc((void**)&CellAbscissa_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&CellOrdinate_d, size_grid*sizeof(float) ));
  gpuErrchk(hipMalloc((void**)&cosns_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&sinns_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&SoundSpeed_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&energy_d, size_grid*sizeof(float) ));
  gpuErrchk(hipMalloc((void**)&AspectRatioRmed_d, NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&press_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&temperature_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vrad_d, size_grid2*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vtheta_d, size_grid2*sizeof(float)));
  //gpuErrchk(hipMalloc((void**)&vt_cent_d, NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&SigmaInf_d, NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Rinf_d, NRAD*sizeof(float)));

  gpuErrchk(hipMemcpy(CellAbscissa_d, CellAbscissa, size_grid*sizeof(float), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(CellOrdinate_d, CellOrdinate, size_grid*sizeof(float), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(cosns_d, cosns, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(sinns_d, sinns, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(SoundSpeed_d, SoundSpeed, size_grid*sizeof(float), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(energy_d, energy, size_grid*sizeof(float), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(AspectRatioRmed_d, AspectRatioRmed, NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(press_d, press, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(temperature_d, temperature, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vrad_d, vrad, size_grid2*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vtheta_d, vtheta, size_grid2*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(vt_cent_d, vt_cent, NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(SigmaInf_d, SigmaInf, NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Rinf_d, Rinf, NRAD*sizeof(float), hipMemcpyHostToDevice));

}
