#include "hip/hip_runtime.h"
#include "Main.cuh"

extern int NRAD, NSEC, LogGrid, size_grid, SelfGravity, ViscosityAlpha;
extern int Adiabatic, Cooling, Corotating, IsDisk, Evanescent, FastTransport;
extern int CentrifugalBalance, ZMPlus = NO, SloppyCFL, *CFL_d, *CFL;

extern string OUTPUTDIR;

extern float RMAX, RMIN, ADIABATICINDEX, FLARINGINDEX, ASPECTRATIO;
extern float SIGMA0, SIGMASLOPE, IMPOSEDDISKDRIFT, PhysicalTime, DT, MASSTAPER;
extern float SGP_eps, TRANSITIONWIDTH, TRANSITIONRATIO, TRANSITIONRADIUS;
extern float LAMBDADOUBLING, PhysicalTimeInitial, mdcp;

extern float *GLOBAL_bufarray, *vt_int, *SigmaInf, *CoolingTimeMed, *QplusMed , *viscosity_array;
extern float *SG_Accr, *array, *Qplus, *SigmaMed,  *EnergyMed, *CellOrdinate, *CellAbscissa;

extern float *Dens_d, *SG_Accr_d, *SG_Acct_d, *GLOBAL_bufarray_d, *array_d;
extern float *Qplus_d, *EnergyInt_d, *EnergyNew_d, *VradNew_d, *Potential_d;
extern float *VthetaInt_d, *powRmed_d, *VthetaNew_d, *SigmaMed_d, *QplusMed_d;
extern float *CoolingTimeMed_d, *EnergyMed_d, *DensInt_d, *DivergenceVelocity_d, *TAURR_d, *TAURP_d;
extern float *TAUPP_d, *Vmoy_d, *CellOrdinate_d, *CellAbscissa_d, *mdcp0_d;

extern double *Surf_d;

extern double *invdiffRmed_d, *invRinf_d, *Rmed_d, *invRmed_d, *invdiffRsup_d, *Rsup_d;

double *invdiffRmed, *invRinf, *Rinf, *Rinf_d, *invRmed, *Rmed, *invdiffRsup, *Rsup;

float *Pressure, *SoundSpeed, *Temperature, *DensStar, *VradInt;
double *invSurf, *Radii, *Surf, *powRmed,  *vt_cent;
float *VthetaInt, *DensInt, *VradNew, *VthetaNew, *EnergyInt, *Potential, *EnergyNew, *TemperInt;

float *Vtheta_d, *SigmaInf_d, *Vrad_d, *SoundSpeed_d, *Energy_d, *Pressure_d;
float *Temperature_d, *viscosity_array_d, *vt_cent_d, *DensStar_d, *TemperInt_d, *VradInt_d, *DT1D_d;
float *DT2D_d, *newDT_d, *Vresidual_d, *Vradial_d, *Vazimutal_d;


float *DT2D;
float exces_mdcp = 0.0, mdcp1, MassTaper;

int CrashedDens, CrashedEnergy;

extern dim3 dimGrid2, dimBlock2, dimGrid4, dimBlock;

int init = 0;

extern double OmegaFrame, HillRadius;

Pair DiskOnPrimaryAcceleration;



__host__ void FillPolar1DArrays ()
{
  FILE *input, *output;
  int i,j;
  float drrsep, temporary;
  float *Radii2, *Rmed2;
  string InputName, OutputName;
  drrsep = (RMAX-RMIN)/(float)NRAD;
  InputName = OUTPUTDIR + "radii.dat";
  OutputName = OUTPUTDIR + "used_rad.dat";

  /* Creo los arreglos de FillPolar1DArrays */
  Radii       = (double *)malloc((NRAD+1)*sizeof(double));
  Rinf        = (double *)malloc((NRAD+1)*sizeof(double));
  Rmed        = (double *)malloc((NRAD+1)*sizeof(double));
  Rsup        = (double *)malloc((NRAD+1)*sizeof(double));
  Surf        = (double *)malloc((NRAD+1)*sizeof(double));
  invRinf     = (double *)malloc((NRAD+1)*sizeof(double));
  invSurf     = (double *)malloc((NRAD+1)*sizeof(double));
  invRmed     = (double *)malloc((NRAD+1)*sizeof(double));
  invdiffRsup = (double *)malloc((NRAD+1)*sizeof(double));
  invdiffRmed = (double *)malloc((NRAD+1)*sizeof(double));
  vt_cent     = (double *)malloc((NRAD+1)*sizeof(double));
  powRmed     = (double *)malloc((NRAD+1)*sizeof(double));
  DT2D     = (float *)malloc(NRAD*NSEC*sizeof(float));

  char InputCharName[100];
  char OutputCharName[100];
  /* string to char InputName */
  strncpy(InputCharName, InputName.c_str(), sizeof(InputCharName));
  InputCharName[sizeof(InputCharName)-1]=0;

  input = fopen (InputCharName, "r");
  if (input == NULL){
    printf("Warning : no `radii.dat' file found. Using default.\n");
    if (LogGrid == YES){
      for (i = 0; i <= NRAD; i++){
        /* Usamos floats para calcular los valores de los arrays, luego
           los pasamos a float */
        Radii[i] = RMIN*exp((double)i/(double)NRAD*log(RMAX/RMIN));
      }
    }
    else {
      for (i = 0; i <= NRAD; i++)
        Radii[i] = RMIN+drrsep*(float)i;
    }
  }
  else {
    printf("Reading 'radii.dat' file.\n");
    for (i = 0; i <= NRAD; i++){
      fscanf (input, "%f", &temporary);
      Radii[i] = (float)temporary;
    }
  }

  for (i = 0; i < NRAD; i++){
    Rinf[i] = Radii[i];
    Rsup[i] = Radii[i+1];
    Rmed[i] = 2.0/3.0*(Rsup[i]*Rsup[i]*Rsup[i]-Rinf[i]*Rinf[i]*Rinf[i]);
    Rmed[i] = Rmed[i] / (Rsup[i]*Rsup[i]-Rinf[i]*Rinf[i]);
    Surf[i] = PI*(Rsup[i]*Rsup[i]-Rinf[i]*Rinf[i])/(double)NSEC;
    invRmed[i] = 1.0/Rmed[i];
    invSurf[i] = 1.0/Surf[i];
    invdiffRsup[i] = 1.0/(Rsup[i]-Rinf[i]);
    invRinf[i] = 1.0/Rinf[i];
  }

  Rinf[NRAD] = Radii[NRAD];

  for (i = 0; i < NRAD; i++) {
    if (i > 0 )invdiffRmed[i] = 1.0/(Rmed[i]-Rmed[i-1]);

    powRmed[i] = pow(Rmed[i],-2.5+SIGMASLOPE);
  }

  /* string to char OutputName */
  strncpy(OutputCharName, OutputName.c_str(), sizeof(OutputCharName));
  OutputCharName[sizeof(OutputCharName)-1]=0;

  output = fopen (OutputCharName, "w");
  if (output == NULL){
    printf ("Can't write %s.\nProgram stopped.\n", OutputCharName);
    exit (1);
  }
  for (i = 0; i <= NRAD; i++){
    fprintf (output, "%.30f\n", Radii[i]);
  }
  fclose (output);
  if (input != NULL) fclose (input);
}



__host__ void InitEuler (float *Vrad, float *Vtheta, float *Dens, float *Energy)
{
  InitTransport ();
  InitViscosity ();
  DensStar        = (float *)malloc(size_grid*sizeof(float));
  DensInt         = (float *)malloc(size_grid*sizeof(float));
  VradNew         = (float *)malloc(size_grid*sizeof(float));
  VradInt         = (float *)malloc(size_grid*sizeof(float));
  VthetaNew       = (float *)malloc(size_grid*sizeof(float));
  VthetaInt       = (float *)malloc(size_grid*sizeof(float));
  EnergyNew       = (float *)malloc(size_grid*sizeof(float));
  EnergyInt       = (float *)malloc(size_grid*sizeof(float));
  TemperInt       = (float *)malloc(size_grid*sizeof(float));
  Potential       = (float *)malloc(size_grid*sizeof(float));
  Pressure        = (float *)malloc(size_grid*sizeof(float));
  SoundSpeed      = (float *)malloc(size_grid*sizeof(float));
  Temperature     = (float *)malloc(size_grid*sizeof(float));
  Qplus           = (float *)malloc(size_grid*sizeof(float));

  Computecudamalloc (Energy);

  InitComputeAccel ();
  /* Rho and Energy are already initialized: cf main.cu*/
  ComputeSoundSpeed ();
  ComputePressureField ();
  ComputeTemperatureField ();
  InitGasVelocities (Vrad, Vtheta);

}



__host__ void AlgoGas (Force *force, float *Dens, float *Vrad, float *Vtheta, float *Energy, float *Label,
  PlanetarySystem *sys, int initialization)
{
  double dt, dtemp =0.0;
  double OmegaNew, domega;
  int gastimestepcfl = 1;
  CrashedDens = 0;
  CrashedEnergy = 0;

  if (Adiabatic){
    ComputeSoundSpeed();
    /* it is necesary to update computation of soundspeed if one uses
      alphaviscosity in Fviscosity. It is not necesary in locally
      isothermal runs since cs is constant. It is computed here for
      the needs of ConditionCFL. */
  }
  if (IsDisk == YES){
    if (SloppyCFL == YES){
      gastimestepcfl = ConditionCFL(Vrad, Vtheta, DT-dtemp);     // case ./bin/fargoGPU -c
    }
  }

  dt = DT / gastimestepcfl;
  int numero = 0;
  while (dtemp < 0.999999999*DT){
    MassTaper = PhysicalTime/(MASSTAPER*2.0*M_PI);
    MassTaper = (MassTaper > 1.0 ? 1.0 : pow(sin(MassTaper*M_PI/2.0), 2.0));
    if(IsDisk == YES){
      if (SloppyCFL == NO){
        gastimestepcfl = 1;
        gastimestepcfl = ConditionCFL(Vrad, Vtheta ,DT-dtemp);
        dt = (DT-dtemp)/(float)gastimestepcfl;
      }
      //printf("dt %f\n", dt);
      AccreteOntoPlanets(Dens, Vrad, Vtheta, dt, sys); // si existe acrecion entra
    }
    dtemp += dt;
    DiskOnPrimaryAcceleration.x = 0.0;
    DiskOnPrimaryAcceleration.y = 0.0;
    if (Corotating == YES) GetPsysInfo (sys, MARK);


    if (IsDisk == YES){
      /* Indirect term star's potential computed here */
      DiskOnPrimaryAcceleration = ComputeAccel (force, Dens, 0.0, 0.0, 0.0, 0.0);

      /* Gravitational potential from star and planet(s) is computed and stored here */
      FillForcesArrays (sys, Dens, Energy);

      /* Planet's velocities are update here from gravitational interaction with disk */
      AdvanceSystemFromDisk (force, Dens, Energy, sys, dt);
    }

    /* Planet's positions and velocities are update from gravitational interaction with star
       and other planets */
    AdvanceSystemRK5 (sys,dt);

    /* Below we correct vtheta, planet's position and velocities if we work in a frame non-centered on the star */
    if (Corotating == YES){
      OmegaNew = GetPsysInfo(sys, GET) / dt;
      domega = OmegaNew - OmegaFrame;
      if (IsDisk == YES) CorrectVtheta (Vtheta, domega);
      OmegaFrame = OmegaNew;
    }
    RotatePsys (sys, OmegaFrame*dt);

    /* Now we update gas */
    if (IsDisk == YES){
      //ApplyBoundaryCondition (Dens, Energy, Vrad, Vtheta, dt);
      gpuErrchk(hipMemcpy(Dens, Dens_d,     size_grid*sizeof(float), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(Energy, Energy_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
      CrashedDens = DetectCrash (Dens);
      CrashedEnergy = DetectCrash (Energy);
      if (CrashedDens == YES || CrashedEnergy == YES){
        fprintf(stdout, "\nCrash! at time %d\n", PhysicalTime);
        printf("c");
      }
      else
        printf(".");
      //if (ZMPlus) compute_anisotropic_pressurecoeff(sys);


      ComputePressureField ();
      Substep1 (Dens, Vrad, Vtheta, dt, init);
      Substep2 (dt);
      host (dt);


      ActualiseGasVrad (Vrad, VradNew);
      ActualiseGasVtheta (Vtheta, VthetaNew);
      //ApplyBoundaryCondition (Dens, Energy, Vrad, Vtheta, dt);

      if (Adiabatic){
        gpuErrchk(hipMemcpy(Vradial_d, Vrad_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(Vazimutal_d, Vtheta_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
        ComputeViscousTerms (Vrad, Vtheta, Dens);
        gpuErrchk(hipMemcpy(Vrad_d, Vradial_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(Vtheta_d, Vazimutal_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));

        Substep3 (Dens, dt);

        ActualiseGasEnergy (Energy, EnergyNew);
      }


      Transport (Dens, Vrad, Vtheta, Energy, Label, dt);
      //ApplyBoundaryCondition(Dens, Energy, Vrad, Vtheta, dt);
      ComputeTemperatureField ();

      mdcp1 = CircumPlanetaryMass (Dens, sys);
      exces_mdcp = mdcp1 - mdcp;
    }
    init = init + 1;

    PhysicalTime += dt;
  }
  printf("\n" );
}



__host__ void Substep1 (float *Dens, float *Vrad, float *Vtheta, float dt, int initialization)
{
  int selfgravityupdate;
  if(initialization == 0) Substep1cudamalloc(Vrad, Vtheta);

  Substep1Kernel<<<dimGrid2, dimBlock2>>>(Pressure_d, Dens_d, VradInt_d, invdiffRmed_d, Potential_d, Rinf_d,
    invRinf_d, Vrad_d, VthetaInt_d, Vtheta_d, Rmed_d,  dt, NRAD, NSEC, OmegaFrame, ZMPlus,
    IMPOSEDDISKDRIFT, SIGMASLOPE);
  gpuErrchk(hipDeviceSynchronize());


  if (SelfGravity){
    selfgravityupdate = YES;

    /* We copy VradInt to Vradial -> device to device */
    gpuErrchk(hipMemcpy(Vradial_d, VradInt_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(Vazimutal_d, VthetaInt_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));

    compute_selfgravity(Dens, dt, selfgravityupdate, 0);
    /* Vradialto VradInt -> device to device */
    gpuErrchk(hipMemcpy(VradInt_d, Vradial_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(VthetaInt_d, Vazimutal_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  }

  gpuErrchk(hipMemcpy(Vradial_d, VradInt_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipMemcpy(Vazimutal_d, VthetaInt_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  ComputeViscousTerms (VradInt, VthetaInt, Dens);


  UpdateVelocitiesWithViscosity(VradInt, VthetaInt, Dens, dt);



  if (!Evanescent)
    ApplySubKeplerianBoundary(VthetaInt);

}



__host__ void Substep2 (float dt)
{
  Substep2Kernel<<<dimGrid2, dimBlock2>>>(Dens_d, VradInt_d, VthetaInt_d, TemperInt_d, NRAD, NSEC, invdiffRmed_d,
  invdiffRsup_d, DensInt_d, Adiabatic, Rmed_d, dt, VradNew_d, VthetaNew_d, Energy_d, EnergyInt_d);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void host (float dt)
{
  kernel<<<dimGrid2, dimBlock2>>>(Dens_d, VradInt_d, VthetaInt_d, TemperInt_d, NRAD, NSEC, invdiffRmed_d,
  invdiffRsup_d, DensInt_d, Adiabatic, Rmed_d, dt, VradNew_d, VthetaNew_d, Energy_d, EnergyInt_d);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void Substep3 (float *Dens, float dt)
{
  for (int i = 0; i < NRAD; i++) viscosity_array[i] = FViscosity(Rmed[i]);
  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  Substep3Kernel<<<dimGrid2, dimBlock2>>>(Dens_d, Qplus_d, viscosity_array_d, TAURR_d, TAURP_d , TAUPP_d, DivergenceVelocity_d,
     NRAD, NSEC, Rmed_d, Cooling, EnergyNew_d, dt, EnergyMed_d, SigmaMed_d, CoolingTimeMed_d, Energy_d,
     ADIABATICINDEX, QplusMed_d);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void Computecudamalloc (float *Energy)
{

  CoolingTimeMed  = (float *)malloc((NRAD+1)*sizeof(float));
  QplusMed        = (float *)malloc((NRAD+1)*sizeof(float));
  viscosity_array = (float *)malloc((NRAD+1)*sizeof(float));

  gpuErrchk(hipMalloc((void**)&Temperature_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Pressure_d,    size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&SoundSpeed_d,  size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DensStar_d,    size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VradInt_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DensInt_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VradNew_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaNew_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Potential_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaInt_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TemperInt_d,   size_grid*sizeof(float)));

  gpuErrchk(hipMemset(TemperInt_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Temperature_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Pressure_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(SoundSpeed_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(DensStar_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(VradInt_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(DensInt_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(VradNew_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(VthetaNew_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Potential_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(VthetaInt_d, 0, size_grid*sizeof(float)));


  gpuErrchk(hipMalloc((void**)&SigmaInf_d,        (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vt_cent_d,         (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&viscosity_array_d, (NRAD+1)*sizeof(float)));

  gpuErrchk(hipMemcpy(SigmaInf_d, SigmaInf,               (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc((void**)&Energy_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&EnergyInt_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Energy_d, 0, size_grid*sizeof(float)));

  if (Adiabatic)
  gpuErrchk(hipMemcpy(Energy_d, Energy,  size_grid*sizeof(float), hipMemcpyHostToDevice));

}


__host__ float ConstructSequence (float *u, float *v, int n)
{
  int i;
  float lapl = 0.0;

  for (i = 1; i < n; i++) u[i] = 2.0*v[i]-u[i-1];
  for (i = 1; i < n-1; i++) lapl += fabs(u[i+1]+u[i-1]-2.0*u[i]);

  return lapl;
}


__host__ void Init_azimutalvelocity_withSG (float *Vtheta)
{
  // !SGZeroMode
  gpuErrchk(hipMemcpy(SG_Accr, SG_Accr_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  Make1Dprofile(1);

  Azimutalvelocity_withSGKernel<<<dimGrid2, dimBlock2>>>(Vtheta_d, Rmed_d, FLARINGINDEX, SIGMASLOPE, ASPECTRATIO,
    GLOBAL_bufarray_d, NRAD, NSEC);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ int DetectCrash (float *array)
{
  int Crash = NO;
  float numCrush;

  gpuErrchk(hipMemcpy(array_d, array, size_grid*sizeof(float), hipMemcpyHostToDevice));
  CrashKernel<<<dimGrid2, dimBlock2>>>(array_d, NRAD, NSEC, Crash);
  gpuErrchk(hipDeviceSynchronize());

  numCrush = DeviceReduce(array_d, size_grid);
  if (numCrush > 0.0) Crash = true;
  return Crash;
}

__host__ void ComputePressureField ()
{
  ComputePressureFieldKernel<<<dimGrid2, dimBlock2>>>(SoundSpeed_d, Dens_d, Pressure_d, Adiabatic, NRAD,
    NSEC, ADIABATICINDEX, Energy_d);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ComputeSoundSpeed ()
{
  ComputeSoundSpeedKernel<<<dimGrid2, dimBlock2>>>(SoundSpeed_d, Dens_d, Rmed_d, Energy_d, NSEC, NRAD,
    Adiabatic, ADIABATICINDEX, FLARINGINDEX, ASPECTRATIO, TRANSITIONWIDTH, TRANSITIONRADIUS,
    TRANSITIONRATIO, PhysicalTime, PhysicalTimeInitial, LAMBDADOUBLING);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ComputeTemperatureField ()
{
  ComputeTemperatureFieldKernel<<<dimGrid2, dimBlock2>>>(Dens_d, Temperature_d, Pressure_d, Energy_d,
    ADIABATICINDEX, Adiabatic, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ActualiseGasVtheta (float *Vtheta, float *VthetaNew)
{
  gpuErrchk(hipMemcpy(Vtheta_d, VthetaNew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ActualiseGasVrad (float *Vrad, float *VradNew)
{
  gpuErrchk(hipMemcpy(Vrad_d, VradNew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ActualiseGasEnergy (float *Energy, float *EnergyNew)
{
  gpuErrchk(hipMemcpy(Energy_d, EnergyNew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void Substep1cudamalloc (float *Vrad, float *Vtheta)
{
  gpuErrchk(hipMemcpy(EnergyMed_d, EnergyMed,           (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(SigmaMed_d, SigmaMed,             (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(QplusMed_d, QplusMed,             (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(CoolingTimeMed_d, CoolingTimeMed, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
}


__host__ int ConditionCFL (float *Vrad, float *Vtheta , float DeltaT)
{
  //gpuErrchk(hipMemset(Vmoy_d, 0, NRAD*sizeof(float)));
  ConditionCFLKernel1D<<<dimGrid4, dimBlock>>>(Rsup_d, Rinf_d, Rmed_d, NRAD, NSEC, Vtheta_d, Vmoy_d);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemset(DT2D_d, 0, NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMemset(DT1D_d, 0, NRAD*sizeof(float)));
  gpuErrchk(hipMemset(CFL_d, 0, sizeof(int)));
  ConditionCFLKernel2D1<<<dimGrid2, dimBlock2>>>(Rsup_d, Rinf_d, Rmed_d, NSEC, NRAD,
    Vresidual_d, Vtheta_d, Vmoy_d, FastTransport, SoundSpeed_d, Vrad_d, DT2D_d);
  gpuErrchk(hipDeviceSynchronize());

  ConditionCFLKernel2D2<<<dimGrid4, dimBlock>>>(newDT_d, DT2D_d, DT1D_d, Vmoy_d, invRmed_d,
    CFL_d, NSEC, NRAD, DeltaT);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(CFL, CFL_d,  sizeof(int), hipMemcpyDeviceToHost));

  return CFL[0];
}


__host__ float CircumPlanetaryMass (float *Dens, PlanetarySystem *sys)
{
  double xpl, ypl, mdcp0;

  xpl = sys->x[0];
  ypl = sys->y[0];

  CircumPlanetaryMassKernel<<<dimGrid2, dimBlock2>>> (Dens_d, Surf_d, CellAbscissa_d, CellOrdinate_d, xpl, ypl, NRAD, NSEC, \
    HillRadius, mdcp0_d);
  gpuErrchk(hipDeviceSynchronize());

  // reduction mdcp
  mdcp0 = DeviceReduce(mdcp0_d, NRAD*NSEC);

  return mdcp0;
}
