#include "hip/hip_runtime.h"
#include "Main.cuh"

#define CFLSECURITY 0.5 /* Maximum fraction of zone size */
              /* swept in one timestep */

#define CVNR 1.41 /* Shocks are spread over CVNR zones:   */
              /* Von Neumann-Richtmyer viscosity constant */
              /* Beware of misprint in Stone and Norman's */
              /* paper : use C2^2 instead of C2           */

extern int NRAD, NSEC, YES, LogGrid, size_grid, SelfGravity, ViscosityAlpha,       \
Adiabaticc, Cooling, Corotating, MARK, NO, IsDisk, GET, Evanescent, FastTransport;

extern boolean CentrifugalBalance, ZMPlus = NO, SloppyCFL;
extern string OUTPUTDIR;

extern float RMAX, RMIN, PI, MU, R, *GLOBAL_bufarray, ADIABATICINDEX, FLARINGINDEX, *vt_int, OmegaFrame,         \
*SigmaInf, G, ASPECTRATIO, SIGMA0, SIGMASLOPE, IMPOSEDDISKDRIFT, *CoolingTimeMed, *QplusMed , *viscosity_array,   \
*Dens_d, *Rmed_d, *SG_Accr, *SG_Accr_d, *SG_Acct_d, *GLOBAL_bufarray_d, *array_d, *array, mdcp, PhysicalTime,     \
*Qplus, *Qplus_d, *EnergyInt_d, *EnergyNew_d, *VradNew_d, *invdiffRsup_d, *Potential_d, *invRinf_d, *VthetaInt_d, \
*powRmed_d, *invdiffRmed_d, *VthetaNew_d, *SigmaMed_d, *SigmaMed, *QplusMed_d, *QplusMed, *CoolingTimeMed_d,      \
*EnergyMed_d, *EnergyMed, *DensInt_d, DT, MASSTAPER, *DivergenceVelocity_d, *TAURR_d, *TAURP_d, *TAUPP_d,   \
*Rsup_d, *Vmoy_d, *invRmed_d, SGP_eps;

float *Pressure, *AspectRatioRmed, *SoundSpeed, *Temperature, *Vtheta_d, *vt_cent, *Rinf_d, *SigmaInf_d, *Vrad_d,   \
*SoundSpeed_d, *Energy_d, *AspectRatioRmed_d, *Pressure_d, *Temperature_d, *viscosity_array_d, exces_mdcp = 0.0,   \
mdcp1, MassTaper, *vt_cent_d, *DensStar_d, *DensStar, *invRmed, *invRinf, *invSurf, *invdiffRmed, \
*invdiffRsup, *Radii, *Rinf, *Rmed, *Rsup, *Surf, *TemperInt_d, *TemperInt, *VradInt, *VradInt_d, *powRmed,         \
*VthetaInt, *DensInt, *VradNew, *VthetaNew, *EnergyInt, *Potential, *EnergyNew, *DT1D_d, *DT2D_d, *newDT_d,         \
*Vresidual_d, *Vradial_d, *Vazimutal_d;


boolean CrashedDens, CrashedEnergy;

extern dim3 dimGrid2, dimBlock2, dimGrid, dimBlock;

int init = 0;
double *Radii2;

Pair DiskOnPrimaryAcceleration;



__host__ void FillPolar1DArrays ()
{
  FILE *input, *output;
  int i,j;
  float drrsep, temporary;
  string InputName, OutputName;
  drrsep = (RMAX-RMIN)/(float)NRAD;
  InputName = OUTPUTDIR + "radii.dat";
  OutputName = OUTPUTDIR + "used_rad.dat";

  /* Creo los arreglos de FillPolar1DArrays */
  Radii       = (float *)malloc((NRAD+1)*sizeof(float));
  Radii2      = (double *)malloc((NRAD+1)*sizeof(double));
  Rinf        = (float *)malloc(NRAD*sizeof(float));
  Rmed        = (float *)malloc(NRAD*sizeof(float));
  Rsup        = (float *)malloc(NRAD*sizeof(float));
  Surf        = (float *)malloc(NRAD*sizeof(float));
  invRinf     = (float *)malloc(NRAD*sizeof(float));
  invSurf     = (float *)malloc(NRAD*sizeof(float));
  invRmed     = (float *)malloc(NRAD*sizeof(float));
  invdiffRsup = (float *)malloc(NRAD*sizeof(float));
  invdiffRmed = (float *)malloc(NRAD*sizeof(float));
  vt_cent     = (float *)malloc(NRAD*sizeof(float));
  powRmed     = (float *)malloc(NRAD*sizeof(float));

  char InputCharName[100];
  char OutputCharName[100];
  /* string to char InputName */
  strncpy(InputCharName, InputName.c_str(), sizeof(InputCharName));
  InputCharName[sizeof(InputCharName)-1]=0;

  input = fopen (InputCharName, "r");
  if (input == NULL){
    printf("Warning : no `radii.dat' file found. Using default.\n");
    if (LogGrid == YES){
      for (i = 0; i <= NRAD; i++){
        /* Usamos doubles para calcular los valores de los arrays, luego
           los pasamos a float */
        Radii2[i] = RMIN*exp((double)i/(double)NRAD*log(RMAX / RMIN));
        Radii[i] = (float) Radii2[i];
      }
    }
    else {
      for (i = 0; i <= NRAD; i++)
        Radii[i] = RMIN+drrsep*i;
    }
  }
  else {
    printf("Reading 'radii.dat' file.\n");
    for (i = 0; i <= NRAD; i++){
      fscanf (input, "%f", &temporary);
      Radii[i] = (float)temporary;
    }
  }

  for (i = 0; i < NRAD; i++){
    Rinf[i] = (float) Radii2[i];
    Rsup[i] = (float) Radii2[i+1];
    Rmed[i] = (float)(2.0/3.0*(Radii2[i+1]*Radii2[i+1]*Radii2[i+1]-Radii2[i]*Radii2[i]*Radii2[i])); // 2/3*(Rsup[i]^3 - Rinf[i]^3)
    Rmed[i] = (float) (Rmed[i] / (Radii2[i+1]*Radii2[i+1]-Radii2[i]*Radii2[i])); // Rmed /(Rsup[i]^2 - Rinf[i]^2)
    Surf[i] = (float) (M_PI*(Radii2[i+1]*Radii2[i+1]-Radii2[i]*Radii2[i])/(float)NSEC);  // (Rsup[i]^2 - Rinf[i]^2)
    invRmed[i] = 1.0/Rmed[i];
    invSurf[i] = 1.0/Surf[i];
    invdiffRsup[i] = (float) (1.0/(Radii2[i+1]-Radii2[i])); // 1.0/(Rsup[i] - Rinf[i])
    invRinf[i] = 1.0/Rinf[i];
  }

  Rinf[NRAD] = Radii2[NRAD];

  for (i = 1; i < NRAD; i++) {
    invdiffRmed[i] = 1.0/(Rmed[i]-Rmed[i-1]);
    powRmed[i] = (float) pow(Rmed[i],-2.5+SIGMASLOPE);
  }

  /* string to char OutputName */
  strncpy(OutputCharName, OutputName.c_str(), sizeof(OutputCharName));
  OutputCharName[sizeof(OutputCharName)-1]=0;

  output = fopen (OutputCharName, "w");
  if (output == NULL){
    printf ("Can't write %s.\nProgram stopped.\n", OutputCharName);
    exit (1);
  }
  for (i = 0; i <= NRAD; i++){
    fprintf (output, "%f\n", Radii[i]);
  }
  fclose (output);
  if (input != NULL) fclose (input);
}



__host__ void InitEuler (float *Vrad, float *Vtheta, float *Dens, float *Energy)
{
  InitTransport ();
  InitViscosity ();
  Pressure        = (float *)malloc(size_grid*sizeof(float));
  SoundSpeed      = (float *)malloc(size_grid*sizeof(float));
  Temperature     = (float *)malloc(size_grid*sizeof(float));
  TemperInt       = (float *)malloc(size_grid*sizeof(float));
  DensStar         = (float *)malloc(size_grid*sizeof(float));
  VradInt         = (float *)malloc(size_grid*sizeof(float));
  AspectRatioRmed = (float *)malloc(NRAD*sizeof(float));
  Potential       = (float *)malloc(size_grid*sizeof(float));
  VthetaInt       = (float *)malloc(size_grid*sizeof(float));
  DensInt         = (float *)malloc(size_grid*sizeof(float));
  VradNew         = (float *)malloc(size_grid*sizeof(float));
  VthetaNew       = (float *)malloc(size_grid*sizeof(float));
  EnergyInt       = (float *)malloc(size_grid*sizeof(float));
  EnergyNew       = (float *)malloc(size_grid*sizeof(float));

  for (int i = 0; i < NRAD; i++) AspectRatioRmed[i] = AspectRatio(Rmed[i]);

  Computecudamalloc (Energy);

  InitComputeAccel ();
  ComputeSoundSpeed ();
  ComputePressureField (Dens, Energy);
  ComputeTemperatureField ();
  InitGasVelocities (Vrad, Vtheta);

}

__host__ void InitGasVelocities (float *Vrad, float *Vtheta)
{
  float r1, t1, r2, t2, r, ri;
  int i;


  /* Pressure is already initialized: cf initeuler in SourceEuler.c ...
    Initialization of azimutal velocity with exact centrifugal balance */

  if (CentrifugalBalance)
  {
    /* vt_int \equiv Romega = grad(P)/sigma + \partial(phi)/\partial(r) - acc_sg_radial
    ./bin/fargoGPU  -b in/template.par */

    gpuErrchk(hipMemcpy(Pressure, Pressure_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
    Make1Dprofile(1);

    /* global axisymmetric pressure field */
    for (i = 1; i < NRAD; i++)
    {
      vt_int[i] = ( GLOBAL_bufarray[i] - GLOBAL_bufarray[i-1]) / \
      (.5*(Sigma(Rmed[i]) + Sigma(Rmed[i-1])))/(Rmed[i]-Rmed[i-1]);
    }
    /* Case of a disk with self-gravity */
    if ( SelfGravity ) // Better test with CL rigid!
    {
      gpuErrchk(hipMemcpy(SG_Accr, SG_Accr_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
      Make1Dprofile(1);

      for (i = 1; i < NRAD; i++)
        vt_int[i] -= ((Radii[i] - Rmed[i-1]) * GLOBAL_bufarray[i] + \
          (Rmed[i] - Radii[i])* GLOBAL_bufarray[i-1]) / (Rmed[i]-Rmed[i-1]);
    }

    for (i = 1; i < NRAD; i++)
      vt_int[i] = sqrt(vt_int[i]*Radii[i]) - Radii[i]*OmegaFrame;

    t1 = vt_cent[0] = vt_int[1]+.75*(vt_int[1]-vt_int[2]);
    r1 = ConstructSequence (vt_cent, vt_int, NRAD);
    vt_cent[0] += .25*(vt_int[1]-vt_int[2]);
    t2 = vt_cent[0];
    r2 = ConstructSequence (vt_cent, vt_int, NRAD);
    t1 = t1-r1/(r2-r1)*(t2-t1);
    vt_cent[0] = t1;
    ConstructSequence (vt_cent, vt_int, NRAD);
    vt_cent[NRAD] = vt_cent[NRAD-1];
  }

  if (!CentrifugalBalance && SelfGravity)
    Init_azimutalvelocity_withSG (Vtheta);

  if (ViscosityAlpha)
  {
    gpuErrchk(hipMemcpy(SoundSpeed, SoundSpeed_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
    Make1Dprofile(1);
  }


  if (Cooling)
  {
    FillCoolingTime();
    FillQplus();
  }

  for (i = 0; i <= NRAD; i++)
  {
    if (i == NRAD) viscosity_array[i] = FViscosity(Rmed[NRAD-1]);
    else viscosity_array[i] = FViscosity(Rmed[i]);
  }

  InitVelocities(Vrad, Vtheta);
}

__host__ void AlgoGas (Force *force, float *Dens, float *Vrad, float *Vtheta, float *Energy, float *Label,
  PlanetarySystem *sys, int initialization)
{

  float dt = 0.0, OmegaNew, domega, dtemp =0.0;
  int GasTimeStepsCFL = 1, gastimestepcfl;
  CrashedDens = false;
  CrashedEnergy = false;
  if (Adiabaticc)
  {
    for (int i = 0; i < NRAD; i++) AspectRatioRmed[i] = AspectRatio(Rmed[i]);
    gpuErrchk(hipMemcpy(AspectRatioRmed_d, AspectRatioRmed, NRAD*sizeof(float), hipMemcpyHostToDevice));

    ComputeSoundSpeed();

    /* it is necesary to update computation of soundspeed if one uses
      alphaviscosity in Fviscosity. It is not necesary in locally
      isothermal runs since cs is constant. It is computed here for
      the needs of ConditionCFL. */
  }
  if (IsDisk == YES)
  {
    // communicateBoundaries -> es para mpi ?
    if (SloppyCFL == YES)
    {
      gastimestepcfl = ConditionCFL(Vrad, Vtheta, DT-dtemp);     // case ./bin/fargoGPU -c
    }

  }

  dt = DT / GasTimeStepsCFL; // es 1

  while (dtemp < 0.99999*DT)
  {
    MassTaper = PhysicalTime/(MASSTAPER*2.0*M_PI);
    MassTaper = (MassTaper > 1.0 ? 1.0 : pow(sin(MassTaper*M_PI/2.0), 2.0));

    if(IsDisk == YES)
    {
      // communicateBoundaries -> mismo que arriba
      if (SloppyCFL == NO)
      {
        gastimestepcfl = 1;
        gastimestepcfl = ConditionCFL(Vrad, Vtheta ,DT-dtemp);
        dt = (DT-dtemp)/(float)GasTimeStepsCFL;
      }
      AccreteOntoPlanets(Dens, Vrad, Vtheta, dt, sys);
    }
    dtemp += dt;
    DiskOnPrimaryAcceleration.x = 0.0;
    DiskOnPrimaryAcceleration.y = 0.0;
    if (Corotating == YES) GetPsysInfo (sys, MARK);

    if (IsDisk == YES)
    {
      /* Indirect term star's potential computed here */
      DiskOnPrimaryAcceleration = ComputeAccel (force, Dens, 0.0, 0.0, 0.0, 0.0);
      /* Gravitational potential from star and planet(s) is computed and stored here */
      FillForcesArrays (sys, Dens, Energy);
      /* Planet's velocities are update here from gravitational interaction with disk */
      AdvanceSystemFromDisk (force, Dens, Energy, sys, dt);

    }

    /* Planet's positions and velocities are update from gravitational interaction with star
       and other planets */
    AdvanceSystemRK5 (sys,dt);

    /* Below we correct vtheta, planet's position and velocities if we work in a frame non-centered on the star */
    if (Corotating == YES)
    {
      OmegaNew = GetPsysInfo(sys, GET) / dt;
      domega = OmegaNew - OmegaFrame;
      if (IsDisk == YES) CorrectVtheta (Vtheta, domega);
      OmegaFrame = OmegaNew;
    }
    RotatePsys (sys, OmegaFrame*dt);

    /* Now we update gas */
    if (IsDisk == YES)
    {
      ApplyBoundaryCondition (Dens, Energy, Vrad, Vtheta, dt);
      gpuErrchk(hipMemcpy(Dens, Dens_d,     size_grid*sizeof(float), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(Energy, Energy_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
      CrashedDens = DetectCrash (Dens);
      CrashedEnergy = DetectCrash (Energy);
      if (CrashedDens == YES || CrashedEnergy == YES)
      {
        //fprintf(stdout, "\nCrash! at time %d\n", PhysicalTime);
      }
      // if (ZMPlus) compute_anisotropic_pressurecoeff(sys);

      ComputePressureField (Dens, Energy);
      Substep1 (Dens, Vrad, Vtheta, dt, init);
      Substep2 (dt);
      ActualiseGasVrad (Vrad, VradNew);
      ActualiseGasVtheta (Vtheta, VthetaNew);
      ApplyBoundaryCondition (Dens, Energy, Vrad, Vtheta, dt);

      if (Adiabaticc)
      {
        ComputeViscousTerms (Vrad, Vtheta, Dens, 1);
        Substep3 (Dens, dt);
        ActualiseGasEnergy (Energy, EnergyNew);
      }
      Transport (Dens, Vrad, Vtheta, Energy, Label, dt);
      ApplyBoundaryCondition(Dens, Energy, Vrad, Vtheta, dt);
      ComputeTemperatureField ();
      mdcp1 = CircumPlanetaryMass (Dens, sys);
      exces_mdcp = mdcp - mdcp1;
      init = init + 1;
    }

    PhysicalTime += dt;
  }
}

__host__ void Substep1 (float *Dens, float *Vrad, float *Vtheta, float dt, int initialization)
{
  boolean selfgravityupdate;
  if(initialization == 0) Substep1cudamalloc(Vrad, Vtheta);

  Substep1Kernel<<<dimGrid2, dimBlock2>>>(Pressure_d, Dens_d, VradInt_d, invdiffRmed_d, Potential_d, Rinf_d,
    invRinf_d, Vrad_d, VthetaInt_d, Vtheta_d, Rmed_d,  dt, NRAD, NSEC, OmegaFrame, ZMPlus,
    IMPOSEDDISKDRIFT, SIGMASLOPE, powRmed_d);
  gpuErrchk(hipDeviceSynchronize());

    if (SelfGravity){
      selfgravityupdate = YES;
      compute_selfgravity(Dens, dt, selfgravityupdate, 2, 0); /* option = 2, using VradInt and VthetaInt arrays */
    }

  ComputeViscousTerms (VradInt, VthetaInt, Dens, 0);
  UpdateVelocitiesWithViscosity(VradInt, VthetaInt, Dens, dt);

  if (!Evanescent) ApplySubKeplerianBoundary(VthetaInt);

}

__host__ void Substep2 (float dt)
{
  Substep2Kernel<<<dimGrid2, dimBlock2>>>(Dens_d, VradInt_d, VthetaInt_d, TemperInt_d, NRAD, NSEC, CVNR, invdiffRmed_d,
  invdiffRsup_d, DensInt_d, Adiabaticc, Rmed_d, dt, VradNew_d, VthetaNew_d, Energy_d, EnergyInt_d);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void Substep3 (float *Dens, float dt)
{
  for (int i = 0; i < NRAD; i++) viscosity_array[i] = FViscosity(Rmed[i]);
  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  Substep3Kernel<<<dimGrid2, dimBlock2>>>(Dens_d, Qplus_d, viscosity_array_d, TAURR_d, TAURP_d , TAUPP_d, DivergenceVelocity_d,
     NRAD, NSEC, Rmed_d, Cooling, EnergyNew_d, dt, EnergyMed_d, SigmaMed_d, CoolingTimeMed_d, Energy_d,
     ADIABATICINDEX, QplusMed_d);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void Computecudamalloc (float *Energy)
{

  CoolingTimeMed  = (float *)malloc(size_grid*sizeof(float));
  QplusMed        = (float *)malloc(size_grid*sizeof(float));
  viscosity_array = (float *)malloc((NRAD+1)*sizeof(float));

  gpuErrchk(hipMalloc((void**)&TemperInt_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Temperature_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Pressure_d,    size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&SoundSpeed_d,  size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DensStar_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VradInt_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&AspectRatioRmed_d, NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DensInt_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VradNew_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaNew_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&EnergyInt_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Potential_d,            size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaInt_d,      size_grid*sizeof(float)));


  gpuErrchk(hipMalloc((void**)&SigmaInf_d,        NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vt_cent_d,         (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Energy_d,       size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&viscosity_array_d, (NRAD+1)*sizeof(float)));

  gpuErrchk(hipMemcpy(SigmaInf_d, SigmaInf,               NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Energy_d, Energy,             size_grid*sizeof(float), hipMemcpyHostToDevice));

}

__host__ float ConstructSequence (float *u, float *v, int n)
{
  int i;
  float lapl = 0.0;

  for (i = 1; i < n; i++) u[i] = 2.0*v[i]-u[i-1];
  for (i = 1; i < n-1; i++) lapl += fabs(u[i+1]+u[i-1]-2.0*u[i]);

  return lapl;
}

__host__ void Init_azimutalvelocity_withSG (float *Vtheta)
{
  // !SGZeroMode
  gpuErrchk(hipMemcpy(SG_Accr, SG_Accr_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  Make1Dprofile(1);

  Azimutalvelocity_withSGKernel<<<dimGrid2, dimBlock2>>>(Vtheta_d, Rmed_d, FLARINGINDEX, SIGMASLOPE, ASPECTRATIO, G, \
    GLOBAL_bufarray_d, NRAD, NSEC);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ boolean DetectCrash (float *array)
{
  boolean Crash = NO;
  float numCrush;

  gpuErrchk(hipMemcpy(array_d, array, size_grid*sizeof(float), hipMemcpyHostToDevice));
  CrashKernel<<<dimGrid2, dimBlock2>>>(array_d, NRAD, NSEC, Crash);
  gpuErrchk(hipDeviceSynchronize());

  numCrush = DeviceReduce(array_d, size_grid);
  if (numCrush > 0.0) Crash = true;
  return Crash;
}

__host__ void ComputePressureField (float *Dens, float *Energy)
{
  ComputePressureFieldKernel<<<dimGrid2, dimBlock2>>>(SoundSpeed_d, Dens_d, Pressure_d, Adiabaticc, NRAD,
    NSEC, ADIABATICINDEX, Energy_d);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void ComputeSoundSpeed ()
{
  ComputeSoundSpeedKernel<<<dimGrid2, dimBlock2>>>(SoundSpeed_d, Dens_d, Rmed_d, Energy_d, NSEC, NRAD,
    Adiabaticc, ADIABATICINDEX, FLARINGINDEX, AspectRatioRmed_d);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void ComputeTemperatureField ()
{

  ComputeTemperatureFieldKernel<<<dimGrid2, dimBlock2>>>(Dens_d, Temperature_d, Pressure_d, Energy_d, MU, R,
    ADIABATICINDEX, Adiabaticc, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void InitVelocities (float *Vrad, float *Vtheta)
{

  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vt_cent_d, vt_cent,     (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  InitGasVelocitiesKernel<<<dimGrid2, dimBlock2>>>(viscosity_array_d, NSEC, NRAD, SelfGravity, Rmed_d,
  G, ASPECTRATIO, FLARINGINDEX, SIGMASLOPE, CentrifugalBalance, Vrad_d, Vtheta_d, ViscosityAlpha,
  IMPOSEDDISKDRIFT, SIGMA0, SigmaInf_d, OmegaFrame, Rinf_d, vt_cent_d);

  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(Vrad, Vrad_d,     size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(Vtheta, Vtheta_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));

  //for (int i = 0; i < NSEC; i++) vrad[i] = vrad[i+NSEC*NRAD] = 0.0;

  gpuErrchk(hipMemcpy(Vrad_d, Vrad,     size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Vtheta_d, Vtheta, size_grid*sizeof(float), hipMemcpyHostToDevice));
}

__host__ void ActualiseGasVtheta (float *Vtheta, float *VthetaNew)
{
  gpuErrchk(hipMemcpy(Vtheta_d, VthetaNew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void ActualiseGasVrad (float *Vrad, float *VradNew)
{
  gpuErrchk(hipMemcpy(Vrad_d, VradNew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void ActualiseGasEnergy (float *Energy, float *EnergyNew)
{
  gpuErrchk(hipMemcpy(Energy_d, EnergyNew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void Substep1cudamalloc (float *Vrad, float *Vtheta)
{
  //gpuErrchk(hipMemcpy(VradInt_d, VradInt,               size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(Potential_d, Potential,           size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(VthetaInt_d, VthetaInt,           size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(Vrad_d, Vrad,                     size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(Vtheta_d, Vtheta,                 size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(TemperInt_d, TemperInt,           size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(DensInt_d, DensInt,               size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(VradNew_d, VradNew,               size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(VthetaNew_d, VthetaNew,           size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(energyInt_d, energyInt,           size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(Qplus_d, Qplus,                   size_grid*sizeof(float), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(energyNew_d, energyNew,           size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(EnergyMed_d, EnergyMed,           NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(SigmaMed_d, SigmaMed,             NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(QplusMed_d, QplusMed,             NRAD*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(CoolingTimeMed_d, CoolingTimeMed, NRAD*sizeof(float), hipMemcpyHostToDevice));
}

__host__ int ConditionCFL (float *Vrad, float *Vtheta , float DeltaT)
{
  ConditionCFLKernel1D<<<dimGrid, dimBlock>>>(Rsup_d, Rinf_d, Rmed_d, NRAD, NSEC, Vtheta_d, Vmoy_d);
  gpuErrchk(hipDeviceSynchronize());

  ConditionCFLKernel2D<<<dimGrid2, dimBlock2>>>(Rsup_d, Rinf_d, Rmed_d, NSEC, NRAD,
    Vresidual_d, Vtheta_d, Vmoy_d, FastTransport, SoundSpeed_d, Vrad_d, DeltaT, DT2D_d,
    CVNR, invRmed_d, DT2D_d, CFLSECURITY, newDT_d);
  gpuErrchk(hipDeviceSynchronize());
}
