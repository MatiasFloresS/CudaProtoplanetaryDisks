#include "hip/hip_runtime.h"
#include "Main.cuh"

extern int NRAD, NSEC, LogGrid, size_grid, SelfGravity, ViscosityAlpha;
extern int Adiabatic, Cooling, Corotating, IsDisk, Evanescent, FastTransport;
extern int CentrifugalBalance, ZMPlus = NO, SloppyCFL, *CFL_d, *CFL;

extern string OUTPUTDIR;

extern float RMAX, RMIN, ADIABATICINDEX, FLARINGINDEX, ASPECTRATIO;
extern float SIGMA0, SIGMASLOPE, IMPOSEDDISKDRIFT, DT, MASSTAPER;
extern float TRANSITIONWIDTH, TRANSITIONRATIO, TRANSITIONRADIUS;
extern float LAMBDADOUBLING;

extern float SGP_eps, PhysicalTime, PhysicalTimeInitial, mdcp, *axifield_d;
extern float *GLOBAL_bufarray, *vt_int, *SigmaInf, *CoolingTimeMed, *QplusMed , *viscosity_array;
extern float *SG_Accr, *array, *Qplus, *SigmaMed,  *EnergyMed, *CellOrdinate, *CellAbscissa;


extern float *Dens_d, *VradNew_d, *VthetaInt_d, *VthetaNew_d, *EnergyInt_d, *EnergyNew_d, *DensInt_d;

extern float *SG_Accr_d, *SG_Acct_d, *GLOBAL_bufarray_d, *array_d;
extern float *Qplus_d;

extern float *Potential_d;

extern float *powRmed_d, *SigmaMed_d, *QplusMed_d;
extern float *CoolingTimeMed_d, *EnergyMed_d, *DivergenceVelocity_d, *TAURR_d, *TAURP_d;
extern float *TAUPP_d, *Vmoy_d, *CellOrdinate_d, *CellAbscissa_d, *mdcp0_d;

extern float *Surf_d;
extern float *example;
extern float *invdiffRmed_d, *invRinf_d, *Rmed_d, *invRmed_d, *invdiffRsup_d, *Rsup_d;

float *invdiffRmed, *invRinf, *Rinf, *Rinf_d, *invRmed, *Rmed, *invdiffRsup, *Rsup;

float *Vtheta_d, *Energy_d, *Vrad_d, *VradInt, *VthetaInt, *VradNew, *VthetaNew, *Vresidual_d, *Vradial_d, *Vazimutal_d;
float *VradInt_d, *EnergyInt, *EnergyNew, *DensInt, *Temperature, *TemperInt, *Temperature_d, *TemperInt_d;

float *Pressure, *SoundSpeed;
float *SoundSpeed_d, *Pressure_d;
float *Potential;

float *DensStar;
float *invSurf, *Radii, *Surf, *powRmed,  *vt_cent;

float *SigmaInf_d;
float *viscosity_array_d, *vt_cent_d, *DensStar_d, *DT1D_d;
float *DT2D_d, *newDT_d;


float *DT2D;
float exces_mdcp = 0.0, mdcp1, MassTaper;

int CrashedDens, CrashedEnergy;

extern dim3 dimGrid2, dimBlock2, dimGrid4, dimBlock;

int init = 0;

extern float OmegaFrame, HillRadius;

Pair DiskOnPrimaryAcceleration;



__host__ void FillPolar1DArrays ()
{
  FILE *input, *output;
  int i,j;
  float drrsep, temporary;
  string InputName, OutputName;
  drrsep = (RMAX-RMIN)/(float)NRAD;
  InputName = OUTPUTDIR + "radii.dat";
  OutputName = OUTPUTDIR + "used_rad.dat";

  /* Creo los arreglos de FillPolar1DArrays */
  Radii       = (float *)malloc((NRAD+1)*sizeof(float));
  Rinf        = (float *)malloc((NRAD+1)*sizeof(float));
  Rmed        = (float *)malloc((NRAD+1)*sizeof(float));
  Rsup        = (float *)malloc((NRAD+1)*sizeof(float));
  Surf        = (float *)malloc((NRAD+1)*sizeof(float));
  invRinf     = (float *)malloc((NRAD+1)*sizeof(float));
  invSurf     = (float *)malloc((NRAD+1)*sizeof(float));
  invRmed     = (float *)malloc((NRAD+1)*sizeof(float));
  invdiffRsup = (float *)malloc((NRAD+1)*sizeof(float));
  invdiffRmed = (float *)malloc((NRAD+1)*sizeof(float));
  vt_cent     = (float *)malloc((NRAD+1)*sizeof(float));
  powRmed     = (float *)malloc((NRAD+1)*sizeof(float));
  DT2D     = (float *)malloc(NRAD*NSEC*sizeof(float));

  char InputCharName[100];
  char OutputCharName[100];
  /* string to char InputName */
  strncpy(InputCharName, InputName.c_str(), sizeof(InputCharName));
  InputCharName[sizeof(InputCharName)-1]=0;

  input = fopen (InputCharName, "r");

  if (input == NULL){
    printf("Warning : no `radii.dat' file found. Using default.\n");
    if (LogGrid == YES){
      for (i = 0; i <= NRAD; i++){
        /* Usamos doubles para calcular los valores de los arrays, luego
           los pasamos a float */
        Radii[i] = (float)RMIN*exp((float)i/(float)NRAD*log((float)RMAX/(float)RMIN));
      }
    }
    else {
      for (i = 0; i <= NRAD; i++)
        Radii[i] = RMIN+drrsep*(float)i;
    }
  }
  else {
    printf("Reading 'radii.dat' file.\n");
    for (i = 0; i <= NRAD; i++){
      fscanf (input, "%f", &temporary);
      Radii[i] = (float)temporary;
    }
  }

  for (i = 0; i < NRAD; i++){
    Rinf[i] = Radii[i];
    Rsup[i] = Radii[i+1];
    Rmed[i] = 2.0/3.0*(Radii[i+1]*Radii[i+1]*Radii[i+1]-Radii[i]*Radii[i]*Radii[i]);
    Rmed[i] = Rmed[i] / (Radii[i+1]*Radii[i+1]-Radii[i]*Radii[i]);
    Surf[i] = PI*(Radii[i+1]*Radii[i+1]-Radii[i]*Radii[i])/(float)NSEC;
    invRmed[i] = 1.0/Rmed[i];
    invSurf[i] = 1.0/Surf[i];
    invdiffRsup[i] = 1.0/(Radii[i+1]-Radii[i]);
    invRinf[i] = 1.0/Radii[i];
  }

  Rinf[NRAD] = Radii[NRAD];

  for (i = 0; i < NRAD; i++) {
    if (i > 0 )invdiffRmed[i] = 1.0/(Rmed[i]-Rmed[i-1]);

    powRmed[i] = pow(Rmed[i],-2.5+SIGMASLOPE);
  }

  /* string to char OutputName */
  strncpy(OutputCharName, OutputName.c_str(), sizeof(OutputCharName));
  OutputCharName[sizeof(OutputCharName)-1]=0;

  output = fopen (OutputCharName, "w");
  if (output == NULL){
    printf ("Can't write %s.\nProgram stopped.\n", OutputCharName);
    exit (1);
  }
  for (i = 0; i <= NRAD; i++){
    fprintf (output, "%.30f\n", Radii[i]);
  }
  fclose (output);
  if (input != NULL) fclose (input);
}



__host__ void InitEuler (float *Vrad, float *Vtheta, float *Dens, float *Energy)
{
  InitTransport ();
  InitViscosity ();
  DensStar        = (float *)malloc(size_grid*sizeof(float));
  DensInt         = (float *)malloc(size_grid*sizeof(float));
  VradNew         = (float *)malloc(size_grid*sizeof(float));
  VradInt         = (float *)malloc(size_grid*sizeof(float));
  VthetaNew       = (float *)malloc(size_grid*sizeof(float));
  VthetaInt       = (float *)malloc(size_grid*sizeof(float));
  EnergyNew       = (float *)malloc(size_grid*sizeof(float));
  EnergyInt       = (float *)malloc(size_grid*sizeof(float));
  TemperInt       = (float *)malloc(size_grid*sizeof(float));
  Potential       = (float *)malloc(size_grid*sizeof(float));
  Pressure        = (float *)malloc(size_grid*sizeof(float));
  SoundSpeed      = (float *)malloc(size_grid*sizeof(float));
  Temperature     = (float *)malloc(size_grid*sizeof(float));
  Qplus           = (float *)malloc(size_grid*sizeof(float));

  Computecudamalloc (Energy);

  InitComputeAccel ();
  /* Rho and Energy are already initialized: cf main.cu*/
  ComputeSoundSpeed ();
  ComputePressureField ();
  ComputeTemperatureField ();
  //ComputeViscosity();
  InitGasVelocities (Vrad, Vtheta);

}



__host__ void AlgoGas (Force *force, float *Dens, float *Vrad, float *Vtheta, float *Energy, float *Label,
  PlanetarySystem *sys, int initialization)
{
  float dt, dtemp =0.0;
  float OmegaNew, domega;
  int gastimestepcfl = 1;
  CrashedDens = 0;
  CrashedEnergy = 0;

  if (Adiabatic){
    ComputeSoundSpeed();
    /* it is necesary to update computation of soundspeed if one uses
      alphaviscosity in Fviscosity. It is not necesary in locally
      isothermal runs since cs is constant. It is computed here for
      the needs of ConditionCFL. */
  }
  if (IsDisk == YES){
    if (SloppyCFL == YES){
      gastimestepcfl = ConditionCFL(Vrad, Vtheta, DT-dtemp);     // case ./bin/fargoGPU -c
    }
  }

  dt = DT / gastimestepcfl;
  //int cont = 0;
  while (dtemp < 0.999999999*DT){
    MassTaper = PhysicalTime/(MASSTAPER*2.0*M_PI);
    MassTaper = (MassTaper > 1.0 ? 1.0 : pow(sin(MassTaper*M_PI/2.0), 2.0));
    if(IsDisk == YES){
      if (SloppyCFL == NO){
        gastimestepcfl = 1;
        gastimestepcfl = ConditionCFL(Vrad, Vtheta ,DT-dtemp); //revisar
        dt = (DT-dtemp)/(float)gastimestepcfl;
        //printf("%d\n", gastimestepcfl);
      }
      AccreteOntoPlanets(Dens, Vrad, Vtheta, dt, sys); // si existe acrecion entra
    }
    dtemp += dt;
    DiskOnPrimaryAcceleration.x = 0.0;
    DiskOnPrimaryAcceleration.y = 0.0;
    if (Corotating == YES) GetPsysInfo (sys, MARK);


    if (IsDisk == YES){
      /* Indirect term star's potential computed here */
      DiskOnPrimaryAcceleration = ComputeAccel (force, Dens, 0.0, 0.0, 0.0, 0.0);

      /* Gravitational potential from star and planet(s) is computed and stored here */
      FillForcesArrays (sys, Dens, Energy);

      /* Planet's velocities are update here from gravitational interaction with disk */
      AdvanceSystemFromDisk (force, Dens, Energy, sys, dt);
    }

    /* Planet's positions and velocities are update from gravitational interaction with star
       and other planets */
    AdvanceSystemRK5 (sys,dt);

    /* Below we correct vtheta, planet's position and velocities if we work in a frame non-centered on the star */
    if (Corotating == YES){
      OmegaNew = GetPsysInfo(sys, GET) / dt;
      domega = OmegaNew - OmegaFrame;
      if (IsDisk == YES) CorrectVtheta (Vtheta, domega);
      OmegaFrame = OmegaNew;
    }
    RotatePsys (sys, OmegaFrame*dt);

    /* Now we update gas */
    if (IsDisk == YES){
      ApplyBoundaryCondition (Dens, Energy, Vrad, Vtheta, dt);
      /*gpuErrchk(hipMemcpy(Dens, Dens_d,     size_grid*sizeof(float), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(Energy, Energy_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
      CrashedDens = DetectCrash (Dens);
      CrashedEnergy = DetectCrash (Energy);
      if (CrashedDens == YES || CrashedEnergy == YES){
        fprintf(stdout, "\nCrash! at time %d\n", PhysicalTime);
        printf("c");
      }
      else*/
      printf(".");
      //if (ZMPlus) compute_anisotropic_pressurecoeff(sys);

      ComputePressureField ();
      Substep1 (Dens, Vrad, Vtheta, dt, init);
      Substep2 (dt);
      host (dt);

      ActualiseGasVrad (Vrad, VradNew);
      ActualiseGasVtheta (Vtheta, VthetaNew);
      ApplyBoundaryCondition (Dens, Energy, Vrad, Vtheta, dt);

      if (Adiabatic){
        //gpuErrchk(hipMemcpy(Vradial_d, Vrad_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
        //gpuErrchk(hipMemcpy(Vazimutal_d, Vtheta_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
        ComputeViscousTerms (Vrad_d, Vtheta_d, Dens);
        //gpuErrchk(hipMemcpy(Vrad_d, Vradial_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
        //gpuErrchk(hipMemcpy(Vtheta_d, Vazimutal_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));

        Substep3 (Dens, dt);
        ActualiseGasEnergy (Energy, EnergyNew);
      }

      Transport (Dens, Vrad, Vtheta, Energy, Label, dt);
      ApplyBoundaryCondition(Dens, Energy, Vrad, Vtheta, dt);
      ComputeTemperatureField ();

      //mdcp1 = CircumPlanetaryMass (Dens, sys);
      //exces_mdcp = mdcp1 - mdcp;
    }
    init = init + 1;
    //cont+=1;
    PhysicalTime += dt;


  }
  printf("\n" );
}



__host__ void Substep1 (float *Dens, float *Vrad, float *Vtheta, float dt, int initialization)
{
  int selfgravityupdate;
  if(initialization == 0) Substep1cudamalloc(Vrad, Vtheta);

  Substep1Kernel<<<dimGrid2, dimBlock2>>>(Pressure_d, Dens_d, VradInt_d, invdiffRmed_d, Potential_d, Rinf_d,
    invRinf_d, Vrad_d, VthetaInt_d, Vtheta_d, Rmed_d,  dt, NRAD, NSEC, OmegaFrame, ZMPlus,
    IMPOSEDDISKDRIFT, SIGMASLOPE);
  gpuErrchk(hipDeviceSynchronize());


  if (SelfGravity){
    selfgravityupdate = YES;

    /* We copy VradInt to Vradial -> device to device */
    gpuErrchk(hipMemcpy(Vradial_d, VradInt_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(Vazimutal_d, VthetaInt_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));

    compute_selfgravity(Dens, dt, selfgravityupdate, 0);
    /* Vradialto VradInt -> device to device */
    gpuErrchk(hipMemcpy(VradInt_d, Vradial_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(VthetaInt_d, Vazimutal_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  }

  //gpuErrchk(hipMemcpy(Vradial_d, VradInt_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  //gpuErrchk(hipMemcpy(Vazimutal_d, VthetaInt_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  ComputeViscousTerms (VradInt_d, VthetaInt_d, Dens);
  UpdateVelocitiesWithViscosity(VradInt, VthetaInt, Dens, dt);

  if (!Evanescent)
    ApplySubKeplerianBoundary(VthetaInt);

}



__host__ void Substep2 (float dt)
{
  Substep2Kernel<<<dimGrid2, dimBlock2>>>(Dens_d, VradInt_d, VthetaInt_d, TemperInt_d, NRAD, NSEC, invdiffRmed_d,
  invdiffRsup_d, DensInt_d, Adiabatic, Rmed_d, dt, VradNew_d, VthetaNew_d, Energy_d, EnergyInt_d);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void host (float dt)
{
  kernel<<<dimGrid2, dimBlock2>>>(Dens_d, VradInt_d, VthetaInt_d, TemperInt_d, NRAD, NSEC, invdiffRmed_d,
  invdiffRsup_d, DensInt_d, Adiabatic, Rmed_d, dt, VradNew_d, VthetaNew_d, Energy_d, EnergyInt_d);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void Substep3 (float *Dens, float dt)
{
  for (int i = 0; i < NRAD; i++) viscosity_array[i] = FViscosity(Rmed[i]);
  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  Substep3Kernel<<<dimGrid2, dimBlock2>>>(Dens_d, Qplus_d, viscosity_array_d, TAURR_d, TAURP_d , TAUPP_d, DivergenceVelocity_d,
     NRAD, NSEC, Rmed_d, Cooling, EnergyNew_d, dt, EnergyMed_d, SigmaMed_d, CoolingTimeMed_d, EnergyInt_d,
     ADIABATICINDEX, QplusMed_d);
  gpuErrchk(hipDeviceSynchronize());
  Substep3Kernel2<<<dimGrid2, dimBlock2>>>(Dens_d, Qplus_d, viscosity_array_d, TAURR_d, TAURP_d , TAUPP_d, DivergenceVelocity_d,
     NRAD, NSEC, Rmed_d, Cooling, EnergyNew_d, dt, EnergyMed_d, SigmaMed_d, CoolingTimeMed_d, EnergyInt_d,
     ADIABATICINDEX, QplusMed_d);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void Computecudamalloc (float *Energy)
{

  CoolingTimeMed  = (float *)malloc((NRAD+1)*sizeof(float));
  QplusMed        = (float *)malloc((NRAD+1)*sizeof(float));
  viscosity_array = (float *)malloc((NRAD+1)*sizeof(float));

  gpuErrchk(hipMalloc((void**)&Temperature_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Pressure_d,    size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&SoundSpeed_d,  size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DensStar_d,    size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VradInt_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DensInt_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VradNew_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaNew_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Potential_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaInt_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TemperInt_d,   size_grid*sizeof(float)));

  gpuErrchk(hipMemset(TemperInt_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Temperature_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Pressure_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(SoundSpeed_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(DensStar_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(VradInt_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(DensInt_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(VradNew_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(VthetaNew_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Potential_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(VthetaInt_d, 0, size_grid*sizeof(float)));


  gpuErrchk(hipMalloc((void**)&SigmaInf_d,        (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vt_cent_d,         (NRAD+1)*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&viscosity_array_d, (NRAD+1)*sizeof(float)));

  gpuErrchk(hipMemcpy(SigmaInf_d, SigmaInf,               (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));


  gpuErrchk(hipMalloc((void**)&Energy_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&EnergyInt_d,   size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Energy_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(EnergyInt_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(vt_cent_d, 0, (NRAD+1)*sizeof(float)));

  if (Adiabatic)
  gpuErrchk(hipMemcpy(Energy_d, Energy,  size_grid*sizeof(float), hipMemcpyHostToDevice));

}


__host__ float ConstructSequence (float *u, float *v, int n)
{
  int i;
  float lapl = 0.0;

  for (i = 1; i < n; i++) u[i] = 2.0*v[i]-u[i-1];
  for (i = 1; i < n-1; i++) lapl += fabs(u[i+1]+u[i-1]-2.0*u[i]);

  return lapl;
}


__host__ void Init_azimutalvelocity_withSG (float *Vtheta)
{
  // !SGZeroMode
  gpuErrchk(hipMemcpy(SG_Accr, SG_Accr_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  Make1Dprofile(1);

  Azimutalvelocity_withSGKernel<<<dimGrid2, dimBlock2>>>(Vtheta_d, Rmed_d, FLARINGINDEX, SIGMASLOPE, ASPECTRATIO,
    axifield_d, NRAD, NSEC);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ int DetectCrash (float *array)
{
  int Crash = NO;
  float numCrush;

  gpuErrchk(hipMemcpy(array_d, array, size_grid*sizeof(float), hipMemcpyHostToDevice));
  CrashKernel<<<dimGrid2, dimBlock2>>>(array_d, NRAD, NSEC, Crash);
  gpuErrchk(hipDeviceSynchronize());

  numCrush = DeviceReduce(array_d, size_grid);
  if (numCrush > 0.0) Crash = true;
  return Crash;
}

__host__ void ComputePressureField ()
{
  ComputePressureFieldKernel<<<dimGrid2, dimBlock2>>>(SoundSpeed_d, Dens_d, Pressure_d, Adiabatic, NRAD,
    NSEC, ADIABATICINDEX, Energy_d);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ComputeSoundSpeed ()
{
  ComputeSoundSpeedKernel<<<dimGrid2, dimBlock2>>>(SoundSpeed_d, Dens_d, Rmed_d, Energy_d, NSEC, NRAD,
    Adiabatic, ADIABATICINDEX, FLARINGINDEX, ASPECTRATIO, TRANSITIONWIDTH, TRANSITIONRADIUS,
    TRANSITIONRATIO, PhysicalTime, PhysicalTimeInitial, LAMBDADOUBLING);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ComputeTemperatureField ()
{
  ComputeTemperatureFieldKernel<<<dimGrid2, dimBlock2>>>(Dens_d, Temperature_d, Pressure_d, Energy_d,
    ADIABATICINDEX, Adiabatic, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ActualiseGasVtheta (float *Vtheta, float *VthetaNew)
{
  gpuErrchk(hipMemcpy(Vtheta_d, VthetaNew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ActualiseGasVrad (float *Vrad, float *VradNew)
{
  gpuErrchk(hipMemcpy(Vrad_d, VradNew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ActualiseGasEnergy (float *Energy, float *EnergyNew)
{
  gpuErrchk(hipMemcpy(Energy_d, EnergyNew_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void Substep1cudamalloc (float *Vrad, float *Vtheta)
{
  gpuErrchk(hipMemcpy(QplusMed_d, QplusMed,             (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(CoolingTimeMed_d, CoolingTimeMed, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));
}


__host__ int ConditionCFL (float *Vrad, float *Vtheta , float DeltaT)
{
  ConditionCFLKernel1D<<<dimGrid4, dimBlock>>>(Rsup_d, Rinf_d, Rmed_d, NRAD, NSEC, Vtheta_d, Vmoy_d);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemset(DT2D_d, 0, NRAD*NSEC*sizeof(float)));
  gpuErrchk(hipMemset(DT1D_d, 0, NRAD*sizeof(float)));
  gpuErrchk(hipMemset(CFL_d, 0, sizeof(int)));


  ConditionCFLKernel2D1<<<dimGrid2, dimBlock2>>>(Rsup_d, Rinf_d, Rmed_d, NSEC, NRAD,
    Vresidual_d, Vtheta_d, Vmoy_d, FastTransport, SoundSpeed_d, Vrad_d, DT2D_d);
  gpuErrchk(hipDeviceSynchronize());


  ConditionCFLKernel2D2<<<dimGrid4, dimBlock>>>(newDT_d, DT2D_d, DT1D_d, Vmoy_d, invRmed_d,
    CFL_d, NSEC, NRAD, DeltaT);
  gpuErrchk(hipDeviceSynchronize());

  ConditionCFLKernel2D3<<<dimGrid4, dimBlock>>>(newDT_d, DT2D_d, DT1D_d, Vmoy_d, invRmed_d,
    CFL_d, NSEC, NRAD, DeltaT);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(CFL, CFL_d,  sizeof(int), hipMemcpyDeviceToHost));

  return CFL[0];
}


__host__ float CircumPlanetaryMass (float *Dens, PlanetarySystem *sys)
{
  float xpl, ypl, mdcp0;
  float cont=0.0;
  xpl = sys->x[0];
  ypl = sys->y[0];

  CircumPlanetaryMassKernel<<<dimGrid2, dimBlock2>>> (Dens_d, Surf_d, CellAbscissa_d, CellOrdinate_d, xpl, ypl, NRAD, NSEC, \
    HillRadius, mdcp0_d);
  gpuErrchk(hipDeviceSynchronize());

  mdcp0 = DeviceReduce(mdcp0_d, NRAD*NSEC);

  return mdcp0;
}
