#include "hip/hip_runtime.h"
#include "SideEuler.cuh"
#include "kernels.cuh"
#include "SourceEuler.cuh"

extern int OpenInner, YES, blocksize, NSEC, size_grid, NonReflecting, Adiabaticc, NRAD, nsec2pot, nrad2pot;
extern float *SigmaMed, *vrad, *dens, *energy, *Rmed, *SoundSpeed, *AspectRatioRmed, *Rinf;
extern float ADIABATICINDEX, FLARINGINDEX;
float *mean_dens, *mean_energy, mean_dens_r, mean_energy_r;
float *cs0, *cs1, cs0_r, cs1_r;

__host__ void ApplyBoundaryCondition (float *vrad, float *vtheta, float *dens, float *energy, float step)
{
  float *vrad_d, *dens_d, *energy_d, *Rmed_d, *SoundSpeed_d, *AspectRatioRmed_d;

  if(OpenInner == YES) OpenBoundaryhost(vrad, dens, energy);

  if (NonReflecting == YES)
  {
    if (Adiabaticc) ComputeSoundSpeedhost(dens, energy);
    NonReflectingBoundaryhost(vrad, dens, energy);

  }
}

__host__ void OpenBoundaryhost(float *vrad, float *dens, float *energy)
{
  float *vrad_d, *dens_d, *energy_d;

  dim3 dimGrid( nsec2pot/blocksize, 1);
  dim3 dimBlock( blocksize, 1);

  gpuErrchk(hipMalloc((void**)&vrad_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dens_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&energy_d,size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(vrad_d, vrad, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dens_d, dens, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(energy_d, energy, size_grid*sizeof(float), hipMemcpyHostToDevice));

  OpenBoundary<<<dimGrid, dimBlock>>> (vrad_d, dens_d, energy_d, NSEC, SigmaMed);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(vrad, vrad_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));

  hipFree(vrad_d);
  hipFree(dens_d);
  hipFree(energy_d);

}

__host__ void NonReflectingBoundaryhost(float *vrad, float *dens, float *energy)
{
  dim3 dimGrid( nsec2pot/blocksize, 1);
  dim3 dimBlock( blocksize, 1);

  ReduceCshost();

  float dangle, i_angle;
  float *dens_d, *energy_d, *SoundSpeed_d, *vrad_d;
  dangle = (pow(Rinf[1],-1.5)-1.0)/(.5*(cs0_r+cs1_r));
  dangle *= (Rmed[1] - Rmed[0]);
  i_angle = (int)(dangle/2.0/HIP_PI_F*(float)NSEC+.5);


  gpuErrchk(hipMalloc((void**)&energy_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dens_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&SoundSpeed_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vrad_d, size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(energy_d, energy, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dens_d, dens, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(SoundSpeed_d, SoundSpeed, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vrad_d, vrad, size_grid*sizeof(float), hipMemcpyHostToDevice));

  NonReflectingBoundary<<<dimGrid, dimBlock>>>(dens_d, energy_d, i_angle, NSEC, vrad_d, SoundSpeed_d, SigmaMed[1]);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(dens, dens_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(energy, energy_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(vrad, vrad_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));

  hipFree(dens_d);
  hipFree(energy_d);
  hipFree(vrad_d);
  hipFree(SoundSpeed_d);

  ReduceMeanHost();

  printf("dens R %f\n", mean_dens_r);
  printf("energy R %f\n", mean_energy_r);
  printf("%f\n", SigmaMed[1]);

}

__host__ void ReduceCshost()
{
  float *cs0_d, *cs1_d, *SoundSpeed_d;

  dim3 dimGrid( nsec2pot/blocksize, 1);
  dim3 dimBlock( blocksize, 1);

  cs0 = (float *)malloc(sizeof(float)*NSEC);
  cs1 = (float *)malloc(sizeof(float)*NSEC);

  gpuErrchk(hipMalloc((void**)&cs0_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&cs1_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&SoundSpeed_d, size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(cs0_d, cs0, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(cs1_d, cs1, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(SoundSpeed_d, SoundSpeed, size_grid*sizeof(float), hipMemcpyHostToDevice));

  ReduceCs<<<dimGrid, dimBlock>>> (SoundSpeed_d, cs0_d, cs1_d, NSEC);

  gpuErrchk(hipDeviceSynchronize());


  cs0_r = deviceReduce(cs0_d, NSEC);
  cs0_r /= NSEC;
  cs1_r = deviceReduce(cs1_d, NSEC);
  cs1_r /= NSEC;

  hipFree(cs0_d);
  hipFree(cs1_d);
  hipFree(SoundSpeed_d);
}

__host__ void ReduceMeanHost()
{
  float *dens_d, *energy_d, *mean_dens_d, *mean_energy_d;

  dim3 dimGrid( nsec2pot/blocksize, 1);
  dim3 dimBlock( blocksize, 1);

  mean_dens = (float *)malloc(sizeof(float)*NSEC);
  mean_energy = (float *)malloc(sizeof(float)*NSEC);

  gpuErrchk(hipMalloc((void**)&dens_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&energy_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mean_dens_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mean_energy_d, NSEC*sizeof(float)));

  gpuErrchk(hipMemcpy(dens_d, dens, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(energy_d, energy, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(mean_dens_d, mean_dens, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(mean_dens_d, mean_energy, NSEC*sizeof(float), hipMemcpyHostToDevice));

  ReduceMean<<<dimGrid, dimBlock>>>(dens_d, energy_d, NSEC, mean_dens_d, mean_energy_d);
  gpuErrchk(hipDeviceSynchronize());

  mean_dens_r = deviceReduce(mean_dens_d, NSEC);
  mean_energy_r = deviceReduce(mean_energy_d, NSEC);

  hipFree(mean_dens_d);
  hipFree(mean_energy_d);
  hipFree(dens_d);
  hipFree(energy_d);
}
