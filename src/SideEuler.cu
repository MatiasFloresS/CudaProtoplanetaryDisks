#include "hip/hip_runtime.h"
#include "SideEuler.cuh"
#include "kernels.cuh"
#include "SourceEuler.cuh"

extern int OpenInner, YES, blocksize, NSEC, size_grid, NonReflecting, Adiabaticc, NRAD, nsec2pot, nrad2pot;
extern float *SigmaMed, *vrad, *dens, *energy, *Rmed, *SoundSpeed, *AspectRatioRmed, *Rinf, *EnergyMed;
extern float ADIABATICINDEX, FLARINGINDEX;
float *mean_dens, *mean_energy, mean_dens_r, mean_energy_r, *mean_dens2, *mean_energy2, mean_dens_r2, mean_energy_r2;
float *cs0, *cs1, cs0_r, cs1_r, csnrm1_r, csnrm2_r, *csnrm1, *csnrm2;

__host__ void ApplyBoundaryCondition (float *vrad, float *vtheta, float *dens, float *energy, float step)
{
  float *vrad_d, *dens_d, *energy_d, *Rmed_d, *SoundSpeed_d, *AspectRatioRmed_d;

  if(OpenInner == YES) OpenBoundaryhost(vrad, dens, energy);

  if (NonReflecting == YES)
  {
    if (Adiabaticc) ComputeSoundSpeedhost(dens, energy);
    NonReflectingBoundaryhost(vrad, dens, energy);

  }
}

__host__ void OpenBoundaryhost(float *vrad, float *dens, float *energy)
{
  float *vrad_d, *dens_d, *energy_d;

  dim3 dimGrid( nsec2pot/blocksize, 1);
  dim3 dimBlock( blocksize, 1);

  gpuErrchk(hipMalloc((void**)&vrad_d,size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dens_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&energy_d,size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(vrad_d, vrad, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dens_d, dens, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(energy_d, energy, size_grid*sizeof(float), hipMemcpyHostToDevice));

  OpenBoundary<<<dimGrid, dimBlock>>> (vrad_d, dens_d, energy_d, NSEC, SigmaMed);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(vrad, vrad_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));

  hipFree(vrad_d);
  hipFree(dens_d);
  hipFree(energy_d);

}

__host__ void NonReflectingBoundaryhost(float *vrad, float *dens, float *energy)
{
  dim3 dimGrid( nsec2pot/blocksize, 1);
  dim3 dimBlock( blocksize, 1);

  ReduceCshost();
  int i;
  float dangle, i_angle, dangle2, i_angle2;
  float *dens_d, *energy_d, *SoundSpeed_d, *vrad_d;
  i = 1;
  dangle = (pow(Rinf[i],-1.5)-1.0)/(.5*(cs0_r+cs1_r));
  dangle *= (Rmed[i] - Rmed[i-1]);
  i_angle = (int)(dangle/2.0/HIP_PI_F*(float)NSEC+.5);

  i = NRAD-1;
  dangle2 = (pow(Rinf[i-1],-1.5)-1.0)/(.5*(csnrm1_r+csnrm2_r));
  dangle2 *= (Rmed[i]-Rmed[i-1]);
  i_angle2 = (int)(dangle/2.0/HIP_PI_F*(float)NSEC+.5);

  gpuErrchk(hipMalloc((void**)&energy_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dens_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&SoundSpeed_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&vrad_d, size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(energy_d, energy, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dens_d, dens, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(SoundSpeed_d, SoundSpeed, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vrad_d, vrad, size_grid*sizeof(float), hipMemcpyHostToDevice));

  NonReflectingBoundary<<<dimGrid, dimBlock>>>(dens_d, energy_d, i_angle, NSEC, vrad_d, SoundSpeed_d, SigmaMed[1], NRAD,
  SigmaMed[i-1], i_angle2);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(dens, dens_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(energy, energy_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(vrad, vrad_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));

  hipFree(dens_d);
  hipFree(energy_d);
  hipFree(vrad_d);
  hipFree(SoundSpeed_d);

  ReduceMeanHost();

  printf("dens R %f\n", mean_dens_r);
  printf("energy R %f\n", mean_energy_r);
  printf("dens R2 %f\n", mean_dens_r2);
  printf("energy R2 %f\n", mean_energy_r2);
  printf("%f\n", SigmaMed[1]);

  MinusMeanHost();

}

__host__ void ReduceCshost()
{
  float *cs0_d, *cs1_d, *SoundSpeed_d, *csnrm1_d, *csnrm2_d;

  dim3 dimGrid( nsec2pot/blocksize, 1);
  dim3 dimBlock( blocksize, 1);

  cs0 = (float *)malloc(sizeof(float)*NSEC);
  cs1 = (float *)malloc(sizeof(float)*NSEC);
  csnrm1 = (float *)malloc(sizeof(float)*NSEC);
  csnrm2 = (float *)malloc(sizeof(float)*NSEC);

  gpuErrchk(hipMalloc((void**)&cs0_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&cs1_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&csnrm1_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&csnrm2_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&SoundSpeed_d, size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(cs0_d, cs0, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(cs1_d, cs1, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(csnrm1_d, csnrm1, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(csnrm2_d, csnrm2, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(SoundSpeed_d, SoundSpeed, size_grid*sizeof(float), hipMemcpyHostToDevice));

  ReduceCs<<<dimGrid, dimBlock>>> (SoundSpeed_d, cs0_d, cs1_d, csnrm1_d, csnrm2_d, NSEC, NRAD);

  gpuErrchk(hipDeviceSynchronize());


  cs0_r = deviceReduce(cs0_d, NSEC);
  cs0_r /= NSEC;
  cs1_r = deviceReduce(cs1_d, NSEC);
  cs1_r /= NSEC;

  csnrm1_r = deviceReduce(csnrm1_d, NSEC);
  csnrm1_r /= NSEC;

  csnrm2_r = deviceReduce(csnrm2_d, NSEC);
  csnrm2_r /= NSEC;

  hipFree(cs0_d);
  hipFree(cs1_d);
  hipFree(csnrm1_d);
  hipFree(csnrm2_d);
  hipFree(SoundSpeed_d);
}

__host__ void ReduceMeanHost()
{
  float *dens_d, *energy_d, *mean_dens_d, *mean_energy_d, *mean_dens_d2, *mean_energy_d2;

  dim3 dimGrid( nsec2pot/blocksize, 1);
  dim3 dimBlock( blocksize, 1);

  mean_dens = (float *)malloc(sizeof(float)*NSEC);
  mean_dens2 = (float *)malloc(sizeof(float)*NSEC);
  mean_energy = (float *)malloc(sizeof(float)*NSEC);
  mean_energy2 = (float *)malloc(sizeof(float)*NSEC);

  gpuErrchk(hipMalloc((void**)&dens_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&energy_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mean_dens_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mean_energy_d, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mean_dens_d2, NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&mean_energy_d2, NSEC*sizeof(float)));

  gpuErrchk(hipMemcpy(dens_d, dens, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(energy_d, energy, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(mean_dens_d, mean_dens, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(mean_energy_d, mean_energy, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(mean_dens_d2, mean_dens2, NSEC*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(mean_energy_d2, mean_energy2, NSEC*sizeof(float), hipMemcpyHostToDevice));

  ReduceMean<<<dimGrid, dimBlock>>>(dens_d, energy_d, NSEC, mean_dens_d, mean_energy_d, mean_dens_d2, mean_energy_d2, NRAD);
  gpuErrchk(hipDeviceSynchronize());

  mean_dens_r = deviceReduce(mean_dens_d, NSEC);
  mean_energy_r = deviceReduce(mean_energy_d, NSEC);
  mean_dens_r2 = deviceReduce(mean_dens_d2, NSEC);
  mean_energy_r2 = deviceReduce(mean_energy_d2, NSEC);

  hipFree(mean_dens_d);
  hipFree(mean_energy_d);
  hipFree(mean_dens_d2);
  hipFree(mean_energy_d2);
  hipFree(dens_d);
  hipFree(energy_d);
}

__host__ void MinusMeanHost()
{
  dim3 dimGrid( nsec2pot/blocksize, 1);
  dim3 dimBlock( blocksize, 1);

  float *dens_d, *energy_d;
  gpuErrchk(hipMalloc((void**)&dens_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&energy_d, size_grid*sizeof(float)));

  gpuErrchk(hipMemcpy(dens_d, dens, size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(energy_d, energy, size_grid*sizeof(float), hipMemcpyHostToDevice));

  MinusMean<<<dimGrid, dimBlock>>>(dens_d, energy_d, SigmaMed[0], mean_dens_r, mean_dens_r2, mean_energy_r, mean_energy_r2,
  EnergyMed[0], NSEC, NRAD, SigmaMed[NRAD-1], EnergyMed[NRAD-1]);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(dens, dens_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(energy, energy_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));

  hipFree(dens_d);
  hipFree(energy_d);

}
