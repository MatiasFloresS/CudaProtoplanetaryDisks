#include "Main.cuh"

extern int NRAD, NSEC, size_grid, AdvecteLabel, OpenInner, Adiabatic, FastTransport;

extern double OmegaFrame;

extern double *Dens_d, *Vrad_d,  *Vtheta_d, *Label_d, *QStar_d, *Qbase_d, *Qbase2_d;
extern double *DensInt_d, *DensStar_d, *Vtheta_d, *Surf_d, *array_d;
extern double *DensStar, *QStar, *Qbase, *DensInt;

extern double *invdiffRmed_d, *Rinf_d, *Rmed_d, *invRmed_d, *Rsup_d, *invSurf_d;

extern dim3 dimGrid2, dimBlock2, dimBlock, dimGrid4;

double *RadMomP, *RadMomM, *ThetaMomP, *ThetaMomM, *Work, *QRStar, *ExtLabel, *dq;
double *VthetaRes, *TempShift;

double *RadMomP_d, *RadMomM_d, *ThetaMomP_d, *ThetaMomM_d, *Work_d, *QRStar_d, *ExtLabel_d;
double *dq_d, *LostByDisk_d, *VMed_d, *VthetaRes_d, *TempShift_d;

double LostMass = 0.0;

static int UniformTransport;

int *NoSplitAdvection_d, *Nshift_d;

__host__ void Transport (double *Dens, double *Vrad, double *Vtheta, double *Energy, double *Label, double dt)
{

  ComputeLRMomenta();

  //if (AdvecteLabel == YES) ComputeExtQty();

  /* No-Alternate Directionnal Splitting */
  OneWindRad (Dens, Vrad, Energy, dt);
  //OneWindTheta (Dens, Vtheta, Energy, dt);
  //ComputeVelocities (Dens, Vrad, Vtheta);
  //if (AdvecteLabel) ComputeSpeQty (Dens, Label, ExtLabel);
}


__host__ void ComputeLRMomenta()
{
  LRMomentaKernel<<<dimGrid2, dimBlock2>>>(RadMomP_d, RadMomM_d, ThetaMomP_d, ThetaMomM_d, Dens_d, Vrad_d, Vtheta_d,
    NRAD, NSEC, Rmed_d, OmegaFrame);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ComputeExtQty()
{
  ExtQtyKernel<<<dimGrid2, dimBlock2>>>(ExtLabel_d, Dens_d, Label_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void OneWindRad (double *Dens, double *Vrad, double *Energy, double dt)
{

  gpuErrchk(hipMemcpy(Qbase2_d, Dens_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // Dens_d -> Qbase2_d
  ComputeStarRad(Dens, Vrad, DensStar, dt);
  gpuErrchk(hipMemcpy(DensStar_d, QStar_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // QStar_d -> DensStar_d

  ActualiseGasDens (DensInt, Dens);

  gpuErrchk(hipMemcpy(Qbase_d, RadMomP_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // RadMomP_d -> Qbase_d
  VanLeerRadial (Vrad, RadMomP, dt, 0);
  gpuErrchk(hipMemcpy(RadMomP_d, Qbase_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // Qbase_d -> RadMomP_d

  gpuErrchk(hipMemcpy(Qbase_d, RadMomM_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // RadMomM_d -> Qbase_d
  VanLeerRadial (Vrad, RadMomM, dt, 0);
  gpuErrchk(hipMemcpy(RadMomM_d, Qbase_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // Qbase_d -> RadMomM_d

  gpuErrchk(hipMemcpy(Qbase_d, ThetaMomP_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // ThetaMomP -> Qbase_d
  VanLeerRadial (Vrad, ThetaMomP, dt, 0);
  gpuErrchk(hipMemcpy(ThetaMomP_d, Qbase_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // Qbase_d -> ThetaMomP

  gpuErrchk(hipMemcpy(Qbase_d, ThetaMomM_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // ThetaMomM -> Qbase_d
  VanLeerRadial (Vrad, ThetaMomP, dt, 0);
  gpuErrchk(hipMemcpy(ThetaMomM_d, Qbase_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // Qbase_d -> ThetaMomM

  if (Adiabatic)
    VanLeerRadial (Vrad, Energy, dt, 0);
  if (AdvecteLabel){
    gpuErrchk(hipMemcpy(Qbase_d, ExtLabel_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // ExtLabel_d -> Qbase_d
    VanLeerRadial (Vrad, ExtLabel, dt, 0);
    gpuErrchk(hipMemcpy(ExtLabel_d, Qbase_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // Qbase_d -> ExtLabel_d
  }

  gpuErrchk(hipMemcpy(Qbase_d, Dens_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // Dens_d -> Qbase_d
  LostMass += VanLeerRadial (Vrad, Dens, dt, 1);
  gpuErrchk(hipMemcpy(Dens_d, Qbase_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // Qbase_d -> Dens_d

}



__host__ void ComputeStarRad(double *Qbase2, double *Vrad, double *QStar, double dt)
{
  StarRadKernel<<<dimGrid2, dimBlock2>>> (Qbase2_d, Vrad_d, QStar_d, dt, NRAD, NSEC, invdiffRmed_d, Rmed_d, dq_d);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ActualiseGasDens(double *DensInt, double *Dens)
{
  gpuErrchk(hipMemcpy(DensInt_d, Dens_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}



__host__ double VanLeerRadial (double *Vrad, double *Qbase, double dt, int ReturnLost)
{
  double Lost = 0.0;
  DivisePolarGrid (Qbase, DensInt, Work);
  gpuErrchk(hipMemcpy(Qbase2_d, Work_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // Work_d -> Qbase_d
  ComputeStarRad (Work, Vrad, QRStar, dt);
  gpuErrchk(hipMemcpy(QRStar_d, QStar_d, size_grid*sizeof(double), hipMemcpyDeviceToDevice)); // Qstar_d -> QRStar_d

  VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, DensStar_d, Vrad_d,
    LostByDisk_d, NSEC, NRAD, dt, OpenInner, Qbase_d, invSurf_d);
  gpuErrchk(hipDeviceSynchronize());

  if (ReturnLost) Lost = DeviceReduce(LostByDisk_d, NSEC);

  return Lost;
}



__host__ void ComputeSpeQty (double *Dens, double *labe, double *ExtLabel)
{
  ComputeSpeQtyKernel<<<dimGrid2, dimBlock2>>>(Label_d, Dens_d, ExtLabel_d, NRAD, NSEC);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeVelocities(double *Dens, double *Label, double *ExtLabel)
{
  ComputeVelocitiesKernel<<<dimGrid2, dimBlock2>>>(Vrad_d, Vtheta_d, Dens_d, Rmed_d, ThetaMomP_d,
    ThetaMomM_d, RadMomP_d, RadMomM_d,  NRAD,  NSEC, OmegaFrame);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void OneWindTheta (double *Dens, double *Vtheta, double *Energy, double dt)
{
  ComputeAverageThetaVelocities (Vtheta, dt);
  ComputeResiduals (Vtheta, dt);
  ComputeConstantResidual (Vtheta, dt); /* Constant residual is in Vtheta from now on */
  UniformTransport = NO;
  QuantitiesAdvection (Dens, VthetaRes, Energy, dt);
  UniformTransport = YES;
  QuantitiesAdvection (Dens, Vtheta, Energy, dt);
  AdvectSHIFT (RadMomP);
  AdvectSHIFT (RadMomM);
  AdvectSHIFT (ThetaMomP);
  AdvectSHIFT (ThetaMomM);
  if (Adiabatic) AdvectSHIFT (Energy);
  if (AdvecteLabel) AdvectSHIFT (ExtLabel);
  AdvectSHIFT (Dens);
}



__host__ void AdvectSHIFT (double *array)
{
  AdvectSHIFTKernel<<<dimGrid2, dimBlock2>>> (array_d, TempShift_d, NSEC, NRAD, Nshift_d);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeConstantResidual (double *Vtheta, double dt)
{
  ComputeConstantResidualKernel<<<dimGrid2, dimBlock2>>>(VMed_d, invRmed_d, Nshift_d, NoSplitAdvection_d,
    NSEC, NRAD, dt, Vtheta_d, VthetaRes_d, Rmed_d, FastTransport);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeAverageThetaVelocities (double *Vtheta, double dt)
{
  ComputeAverageThetaVelocitiesKernel<<<dimGrid4, dimBlock>>>(Vtheta_d, VMed_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeResiduals (double *Vtheta, double dt)
{
  ComputeResidualsKernel<<<dimGrid2, dimBlock2>>>(VthetaRes_d, VMed_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void VanLeerTheta (double *Vazimutal, double *Qbase, double dt)
{
  DivisePolarGrid (Qbase, DensInt, Work);
  ComputeStarTheta (Work, Vazimutal, QRStar, dt);

  VanLeerThetaKernel<<<dimGrid2, dimBlock2>>>(Rsup_d, Rinf_d, Surf_d, dt, NRAD, NSEC, UniformTransport,
    NoSplitAdvection_d, QRStar_d, DensStar_d, Vtheta_d, Qbase_d);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void QuantitiesAdvection (double *Dens, double *Vazimutal, double *Energy, double dt)
{
  ComputeStarTheta (Dens, Vazimutal, DensStar, dt);
  ActualiseGasDens (DensInt, Dens);
  VanLeerTheta (Vazimutal, RadMomP, dt);
  VanLeerTheta (Vazimutal, RadMomM, dt);
  VanLeerTheta (Vazimutal, ThetaMomP, dt);
  VanLeerTheta (Vazimutal, ThetaMomM, dt);
  if (Adiabatic) VanLeerTheta (Vazimutal, Energy, dt);
  if (AdvecteLabel) VanLeerTheta (Vazimutal, ExtLabel, dt);
  VanLeerTheta (Vazimutal, Dens, dt); /* MUST be the last line */
}



__host__ void ComputeStarTheta (double *Qbase, double *Vtheta, double *QStar, double dt)
{
    StarThetaKernel<<<dimGrid2, dimBlock2>>> (Qbase_d, Rmed_d, Vtheta_d, QStar_d, NRAD, NSEC, dq_d, dt);
    gpuErrchk(hipDeviceSynchronize());
}



__host__ void InitTransport ()
{
  RadMomP         = (double *)malloc(size_grid*sizeof(double));
  RadMomM         = (double *)malloc(size_grid*sizeof(double));
  ThetaMomP       = (double *)malloc(size_grid*sizeof(double));
  ThetaMomM       = (double *)malloc(size_grid*sizeof(double));
  Work            = (double *)malloc(size_grid*sizeof(double));
  QRStar          = (double *)malloc(size_grid*sizeof(double));
  ExtLabel        = (double *)malloc(size_grid*sizeof(double));
  VthetaRes       = (double *)malloc(size_grid*sizeof(double));
  TempShift       = (double *)malloc(size_grid*sizeof(double));
  dq              = (double *)malloc(size_grid*sizeof(double));
  InitTransportDevice();

}



__host__ void InitTransportDevice()
{
  gpuErrchk(hipMalloc((void**)&RadMomP_d,        size_grid*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&RadMomM_d,        size_grid*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&ThetaMomP_d,      size_grid*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&ThetaMomM_d,      size_grid*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&Work_d,           size_grid*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&QRStar_d,         size_grid*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&ExtLabel_d,       size_grid*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&dq_d,             size_grid*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&LostByDisk_d,     NSEC*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&VthetaRes_d,      size_grid*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&TempShift_d,      size_grid*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&VMed_d,             NRAD*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&Nshift_d,           NRAD*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&NoSplitAdvection_d, NRAD*sizeof(int)));


  gpuErrchk(hipMemset(RadMomP_d, 0, size_grid*sizeof(double)));
  gpuErrchk(hipMemset(RadMomM_d, 0, size_grid*sizeof(double)));
  gpuErrchk(hipMemset(ThetaMomP_d, 0, size_grid*sizeof(double)));
  gpuErrchk(hipMemset(ThetaMomM_d, 0, size_grid*sizeof(double)));
  gpuErrchk(hipMemset(Work_d, 0, size_grid*sizeof(double)));
  gpuErrchk(hipMemset(QRStar_d, 0, size_grid*sizeof(double)));
  gpuErrchk(hipMemset(ExtLabel_d, 0, size_grid*sizeof(double)));
  gpuErrchk(hipMemset(dq_d, 0, size_grid*sizeof(double)));
  gpuErrchk(hipMemset(LostByDisk_d, 0, NSEC*sizeof(double)));
  gpuErrchk(hipMemset(TempShift_d, 0, size_grid*sizeof(double)));
  gpuErrchk(hipMemset(VMed_d, 0, NRAD*sizeof(double)));
  gpuErrchk(hipMemset(Nshift_d, 0, NRAD*sizeof(int)));
  gpuErrchk(hipMemset(NoSplitAdvection_d, 0, NRAD*sizeof(int)));

}
