#include "Main.cuh"

extern int NRAD, NSEC, size_grid, AdvecteLabel, OpenInner, Adiabatic, FastTransport;

extern float OmegaFrame;

extern float *Label_d, *QStar_d, *Qbase_d, *Qbase2_d;
extern float *DensStar_d, *array_d;
extern float *DensStar, *QStar, *Qbase;
extern float *invdiffRmed_d, *Rinf_d, *Rmed_d, *invRmed_d, *Rsup_d, *invSurf_d,  *Surf_d;

extern float *Dens_d, *Vrad_d, *Vtheta_d, *Energy_d,  *Vazimutal_d, *DensInt_d, *DensInt;

extern dim3 dimGrid2, dimBlock2, dimBlock, dimGrid4;

float *RadMomP, *RadMomM, *ThetaMomP, *ThetaMomM, *Work;
float *VthetaRes, *RadMomP_d, *RadMomM_d, *ThetaMomP_d, *ThetaMomM_d, *VthetaRes_d, *Work_d, *TempShift_d;

float *QRStar, *ExtLabel, *dq;
float *TempShift;
float *QRStar_d, *ExtLabel_d;
float *dq_d, *LostByDisk_d, *VMed_d;
float LostMass = 0.0;

static int UniformTransport;

int *NoSplitAdvection_d;
int *Nshift_d;

__host__ void Transport (float *Dens, float *Vrad, float *Vtheta, float *Energy, float *Label, float dt)
{

  ComputeLRMomenta(); // bien hasta aca
  //if (AdvecteLabel == YES) ComputeExtQty();

  /* No-Alternate Directionnal Splitting */
  OneWindRad (Dens, Vrad, Energy, dt);
  OneWindTheta (Dens, Vtheta, Energy, dt);
  ComputeVelocities (Dens, Vrad, Vtheta);
  //if (AdvecteLabel) ComputeSpeQty (Dens, Label, ExtLabel);
}


__host__ void ComputeLRMomenta()
{
  LRMomentaKernel<<<dimGrid2, dimBlock2>>>(RadMomP_d, RadMomM_d, ThetaMomP_d, ThetaMomM_d, Dens_d, Vrad_d, Vtheta_d,
    NRAD, NSEC, Rmed_d, OmegaFrame);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ComputeExtQty()
{
  ExtQtyKernel<<<dimGrid2, dimBlock2>>>(ExtLabel_d, Dens_d, Label_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void OneWindRad (float *Dens, float *Vrad, float *Energy, float dt)
{

  ComputeStarRad(Vrad, dt, 0);
  ActualiseGasDens (DensInt, Dens);

  VanLeerRadial (Vrad, dt, 0, 0);
  VanLeerRadial (Vrad, dt, 0, 1);
  VanLeerRadial (Vrad, dt, 0, 2);
  VanLeerRadial (Vrad, dt, 0, 3);

  if (Adiabatic)
    VanLeerRadial (Vrad, dt, 0, 4);

  if (AdvecteLabel == YES)
    VanLeerRadial (Vrad, dt, 0, 5);

  LostMass += VanLeerRadial (Vrad, dt, 1, 6);

}

__host__ void ComputeStarRad(float *Vrad, float dt, int option)
{
  if(option == 0){
    StarRadKernel<<<dimGrid2, dimBlock2>>> (Dens_d, Vrad_d, DensStar_d, dt, NRAD, NSEC, invdiffRmed_d, Rmed_d, dq_d);
    gpuErrchk(hipDeviceSynchronize());
    StarRadKernel2<<<dimGrid2, dimBlock2>>> (Dens_d, Vrad_d, DensStar_d, dt, NRAD, NSEC, invdiffRmed_d, Rmed_d, dq_d);
    gpuErrchk(hipDeviceSynchronize());
  }
  else{
    StarRadKernel<<<dimGrid2, dimBlock2>>> (Work_d, Vrad_d, QRStar_d, dt, NRAD, NSEC, invdiffRmed_d, Rmed_d, dq_d);
    gpuErrchk(hipDeviceSynchronize());
    StarRadKernel2<<<dimGrid2, dimBlock2>>> (Work_d, Vrad_d, QRStar_d, dt, NRAD, NSEC, invdiffRmed_d, Rmed_d, dq_d);
    gpuErrchk(hipDeviceSynchronize());
  }
}



__host__ void ActualiseGasDens(float *DensInt, float *Dens)
{
  gpuErrchk(hipMemcpy(DensInt_d, Dens_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
}



__host__ float VanLeerRadial (float *Vrad, float dt, int ReturnLost, int option)
{
  float Lost = 0.0;
  if(option == 0) DivisePolarGrid (RadMomP_d, DensInt_d, Work_d);
  if(option == 1) DivisePolarGrid (RadMomM_d, DensInt_d, Work_d);
  if(option == 2) DivisePolarGrid (ThetaMomP_d, DensInt_d, Work_d);
  if(option == 3) DivisePolarGrid (ThetaMomM_d, DensInt_d, Work_d);
  if(option == 4) DivisePolarGrid (Energy_d, DensInt_d, Work_d);
  if(option == 6) DivisePolarGrid (Dens_d, DensInt_d, Work_d);
  ComputeStarRad (Vrad, dt, 1);

  if (option == 0) {
    VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, DensStar_d, Vrad_d,
      LostByDisk_d, NSEC, NRAD, dt, OpenInner, RadMomP_d, invSurf_d);
    gpuErrchk(hipDeviceSynchronize());
  }

  if (option == 1) {
    VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, DensStar_d, Vrad_d,
      LostByDisk_d, NSEC, NRAD, dt, OpenInner, RadMomM_d, invSurf_d);
    gpuErrchk(hipDeviceSynchronize());
  }

  if (option == 2) {
    VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, DensStar_d, Vrad_d,
      LostByDisk_d, NSEC, NRAD, dt, OpenInner, ThetaMomP_d, invSurf_d);
    gpuErrchk(hipDeviceSynchronize());
  }

  if (option == 3) {
    VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, DensStar_d, Vrad_d,
      LostByDisk_d, NSEC, NRAD, dt, OpenInner, ThetaMomM_d, invSurf_d);
    gpuErrchk(hipDeviceSynchronize());
  }

  if (option == 4) {
    VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, DensStar_d, Vrad_d,
      LostByDisk_d, NSEC, NRAD, dt, OpenInner, Energy_d, invSurf_d);
    gpuErrchk(hipDeviceSynchronize());
  }

  if (option == 6) {
    VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, DensStar_d, Vrad_d,
      LostByDisk_d, NSEC, NRAD, dt, OpenInner, Dens_d, invSurf_d);
    gpuErrchk(hipDeviceSynchronize());
  }

  if (ReturnLost) Lost = DeviceReduce(LostByDisk_d, NSEC);

  return Lost;
}



__host__ void ComputeSpeQty (float *Dens, float *label, float *ExtLabel)
{
  ComputeSpeQtyKernel<<<dimGrid2, dimBlock2>>>(Label_d, Dens_d, ExtLabel_d, NRAD, NSEC);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeVelocities(float *Dens, float *Vrad, float *Vtheta)
{
  ComputeVelocitiesKernel<<<dimGrid2, dimBlock2>>>(Vrad_d, Vtheta_d, Dens_d, Rmed_d, ThetaMomP_d,
    ThetaMomM_d, RadMomP_d, RadMomM_d,  NRAD,  NSEC, OmegaFrame);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void OneWindTheta (float *Dens, float *Vtheta, float *Energy, float dt)
{
  ComputeAverageThetaVelocities (Vtheta, dt);
  ComputeResiduals (Vtheta, dt);
  ComputeConstantResidual (Vtheta, dt); /* Constant residual is in Vtheta from now on */
  UniformTransport = NO;
  QuantitiesAdvection (Dens, VthetaRes_d, Energy, dt, 0);
  UniformTransport = YES;
  QuantitiesAdvection (Dens, Vtheta_d, Energy, dt, 1);

  AdvectSHIFT (RadMomP_d);
  AdvectSHIFT (RadMomM_d);
  AdvectSHIFT (ThetaMomP_d);
  AdvectSHIFT (ThetaMomM_d);
  if (Adiabatic) AdvectSHIFT (Energy_d);
  //if (AdvecteLabel) AdvectSHIFT (ExtLabel_d);
  AdvectSHIFT (Dens_d);
}

__host__ void ComputeAverageThetaVelocities (float *Vtheta, float dt)
{
  ComputeAverageThetaVelocitiesKernel<<<dimGrid4, dimBlock>>>(Vtheta_d, VMed_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ComputeResiduals (float *Vtheta, float dt)
{
  ComputeResidualsKernel<<<dimGrid2, dimBlock2>>>(VthetaRes_d, VMed_d, NSEC, NRAD, Vtheta_d);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void AdvectSHIFT (float *array_d)
{
  AdvectSHIFTKernel<<<dimGrid2, dimBlock2>>> (array_d, TempShift_d, NSEC, NRAD, Nshift_d);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(array_d, TempShift_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));

}



__host__ void ComputeConstantResidual (float *Vtheta, float dt)
{
  ComputeConstantResidualKernel<<<dimGrid2, dimBlock2>>>(VMed_d, invRmed_d, Nshift_d, NoSplitAdvection_d,
    NSEC, NRAD, dt, Vtheta_d, VthetaRes_d, Rmed_d, FastTransport);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void QuantitiesAdvection (float *Dens, float *Vazimutal_d, float *Energy, float dt, int option)
{

  ComputeStarTheta (Dens_d, Vazimutal_d, DensStar_d, dt);
  ActualiseGasDens (DensInt, Dens);
  VanLeerTheta (Vazimutal_d, RadMomP_d, dt);
  VanLeerTheta (Vazimutal_d, RadMomM_d, dt);
  VanLeerTheta (Vazimutal_d, ThetaMomP_d, dt);
  VanLeerTheta (Vazimutal_d, ThetaMomM_d, dt);

  if (Adiabatic)
    VanLeerTheta (Vazimutal_d, Energy_d, dt);
  //if (AdvecteLabel)
    //VanLeerTheta (Vazimutal_d, ExtLabel_d, dt);
  VanLeerTheta (Vazimutal_d, Dens_d, dt); /* MUST be the last line */

}



__host__ void VanLeerTheta (float *Vazimutal_d, float *Qbase_d, float dt)
{
  DivisePolarGrid (Qbase_d, DensInt_d, Work_d);
  ComputeStarTheta (Work_d, Vazimutal_d, QRStar_d, dt);

  VanLeerThetaKernel<<<dimGrid2, dimBlock2>>>(Rsup_d, Rinf_d, Surf_d, dt, NRAD, NSEC, 0,
    NoSplitAdvection_d, QRStar_d, DensStar_d, Vazimutal_d, Qbase_d);
    gpuErrchk(hipDeviceSynchronize());
  }



__host__ void ComputeStarTheta (float *Qbase_d, float *Vazimutal_d, float *QStar_d, float dt)
{
    //gpuErrchk(hipMemset(dq_d, 0, size_grid*sizeof(float)));
    StarThetaKernel<<<dimGrid2, dimBlock2>>> (Qbase_d, Rmed_d, NRAD, NSEC, dq_d, dt);
    gpuErrchk(hipDeviceSynchronize());

    StarThetaKernel2<<<dimGrid2, dimBlock2>>>(Qbase_d, Rmed_d, Vazimutal_d, QStar_d, NRAD, NSEC, dq_d, dt);
    gpuErrchk(hipDeviceSynchronize());
}



__host__ void InitTransport ()
{
  RadMomP         = (float *)malloc(size_grid*sizeof(float));
  RadMomM         = (float *)malloc(size_grid*sizeof(float));
  ThetaMomP       = (float *)malloc(size_grid*sizeof(float));
  ThetaMomM       = (float *)malloc(size_grid*sizeof(float));
  Work            = (float *)malloc(size_grid*sizeof(float));
  QRStar          = (float *)malloc(size_grid*sizeof(float));
  ExtLabel        = (float *)malloc(size_grid*sizeof(float));
  VthetaRes       = (float *)malloc(size_grid*sizeof(float));
  TempShift       = (float *)malloc(size_grid*sizeof(float));
  dq              = (float *)malloc(size_grid*sizeof(float));
  InitTransportDevice();

}



__host__ void InitTransportDevice()
{
  gpuErrchk(hipMalloc((void**)&RadMomP_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&RadMomM_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ThetaMomP_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ThetaMomM_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Work_d,           size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&QRStar_d,         size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ExtLabel_d,       size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dq_d,             size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&LostByDisk_d,     NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaRes_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TempShift_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VMed_d,             NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Nshift_d,           NRAD*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&NoSplitAdvection_d, NRAD*sizeof(int)));


  gpuErrchk(hipMemset(RadMomP_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(RadMomM_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(ThetaMomP_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(ThetaMomM_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Work_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(QRStar_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(ExtLabel_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(dq_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(LostByDisk_d, 0, NSEC*sizeof(float)));
  gpuErrchk(hipMemset(VthetaRes_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(TempShift_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(VMed_d, 0, NRAD*sizeof(float)));
  gpuErrchk(hipMemset(Nshift_d, 0, NRAD*sizeof(int)));
  gpuErrchk(hipMemset(NoSplitAdvection_d, 0, NRAD*sizeof(int)));

}
