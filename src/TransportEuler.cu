#include "Main.cuh"

extern int NRAD, NSEC, size_grid, nsec2pot, blocksize, nrad2pot, AdvecteLabel, YES, OpenInner,         \
Adiabaticc, FastTransport;

extern float OmegaFrame1, *Dens_d, *Vrad_d, *Rmed_d, *Vtheta_d, *label_d, *densStar, *invdiffRmed_d,   \
*QStar, *Qbase, *QStar_d, *Qbase_d, *DensInt, *DensInt_d, *RhoStar, *RhoStar_d, *Rinf_d, *Rsup_d;

extern dim3 dimGrid2, dimBlock2, dimBlock, dimGrid4;

float *RadMomP, *RadMomM, *ThetaMomP, *ThetaMomM, *Work, *QRStar, *Extlabel, *RadMomP_d, *RadMomM_d,    \
*ThetaMomP_d, *ThetaMomM_d, *Work_d, *QRStar_d, *Extlabel_d, *dq, *dq_d, *LostByDisk_d, LostMass = 0.0, \
*VMed_d, *VthetaRes_d;

__host__ void Transport (float *Dens, float *Vrad, float *Vtheta, float *energy, float *label, float dt)
{

  ComputeLRMomenta();

  if (AdvecteLabel == YES) ComputeExtQty();

  /* No-Alternate Directionnal Splitting */
  OneWindRad (Dens, Vrad, energy, dt);
  OneWindTheta (Dens, Vtheta, energy, dt);
}


__host__ void ComputeLRMomenta()
{
  LRMomentaKernel<<<dimGrid2, dimBlock2>>>(RadMomP_d, RadMomM_d, ThetaMomP_d, ThetaMomM_d, Dens_d, Vrad_d, Vtheta_d,
    NRAD, NSEC, Rmed_d, OmegaFrame1);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void ComputeExtQty()
{
  ExtQtyKernel<<<dimGrid2, dimBlock2>>>(Extlabel_d, Dens_d, label_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void OneWindRad (float *Dens, float *Vrad, float *energy, float dt)
{
  gpuErrchk(hipMemcpy(Qbase_d, Dens_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice)); // dens_d -> Qbase_d
  ComputeStarRad(Dens, Vrad, RhoStar, dt);

  ActualiseGasDens (DensInt, Dens);
  VanLeerRadial (Vrad, RadMomP, dt, 0);
  VanLeerRadial (Vrad, RadMomM, dt, 0);
  VanLeerRadial (Vrad, ThetaMomP, dt, 0);
  VanLeerRadial (Vrad, ThetaMomP, dt, 0);
  if (Adiabaticc)
    VanLeerRadial (Vrad, energy, dt, 0);
  if (AdvecteLabel)
    VanLeerRadial (Vrad, Extlabel, dt, 0);
  LostMass += VanLeerRadial (Vrad, Dens, dt, 1);

}

__host__ void OneWindTheta (float *Dens, float *Vtheta, float *energy, float dt)
{
  ComputeAverageThetaVelocities (Vtheta, dt);
  ComputeResiduals (Vtheta, dt);
  ComputeConstantResidual (Vtheta, dt); /* Constant residual is in Vtheta from now on */

}

__host__ void ComputeConstantResidual (float *Vtheta, float dt)
{
  //ComputeConstantResidualKernel<<<>>>()
  gpuErrchk(hipDeviceSynchronize());

}
__host__ void ComputeAverageThetaVelocities (float *Vtheta, float dt)
{
  ComputeAverageThetaVelocitiesKernel<<<dimGrid4, dimBlock>>>(Vtheta_d, VMed_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void ComputeResiduals (float *Vtheta, float dt)
{
  ComputeResidualsKernel<<<dimGrid2, dimBlock2>>>(VthetaRes_d, VMed_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ActualiseGasDens(float *DensInt, float *Dens)
{
  gpuErrchk(hipMemcpy(DensInt_d, Qbase_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void ComputeStarRad(float *Qbase, float *Vrad, float *QStar, float dt)
{
  StarRadKernel<<<dimGrid2, dimBlock2>>> (Qbase_d, Vrad_d, QStar_d, dt, NRAD, NSEC, invdiffRmed_d, Rmed_d, dq_d);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ float VanLeerRadial (float *Vrad, float *Qbase, float dt, int ReturnLost)
{
  float Lost = 0.0;
  DivisePolarGrid (Qbase, DensInt, Work);
  gpuErrchk(hipMemcpy(Qbase_d, Work_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice)); // Work_d -> Qbase_d
  ComputeStarRad (Work, Vrad, QRStar, dt);

  VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, RhoStar_d, Vrad_d,
    LostByDisk_d, NSEC, NRAD, dt, OpenInner);
  gpuErrchk(hipDeviceSynchronize());

  if (ReturnLost) Lost = DeviceReduce(LostByDisk_d, NSEC);

  return Lost;
}

__host__ void InitTransport ()
{
  RadMomP         = (float *)malloc(size_grid*sizeof(float));
  RadMomM         = (float *)malloc(size_grid*sizeof(float));
  ThetaMomP       = (float *)malloc(size_grid*sizeof(float));
  ThetaMomM       = (float *)malloc(size_grid*sizeof(float));
  Work            = (float *)malloc(size_grid*sizeof(float));
  QRStar          = (float *)malloc(size_grid*sizeof(float));
  Extlabel        = (float *)malloc(size_grid*sizeof(float));
  //VthetaRes
  //Elongations
  //tempshift
  dq              = (float *)malloc(size_grid*sizeof(float));
  InitTransportDevice();

}

__host__ void InitTransportDevice()
{
  gpuErrchk(hipMalloc((void**)&RadMomP_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&RadMomM_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ThetaMomP_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ThetaMomM_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Work_d,           size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&QRStar_d,         size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Extlabel_d,       size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dq_d,             size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&LostByDisk_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaRes_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VMed_d,     NRAD*sizeof(float)));
}
