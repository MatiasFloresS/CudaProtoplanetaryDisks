#include "Main.cuh"

extern int NRAD, NSEC, size_grid, nsec2pot, blocksize, nrad2pot, AdvecteLabel, YES, OpenInner,         \
Adiabaticc, FastTransport, NO;

extern float OmegaFrame1, *Dens_d, *Vrad_d, *Rmed_d, *Vtheta_d, *label_d, *DensStar, *invdiffRmed_d,   \
*QStar, *Qbase, *QStar_d, *Qbase_d, *DensInt, *DensInt_d, *DensStar_d, *Rinf_d, *Rsup_d,      \
*invRmed_d, *Vtheta_d;

extern dim3 dimGrid2, dimBlock2, dimBlock, dimGrid4;

float *RadMomP, *RadMomM, *ThetaMomP, *ThetaMomM, *Work, *QRStar, *Extlabel, *RadMomP_d, *RadMomM_d,    \
*ThetaMomP_d, *ThetaMomM_d, *Work_d, *QRStar_d, *Extlabel_d, *dq, *dq_d, *LostByDisk_d, LostMass = 0.0, \
*VMed_d, *VthetaRes_d, *Nshift_d, *NoSplitAdvection_d, *VthetaRes;

static bool UniformTransport;

__host__ void Transport (float *Dens, float *Vrad, float *Vtheta, float *energy, float *label, float dt)
{

  ComputeLRMomenta();

  if (AdvecteLabel == YES) ComputeExtQty();

  /* No-Alternate Directionnal Splitting */
  OneWindRad (Dens, Vrad, energy, dt);
  OneWindTheta (Dens, Vtheta, energy, dt);
}


__host__ void ComputeLRMomenta()
{
  LRMomentaKernel<<<dimGrid2, dimBlock2>>>(RadMomP_d, RadMomM_d, ThetaMomP_d, ThetaMomM_d, Dens_d, Vrad_d, Vtheta_d,
    NRAD, NSEC, Rmed_d, OmegaFrame1);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void ComputeExtQty()
{
  ExtQtyKernel<<<dimGrid2, dimBlock2>>>(Extlabel_d, Dens_d, label_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void OneWindRad (float *Dens, float *Vrad, float *energy, float dt)
{
  gpuErrchk(hipMemcpy(Qbase_d, Dens_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice)); // dens_d -> Qbase_d
  ComputeStarRad(Dens, Vrad, DensStar, dt);

  ActualiseGasDens (DensInt, Dens);
  VanLeerRadial (Vrad, RadMomP, dt, 0);
  VanLeerRadial (Vrad, RadMomM, dt, 0);
  VanLeerRadial (Vrad, ThetaMomP, dt, 0);
  VanLeerRadial (Vrad, ThetaMomP, dt, 0);
  if (Adiabaticc)
    VanLeerRadial (Vrad, energy, dt, 0);
  if (AdvecteLabel)
    VanLeerRadial (Vrad, Extlabel, dt, 0);
  LostMass += VanLeerRadial (Vrad, Dens, dt, 1);

}

__host__ void OneWindTheta (float *Dens, float *Vtheta, float *energy, float dt)
{
  ComputeAverageThetaVelocities (Vtheta, dt);
  ComputeResiduals (Vtheta, dt);
  ComputeConstantResidual (Vtheta, dt); /* Constant residual is in Vtheta from now on */
  UniformTransport = NO;
  QuantitiesAdvection (Dens, VthetaRes, energy, dt);
  UniformTransport = YES;
  QuantitiesAdvection (Dens, Vtheta, energy, dt);

}

__host__ void ComputeConstantResidual (float *Vtheta, float dt)
{
  ComputeConstantResidualKernel<<<dimGrid2, dimBlock2>>>(VMed_d, invRmed_d, Nshift_d, NoSplitAdvection_d,
    NSEC, NRAD, dt, YES, NO, Vtheta_d, VthetaRes_d, Rmed_d, FastTransport);
  gpuErrchk(hipDeviceSynchronize());

}



__host__ void ComputeAverageThetaVelocities (float *Vtheta, float dt)
{
  ComputeAverageThetaVelocitiesKernel<<<dimGrid4, dimBlock>>>(Vtheta_d, VMed_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeResiduals (float *Vtheta, float dt)
{
  ComputeResidualsKernel<<<dimGrid2, dimBlock2>>>(VthetaRes_d, VMed_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ActualiseGasDens(float *DensInt, float *Dens)
{
  gpuErrchk(hipMemcpy(DensInt_d, Qbase_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeStarRad(float *Qbase, float *Vrad, float *QStar, float dt)
{
  StarRadKernel<<<dimGrid2, dimBlock2>>> (Qbase_d, Vrad_d, QStar_d, dt, NRAD, NSEC, invdiffRmed_d, Rmed_d, dq_d);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ float VanLeerRadial (float *Vrad, float *Qbase, float dt, int ReturnLost)
{
  float Lost = 0.0;
  DivisePolarGrid (Qbase, DensInt, Work);
  gpuErrchk(hipMemcpy(Qbase_d, Work_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice)); // Work_d -> Qbase_d
  ComputeStarRad (Work, Vrad, QRStar, dt);

  VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, DensStar_d, Vrad_d,
    LostByDisk_d, NSEC, NRAD, dt, OpenInner);
  gpuErrchk(hipDeviceSynchronize());

  if (ReturnLost) Lost = DeviceReduce(LostByDisk_d, NSEC);

  return Lost;
}



__host__ void QuantitiesAdvection (float *Dens, float *Vradial, float *energy, float dt)
{
  ComputeStarTheta (Dens, Vradial, DensStar, dt);
}



__host__ void ComputeStarTheta (float *Qbase, float *Vtheta, float *QStar, float dt)
{
    StarThetaKernel<<<dimGrid2, dimBlock2>>> (Qbase_d, Rmed_d, Vtheta_d, QStar_d, NRAD, NSEC, dq_d, dt);
}



__host__ void InitTransport ()
{
  RadMomP         = (float *)malloc(size_grid*sizeof(float));
  RadMomM         = (float *)malloc(size_grid*sizeof(float));
  ThetaMomP       = (float *)malloc(size_grid*sizeof(float));
  ThetaMomM       = (float *)malloc(size_grid*sizeof(float));
  Work            = (float *)malloc(size_grid*sizeof(float));
  QRStar          = (float *)malloc(size_grid*sizeof(float));
  Extlabel        = (float *)malloc(size_grid*sizeof(float));
  VthetaRes       = (float *)malloc(size_grid*sizeof(float));
  //tempshift
  dq              = (float *)malloc(size_grid*sizeof(float));
  InitTransportDevice();

}

__host__ void InitTransportDevice()
{
  gpuErrchk(hipMalloc((void**)&RadMomP_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&RadMomM_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ThetaMomP_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ThetaMomM_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Work_d,           size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&QRStar_d,         size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Extlabel_d,       size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dq_d,             size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&LostByDisk_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaRes_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VMed_d,             NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Nshift_d,           NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&NoSplitAdvection_d, NRAD*sizeof(float)));
}
