#include "Main.cuh"

extern int NRAD, NSEC, size_grid, AdvecteLabel, YES, OpenInner,         \
Adiabaticc, FastTransport, NO;

extern float OmegaFrame1, *Dens_d, *Vrad_d, *Rmed_d, *Vtheta_d, *Label_d, *DensStar, *invdiffRmed_d,   \
*QStar, *Qbase, *QStar_d, *Qbase_d, *DensInt, *DensInt_d, *DensStar_d, *Rinf_d, *Rsup_d, *invRmed_d,   \
*Vtheta_d, *Surf_d, *array_d;

extern dim3 dimGrid2, dimBlock2, dimBlock, dimGrid4;

float *RadMomP, *RadMomM, *ThetaMomP, *ThetaMomM, *Work, *QRStar, *ExtLabel, *RadMomP_d, *RadMomM_d,    \
*ThetaMomP_d, *ThetaMomM_d, *Work_d, *QRStar_d, *ExtLabel_d, *dq, *dq_d, *LostByDisk_d, LostMass = 0.0, \
*VMed_d, *VthetaRes_d, *VthetaRes, *TempShift, *TempShift_d;

static bool UniformTransport;

int *NoSplitAdvection_d, *Nshift_d;

__host__ void Transport (float *Dens, float *Vrad, float *Vtheta, float *Energy, float *Label, float dt)
{

  ComputeLRMomenta();

  if (AdvecteLabel == YES) ComputeExtQty();

  /* No-Alternate Directionnal Splitting */
  OneWindRad (Dens, Vrad, Energy, dt);
  OneWindTheta (Dens, Vtheta, Energy, dt);
  ComputeVelocities (Dens, Vrad, Vtheta);
  if (AdvecteLabel) ComputeSpeQty (Dens, Label, ExtLabel);
}



__host__ void ComputeSpeQty (float *Dens, float *labe, float *ExtLabel)
{
  ComputeSpeQtyKernel<<<dimGrid2, dimBlock2>>>(Label_d, Dens_d, ExtLabel_d, NRAD, NSEC);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeVelocities(float *Dens, float *Label, float *ExtLabel)
{
  ComputeVelocitiesKernel<<<dimGrid2, dimBlock2>>>(Vrad_d, Vtheta_d, Dens_d, Rmed_d, ThetaMomP_d,
    ThetaMomM_d, RadMomP_d, RadMomM_d,  NRAD,  NSEC, OmegaFrame1);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeLRMomenta()
{
  LRMomentaKernel<<<dimGrid2, dimBlock2>>>(RadMomP_d, RadMomM_d, ThetaMomP_d, ThetaMomM_d, Dens_d, Vrad_d, Vtheta_d,
    NRAD, NSEC, Rmed_d, OmegaFrame1);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeExtQty()
{
  ExtQtyKernel<<<dimGrid2, dimBlock2>>>(ExtLabel_d, Dens_d, Label_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void OneWindRad (float *Dens, float *Vrad, float *Energy, float dt)
{
  gpuErrchk(hipMemcpy(Qbase_d, Dens_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice)); // dens_d -> Qbase_d
  ComputeStarRad(Dens, Vrad, DensStar, dt);

  ActualiseGasDens (DensInt, Dens);
  VanLeerRadial (Vrad, RadMomP, dt, 0);
  VanLeerRadial (Vrad, RadMomM, dt, 0);
  VanLeerRadial (Vrad, ThetaMomP, dt, 0);
  VanLeerRadial (Vrad, ThetaMomP, dt, 0);
  if (Adiabaticc)
    VanLeerRadial (Vrad, Energy, dt, 0);
  if (AdvecteLabel)
    VanLeerRadial (Vrad, ExtLabel, dt, 0);
  LostMass += VanLeerRadial (Vrad, Dens, dt, 1);

}



__host__ void OneWindTheta (float *Dens, float *Vtheta, float *Energy, float dt)
{
  ComputeAverageThetaVelocities (Vtheta, dt);
  ComputeResiduals (Vtheta, dt);
  ComputeConstantResidual (Vtheta, dt); /* Constant residual is in Vtheta from now on */
  UniformTransport = NO;
  QuantitiesAdvection (Dens, VthetaRes, Energy, dt);
  UniformTransport = YES;
  QuantitiesAdvection (Dens, Vtheta, Energy, dt);
  AdvectSHIFT (RadMomP);
  AdvectSHIFT (RadMomM);
  AdvectSHIFT (ThetaMomP);
  AdvectSHIFT (ThetaMomM);
  if (Adiabaticc) AdvectSHIFT (Energy);
  if (AdvecteLabel) AdvectSHIFT (ExtLabel);
  AdvectSHIFT (Dens);
}



__host__ void AdvectSHIFT (float *array)
{
  AdvectSHIFTKernel<<<dimGrid2, dimBlock2>>> (array_d, TempShift_d, NSEC, NRAD, Nshift_d);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeConstantResidual (float *Vtheta, float dt)
{
  ComputeConstantResidualKernel<<<dimGrid2, dimBlock2>>>(VMed_d, invRmed_d, Nshift_d, NoSplitAdvection_d,
    NSEC, NRAD, dt, YES, NO, Vtheta_d, VthetaRes_d, Rmed_d, FastTransport);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeAverageThetaVelocities (float *Vtheta, float dt)
{
  ComputeAverageThetaVelocitiesKernel<<<dimGrid4, dimBlock>>>(Vtheta_d, VMed_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeResiduals (float *Vtheta, float dt)
{
  ComputeResidualsKernel<<<dimGrid2, dimBlock2>>>(VthetaRes_d, VMed_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ActualiseGasDens(float *DensInt, float *Dens)
{
  gpuErrchk(hipMemcpy(DensInt_d, Qbase_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeStarRad(float *Qbase, float *Vrad, float *QStar, float dt)
{
  StarRadKernel<<<dimGrid2, dimBlock2>>> (Qbase_d, Vrad_d, QStar_d, dt, NRAD, NSEC, invdiffRmed_d, Rmed_d, dq_d);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ float VanLeerRadial (float *Vrad, float *Qbase, float dt, int ReturnLost)
{
  float Lost = 0.0;
  DivisePolarGrid (Qbase, DensInt, Work);
  gpuErrchk(hipMemcpy(Qbase_d, Work_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice)); // Work_d -> Qbase_d
  ComputeStarRad (Work, Vrad, QRStar, dt);

  VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, DensStar_d, Vrad_d,
    LostByDisk_d, NSEC, NRAD, dt, OpenInner);
  gpuErrchk(hipDeviceSynchronize());

  if (ReturnLost) Lost = DeviceReduce(LostByDisk_d, NSEC);

  return Lost;
}



__host__ void VanLeerTheta (float *Vazimutal, float *Qbase, float dt)
{
  DivisePolarGrid (Qbase, DensInt, Work);
  ComputeStarTheta (Work, Vazimutal, QRStar, dt);

  VanLeerThetaKernel<<<dimGrid2, dimBlock2>>>(Rsup_d, Rinf_d, Surf_d, dt, NRAD, NSEC, UniformTransport,
    NoSplitAdvection_d, QRStar_d, DensStar_d, Vtheta_d, Qbase_d, NO);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void QuantitiesAdvection (float *Dens, float *Vazimutal, float *Energy, float dt)
{
  ComputeStarTheta (Dens, Vazimutal, DensStar, dt);
  ActualiseGasDens (DensInt, Dens);
  VanLeerTheta (Vazimutal, RadMomP, dt);
  VanLeerTheta (Vazimutal, RadMomM, dt);
  VanLeerTheta (Vazimutal, ThetaMomP, dt);
  VanLeerTheta (Vazimutal, ThetaMomM, dt);
  if (Adiabaticc) VanLeerTheta (Vazimutal, Energy, dt);
  if (AdvecteLabel) VanLeerTheta (Vazimutal, ExtLabel, dt);
  VanLeerTheta (Vazimutal, Dens, dt); /* MUST be the last line */
}



__host__ void ComputeStarTheta (float *Qbase, float *Vtheta, float *QStar, float dt)
{
    StarThetaKernel<<<dimGrid2, dimBlock2>>> (Qbase_d, Rmed_d, Vtheta_d, QStar_d, NRAD, NSEC, dq_d, dt);
    gpuErrchk(hipDeviceSynchronize());
}



__host__ void InitTransport ()
{
  RadMomP         = (float *)malloc(size_grid*sizeof(float));
  RadMomM         = (float *)malloc(size_grid*sizeof(float));
  ThetaMomP       = (float *)malloc(size_grid*sizeof(float));
  ThetaMomM       = (float *)malloc(size_grid*sizeof(float));
  Work            = (float *)malloc(size_grid*sizeof(float));
  QRStar          = (float *)malloc(size_grid*sizeof(float));
  ExtLabel        = (float *)malloc(size_grid*sizeof(float));
  VthetaRes       = (float *)malloc(size_grid*sizeof(float));
  TempShift       = (float *)malloc(size_grid*sizeof(float));
  dq              = (float *)malloc(size_grid*sizeof(float));
  InitTransportDevice();

}



__host__ void InitTransportDevice()
{
  gpuErrchk(hipMalloc((void**)&RadMomP_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&RadMomM_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ThetaMomP_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ThetaMomM_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Work_d,           size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&QRStar_d,         size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ExtLabel_d,       size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dq_d,             size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&LostByDisk_d,     size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaRes_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TempShift_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VMed_d,             NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Nshift_d,           NRAD*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&NoSplitAdvection_d, NRAD*sizeof(int)));
}
