#include "Main.cuh"

extern int NRAD, NSEC, size_grid, AdvecteLabel, OpenInner, Adiabatic, FastTransport;

extern double OmegaFrame;

extern float *Dens_d, *Vrad_d,  *Vtheta_d, *Label_d, *QStar_d, *Qbase_d, *Qbase2_d;
extern float *DensInt_d, *DensStar_d, *array_d;
extern float *DensStar, *QStar, *Qbase, *DensInt,  *Vazimutal_d;

extern float *invdiffRmed_d, *Rinf_d, *Rmed_d, *invRmed_d, *Rsup_d, *invSurf_d,  *Surf_d;

extern dim3 dimGrid2, dimBlock2, dimBlock, dimGrid4;

float *RadMomP, *RadMomM, *ThetaMomP, *ThetaMomM, *Work, *QRStar, *ExtLabel, *dq;
float *VthetaRes, *TempShift;

float *RadMomP_d, *RadMomM_d, *ThetaMomP_d, *ThetaMomM_d, *Work_d, *QRStar_d, *ExtLabel_d;
float *dq_d, *LostByDisk_d, *VMed_d, *VthetaRes_d, *TempShift_d;

float LostMass = 0.0;

static int UniformTransport;

int *NoSplitAdvection_d;
long *Nshift_d;

__host__ void Transport (float *Dens, float *Vrad, float *Vtheta, float *Energy, float *Label, float dt)
{

  ComputeLRMomenta(); // bien hasta aca

  if (AdvecteLabel == YES) ComputeExtQty();

  /* No-Alternate Directionnal Splitting */
  OneWindRad (Dens, Vrad, Energy, dt);
  OneWindTheta (Dens, Vtheta, Energy, dt);
  ComputeVelocities (Dens, Vrad, Vtheta);
  //if (AdvecteLabel) ComputeSpeQty (Dens, Label, ExtLabel);
}


__host__ void ComputeLRMomenta()
{
  LRMomentaKernel<<<dimGrid2, dimBlock2>>>(RadMomP_d, RadMomM_d, ThetaMomP_d, ThetaMomM_d, Dens_d, Vrad_d, Vtheta_d,
    NRAD, NSEC, Rmed_d, OmegaFrame);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ComputeExtQty()
{
  ExtQtyKernel<<<dimGrid2, dimBlock2>>>(ExtLabel_d, Dens_d, Label_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void OneWindRad (float *Dens, float *Vrad, float *Energy, float dt)
{

  ComputeStarRad(Dens_d, Vrad, DensStar_d, dt);

  ActualiseGasDens (DensInt, Dens);

  VanLeerRadial (Vrad, RadMomP_d, dt, 0);
  VanLeerRadial (Vrad, RadMomM_d, dt, 0);
  VanLeerRadial (Vrad, ThetaMomP_d, dt, 0);
  VanLeerRadial (Vrad, ThetaMomM_d, dt, 0);

  if (Adiabatic)
    VanLeerRadial (Vrad, Energy, dt, 0);

  if (AdvecteLabel == YES)
    VanLeerRadial (Vrad, ExtLabel_d, dt, 0);

  LostMass += VanLeerRadial (Vrad, Dens_d, dt, 1);

}



__host__ void ComputeStarRad(float *Qbase2_d, float *Vrad, float *QStar_d, float dt)
{
  StarRadKernel<<<dimGrid2, dimBlock2>>> (Qbase2_d, Vrad_d, QStar_d, dt, NRAD, NSEC, invdiffRmed_d, Rmed_d, dq_d);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ActualiseGasDens(float *DensInt, float *Dens)
{
  gpuErrchk(hipMemcpy(DensInt_d, Dens_d, size_grid*sizeof(float), hipMemcpyDeviceToDevice));
  gpuErrchk(hipDeviceSynchronize());
}



__host__ float VanLeerRadial (float *Vrad, float *Qbase_d, float dt, int ReturnLost)
{
  float Lost = 0.0;
  DivisePolarGrid (Qbase_d, DensInt_d, Work_d);
  ComputeStarRad (Work_d, Vrad, QRStar_d, dt);

  VanLeerRadialKernel<<<dimGrid2, dimBlock2>>>(Rinf_d, Rsup_d, QRStar_d, DensStar_d, Vrad_d,
    LostByDisk_d, NSEC, NRAD, dt, OpenInner, Qbase_d, invSurf_d);
  gpuErrchk(hipDeviceSynchronize());

  if (ReturnLost) Lost = DeviceReduce(LostByDisk_d, NSEC);

  return Lost;
}



__host__ void ComputeSpeQty (float *Dens, float *labe, float *ExtLabel)
{
  ComputeSpeQtyKernel<<<dimGrid2, dimBlock2>>>(Label_d, Dens_d, ExtLabel_d, NRAD, NSEC);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeVelocities(float *Dens, float *Vrad, float *Vtheta)
{
  ComputeVelocitiesKernel<<<dimGrid2, dimBlock2>>>(Vrad_d, Vtheta_d, Dens_d, Rmed_d, ThetaMomP_d,
    ThetaMomM_d, RadMomP_d, RadMomM_d,  NRAD,  NSEC, OmegaFrame);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void OneWindTheta (float *Dens, float *Vtheta, float *Energy, float dt)
{
  ComputeAverageThetaVelocities (Vtheta, dt);
  ComputeResiduals (Vtheta, dt);
  ComputeConstantResidual (Vtheta, dt); /* Constant residual is in Vtheta from now on */
  UniformTransport = NO;

  QuantitiesAdvection (Dens, VthetaRes_d, Energy, dt, 0);

  UniformTransport = YES;

  QuantitiesAdvection (Dens, Vtheta_d, Energy, dt, 1);

  AdvectSHIFT (RadMomP_d);
  AdvectSHIFT (RadMomM_d);
  AdvectSHIFT (ThetaMomP_d);
  AdvectSHIFT (ThetaMomM_d);
  if (Adiabatic) AdvectSHIFT (Energy);
  if (AdvecteLabel) AdvectSHIFT (ExtLabel_d);
  AdvectSHIFT (Dens_d);
}

__host__ void ComputeAverageThetaVelocities (float *Vtheta, float dt)
{
  ComputeAverageThetaVelocitiesKernel<<<dimGrid4, dimBlock>>>(Vtheta_d, VMed_d, NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void ComputeResiduals (float *Vtheta, float dt)
{
  ComputeResidualsKernel<<<dimGrid2, dimBlock2>>>(VthetaRes_d, VMed_d, NSEC, NRAD, Vtheta_d);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void AdvectSHIFT (float *array_d)
{
  AdvectSHIFTKernel<<<dimGrid2, dimBlock2>>> (array_d, TempShift_d, NSEC, NRAD, Nshift_d);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void ComputeConstantResidual (float *Vtheta, float dt)
{
  ComputeConstantResidualKernel<<<dimGrid2, dimBlock2>>>(VMed_d, invRmed_d, Nshift_d, NoSplitAdvection_d,
    NSEC, NRAD, dt, Vtheta_d, VthetaRes_d, Rmed_d, FastTransport);
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void QuantitiesAdvection (float *Dens, float *Vazimutal_d, float *Energy, float dt, int option)
{

  ComputeStarTheta (Dens_d, Vazimutal_d, DensStar_d, dt);
  ActualiseGasDens (DensInt, Dens);
  VanLeerTheta (Vazimutal_d, RadMomP_d, dt);
  VanLeerTheta (Vazimutal_d, RadMomM_d, dt);
  VanLeerTheta (Vazimutal_d, ThetaMomP_d, dt);
  VanLeerTheta (Vazimutal_d, ThetaMomM_d, dt);

  if (Adiabatic)
    VanLeerTheta (Vazimutal_d, Energy, dt);
  if (AdvecteLabel)
    VanLeerTheta (Vazimutal_d, ExtLabel_d, dt);
  VanLeerTheta (Vazimutal_d, Dens_d, dt); /* MUST be the last line */

}



__host__ void VanLeerTheta (float *Vazimutal_d, float *Qbase_d, float dt)
{
  DivisePolarGrid (Qbase_d, DensInt_d, Work_d);
  ComputeStarTheta (Work_d, Vazimutal_d, QRStar_d, dt);

  VanLeerThetaKernel<<<dimGrid2, dimBlock2>>>(Rsup_d, Rinf_d, Surf_d, dt, NRAD, NSEC, UniformTransport,
    NoSplitAdvection_d, QRStar_d, DensStar_d, Vazimutal_d, Qbase_d);
    gpuErrchk(hipDeviceSynchronize());
  }



__host__ void ComputeStarTheta (float *Qbase_d, float *Vazimutal_d, float *QStar_d, float dt)
{
    StarThetaKernel<<<dimGrid2, dimBlock2>>> (Qbase_d, Rmed_d, Vazimutal_d, QStar_d, NRAD, NSEC, dq_d, dt);
    gpuErrchk(hipDeviceSynchronize());
}



__host__ void InitTransport ()
{
  RadMomP         = (float *)malloc(size_grid*sizeof(float));
  RadMomM         = (float *)malloc(size_grid*sizeof(float));
  ThetaMomP       = (float *)malloc(size_grid*sizeof(float));
  ThetaMomM       = (float *)malloc(size_grid*sizeof(float));
  Work            = (float *)malloc(size_grid*sizeof(float));
  QRStar          = (float *)malloc(size_grid*sizeof(float));
  ExtLabel        = (float *)malloc(size_grid*sizeof(float));
  VthetaRes       = (float *)malloc(size_grid*sizeof(float));
  TempShift       = (float *)malloc(size_grid*sizeof(float));
  dq              = (float *)malloc(size_grid*sizeof(float));
  InitTransportDevice();

}



__host__ void InitTransportDevice()
{
  gpuErrchk(hipMalloc((void**)&RadMomP_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&RadMomM_d,        size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ThetaMomP_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ThetaMomM_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Work_d,           size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&QRStar_d,         size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&ExtLabel_d,       size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dq_d,             size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&LostByDisk_d,     NSEC*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VthetaRes_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TempShift_d,      size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&VMed_d,             NRAD*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&Nshift_d,           NRAD*sizeof(long)));
  gpuErrchk(hipMalloc((void**)&NoSplitAdvection_d, NRAD*sizeof(int)));


  gpuErrchk(hipMemset(RadMomP_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(RadMomM_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(ThetaMomP_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(ThetaMomM_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(Work_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(QRStar_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(ExtLabel_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(dq_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(LostByDisk_d, 0, NSEC*sizeof(float)));
  gpuErrchk(hipMemset(TempShift_d, 0, size_grid*sizeof(float)));
  gpuErrchk(hipMemset(VMed_d, 0, NRAD*sizeof(float)));
  gpuErrchk(hipMemset(Nshift_d, 0, NRAD*sizeof(long)));
  gpuErrchk(hipMemset(NoSplitAdvection_d, 0, NRAD*sizeof(int)));

}
