#include "hip/hip_runtime.h"
#include "Main.cuh"

extern dim3 dimGrid2, dimBlock2, dimGrid3;

extern hipfftDoubleComplex *SGP_Kt_dc, *SGP_Kr_dc, *SGP_St_dc, *SGP_Sr_dc, *Gr_dc, *Gphi_dc;
extern hipfftDoubleComplex *Gr_d, *Gphi_d, *SGP_Kt_d, *SGP_Kr_d, *SGP_Sr_d, *SGP_St_d;

extern int NSEC, NRAD, size_grid;

extern float *Vrad_d, *Vtheta_d, *Dens_d, *Vradial_d, *Vazimutal_d, *VthetaInt_d, *VradInt_d;


extern float *Kr_aux_d, *Kt_aux_d, *SG_Acct_d, *SG_Accr_d;
extern float *SG_Accr, *GLOBAL_AxiSGAccr, *axifield_d;
extern float *GLOBAL_AxiSGAccr, *Radii;
extern float *invdiffRmed_d, *Rinf_d, *Rmed, *Rmed_d,  *Radii_d;
extern float SGP_eps, SGP_rstep, SGP_tstep, ECCENTRICITY;

extern hipfftHandle planf, planb;

float *Kr_aux, *Kt_aux;

__host__ void compute_selfgravity (float *Dens, float DeltaT, int SGUpdate, int initialization)
{
  /* We compute Kernel */
  if (initialization)
    compute_kernel();
  /* We compute Density */
  compute_FFT ();
  /* Execute plan2d forward Complex-to-Complex */
  ExecuteExeC2Cforward();
  /* Convolution Kr*Sr and Kt*St in Fourier space */
  compute_Convolution();
  /* Execute plan2d backward Complex-to-Complex */
  ExecuteExeC2Cbackward();
  /* Here we compute radial and azimutal components of sg acceleration
     as a convolution product of reduced density and kernel arrays */
  compute_sgacc();

  if (SGUpdate) {
    /* Computes polar components of acceleration and
      updates values of vrad, vtheta at each step */
      update_sgvelocity(DeltaT);
  }
}



__host__ void compute_kernel ()
{
  float u, theta, Kr, Kt, base;
  int i,j;
  /* Si se elige la opcion SelfGravity se crean los arreglos para calcular
     los Kernels Kr, Kt */
  Kr_aux      = (float *)malloc(2*size_grid*sizeof(float));
  Kt_aux      = (float *)malloc(2*size_grid*sizeof(float));

  /* Aca calculo los kernels Kr y Kt en CPU ya que son constantes */
  for (i = 0; i < 2*NRAD; i++){
    if(i < NRAD)
      u = logf(Radii[i]/Radii[0]);
    else
      u = -logf(Radii[2*NRAD-i]/Radii[0]);

    for (j = 0; j < NSEC; j++){
      theta = 2.0*PI*(float)j  / (float)NSEC;
      Kr = 1.0 + SGP_eps*SGP_eps - cosf(theta)*expf(-u);
      base = SGP_eps*SGP_eps*expf(u) + 2.0*(coshf(u) - cosf(theta)) ;
      Kr *= powf(base , -1.5);

      Kt = sinf(theta);
      Kt *= powf(base, -1.5);

      Kr_aux[i*NSEC+j] = (float) Kr;
      Kt_aux[i*NSEC+j] = (float) Kt;
    }
  }

  gpuErrchk(hipMemcpy(Kr_aux_d, Kr_aux, 2*size_grid*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(Kt_aux_d, Kt_aux, 2*size_grid*sizeof(float), hipMemcpyHostToDevice));
}


__host__ void compute_FFT ()
{
  ComputeFFTKernel<<<dimGrid3, dimBlock2>>>(Radii_d, SGP_Kr_d, SGP_Kt_d, SGP_eps, NRAD, NSEC, SGP_Sr_d,
    SGP_St_d, Dens_d, Rmed_d, Kr_aux_d, Kt_aux_d);
    gpuErrchk(hipDeviceSynchronize());
}



__host__ void compute_Convolution ()
{
  ComputeConvolutionKernel<<<dimGrid3, dimBlock2>>>(Gr_dc, Gphi_dc, SGP_Kr_dc, SGP_Kt_dc, SGP_Sr_dc, SGP_St_dc,
    NSEC, NRAD);
  gpuErrchk(hipDeviceSynchronize());
}


__host__ void compute_sgacc ()
{
  ComputeSgAccKernel<<<dimGrid2, dimBlock2>>>(SG_Accr_d, SG_Acct_d, Dens_d, SGP_rstep, SGP_tstep, SGP_eps,
    NRAD, NSEC, Rmed_d, Gr_d, Gphi_d);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void update_sgvelocity (float DeltaT)
{
  Update_sgvelocityKernel <<<dimGrid2, dimBlock2>>>(Vradial_d, Vazimutal_d, SG_Accr_d, SG_Acct_d, Rinf_d, Rmed_d,
    invdiffRmed_d, DeltaT , NRAD,  NSEC);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void ExecuteExeC2Cforward ()
{
  if ((hipfftExecZ2Z(planf, (hipfftDoubleComplex *)SGP_St_d, (hipfftDoubleComplex *)SGP_St_dc, HIPFFT_FORWARD))!= \
    HIPFFT_SUCCESS){
      printf("cufft execC2C error\n");
      exit(-1);
  }
  gpuErrchk(hipDeviceSynchronize());

  if ((hipfftExecZ2Z(planf,(hipfftDoubleComplex *)SGP_Kt_d, (hipfftDoubleComplex *)SGP_Kt_dc, HIPFFT_FORWARD))!= \
    HIPFFT_SUCCESS){
      printf("cufft execC2C error\n");
      exit(-1);
  }
  gpuErrchk(hipDeviceSynchronize());

  if ((hipfftExecZ2Z(planf, (hipfftDoubleComplex *)SGP_Sr_d, (hipfftDoubleComplex *)SGP_Sr_dc, HIPFFT_FORWARD))!= \
    HIPFFT_SUCCESS){
      printf("cufft execC2C error\n");
      exit(-1);
  }
  gpuErrchk(hipDeviceSynchronize());

  if ((hipfftExecZ2Z(planf, (hipfftDoubleComplex *)SGP_Kr_d, (hipfftDoubleComplex *)SGP_Kr_dc, HIPFFT_FORWARD))!= \
    HIPFFT_SUCCESS){
      printf("cufft execC2C error\n");
      exit(-1);
  }
  gpuErrchk(hipDeviceSynchronize());

}

__host__ void ExecuteExeC2Cbackward ()
{
  if ((hipfftExecZ2Z(planb, (hipfftDoubleComplex *)Gphi_dc, (hipfftDoubleComplex *)Gphi_d, HIPFFT_BACKWARD))!= \
    HIPFFT_SUCCESS){
      printf("cufft execC2C error\n");
      exit(-1);
  }
  gpuErrchk(hipDeviceSynchronize());

  if ((hipfftExecZ2Z(planb, (hipfftDoubleComplex *)Gr_dc, (hipfftDoubleComplex *)Gr_d, HIPFFT_BACKWARD))!= \
    HIPFFT_SUCCESS){
      printf("cufft execC2C error\n");
      exit(-1);
  }
  gpuErrchk(hipDeviceSynchronize());
}



__host__ void Init_planetarysys_withSG (PlanetarySystem *sys)
{
  /*  !SGZeroMode case */
  Make1Dprofile (1);
  gpuErrchk(hipMemcpy(GLOBAL_AxiSGAccr, axifield_d, NRAD*sizeof(float), hipMemcpyDeviceToHost));

  float r, dist, ri, rip1, dr, sgacc;
  int ipl, k;

  /* Planetary system initialization in self-gravity cases:
     planets are put in a fixed circular orbit, we need to know radial sg acceleration
     felt by planets.*/

  for (k = 0; k < sys->nb; k++){
    r = sys->x[k];
    /* dist denotes the planet's semi-major axis */
    dist = (float) (r / (1. + ECCENTRICITY));
    ipl = 0;
    while (Rmed[ipl] <= dist) ipl++;
    ri = Rmed[ipl];
    rip1 = Rmed[ipl+1];
    dr = rip1 - ri;
    sgacc = (dist - ri)*GLOBAL_AxiSGAccr[ipl+1] + (rip1 - dist)*GLOBAL_AxiSGAccr[ipl];
    sgacc /= dr;

    /* sgacc is the radial sg acc. at the planet's semi-major axis */
    sys->vy[k] *= (float) sqrtf(1. - dist*dist*sgacc);
  }
}
