#include "hip/hip_runtime.h"
#include "Main.cuh"

extern int NRAD, NSEC, size_grid;

extern float *Label_d;
extern float *Rmed,  *Rmed_d;

extern dim3 dimGrid2, dimBlock2;

__host__ void Initialization (float *Dens, float *Vrad, float *Vtheta, float *Energy, float *Label, PlanetarySystem *sys)
{
  InitEuler (Vrad, Vtheta, Dens, Energy);
  InitLabel (Label, sys);
  WriteDim();
}

__host__ void InitLabel (float *Label, PlanetarySystem *sys)
{
  float xp,yp, rhill, rp;
  xp = sys->x[0];
  yp = sys->y[0];
  rp = sqrt(xp*xp+yp*yp);
  rhill = rp * pow(sys->mass[0]/3., 1./3);

  InitLabelKernel<<<dimGrid2, dimBlock2>>>(Label_d, xp, yp, rhill, Rmed_d, NRAD, NSEC);
  gpuErrchk(hipDeviceSynchronize());
}
