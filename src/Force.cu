#include "hip/hip_runtime.h"
#include "Main.cuh"

extern string OUTPUTDIR;

extern float ROCHESMOOTHING, THICKNESSSMOOTHING, FLARINGINDEX;

extern float *CellAbscissa, *CellOrdinate, *forcesxi, *forcesyi, *forcesxo, *forcesyo;

extern float *CellAbscissa_d, *CellOrdinate_d;
extern float *fxi_d, *fxo_d, *fyi_d, *fyo_d;
extern float *Rmed, *Rmed_d, *Surf,  *Surf_d, *example;

extern float *Dens_d;

extern int RocheSmoothing, size_grid, NRAD, NSEC, SelfGravity;

extern dim3 dimGrid2, dimBlock2;

__host__ void UpdateLog (Force *force, PlanetarySystem *sys, float *Dens, float *Energy, int TimeStep,
  float PhysicalTime, int dimfxy)
{
  FILE *out;
  float x, y, r, m, vx, vy, smoothing, a, rh;
  float *globalforce;
  char filename[500];
  char filename2[500];
  int i, nb;
  nb=sys->nb;
  string input;
  input = OUTPUTDIR +"tqwk";
  strncpy(filename, input.c_str(), sizeof(filename));
  filename[sizeof(filename)-1]=0;
  for (i = 0; i < nb; i++){
    x = sys->x[i];
    y = sys->y[i];
    vx = sys->vx[i];
    vy = sys->vy[i];
    r = sqrtf(x*x+y*y);
    m = sys->mass[i];
    a = sqrtf(x*x+y*y);
    rh = powf(m/3., 1./3.)*a+1e-15;

    if (RocheSmoothing) smoothing = r*powf(m/3.,1./3.)*ROCHESMOOTHING;
    else smoothing = Compute_smoothing(r);

    ComputeForce (force, Dens, x, y, smoothing, m, dimfxy, a, rh);

    globalforce = force->GlobalForce;
    sprintf (filename2, "%s%d.dat", filename,i);
    out = fopen(filename2, "a");
    if (out == NULL){
      fprintf(stderr, "Can't open %s\n",filename2 );
      fprintf(stderr, "Aborted.\n");
    }

    fprintf(out, "%d\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\n", TimeStep, \
      x*force->fy_inner-y*force->fx_inner, \
      x*force->fy_outer-y*force->fx_outer, \
      x*force->fy_ex_inner-y*force->fx_ex_inner, \
      x*force->fy_ex_outer-y*force->fx_ex_outer, \
      vx*force->fx_inner+vy*force->fy_inner , \
      vx*force->fx_outer+vy*force->fy_outer , \
      vx*force->fx_ex_inner+vy*force->fy_ex_inner , \
      vx*force->fx_ex_outer+vy*force->fy_ex_outer , PhysicalTime);
    fclose (out);

    if (!SelfGravity){
      for (int k = 0; k < dimfxy; k++) {
        sprintf( filename2, "%s%d_%d.dat", filename, i, k);
        out = fopen(filename2, "a");
        if (out == NULL){
          fprintf(stderr, "Can't open %s\n", filename2);
          fprintf(stderr, "Aborted.\n");
        }
        fprintf(out, "%d\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\n", TimeStep, \
          x*globalforce[2*dimfxy+k]-y*globalforce[k], \
          x*globalforce[3*dimfxy+k]-y*globalforce[dimfxy+k], \
          vx*globalforce[k]+vy*globalforce[2*dimfxy+k], \
          vx*globalforce[dimfxy+k]+vy*globalforce[3*dimfxy+k], PhysicalTime);
        fclose (out);
      }
    }
  }
}



__host__ Force *AllocateForce (int dimfxy)
{
  Force *force;
  float *globalforce;
  force = (Force *)malloc(sizeof(Force));
  globalforce = (float *)malloc(sizeof(float)*4*dimfxy);
  for (int i = 0; i < 4*dimfxy; i++)
    globalforce[i] = 0.0;
  force->GlobalForce = globalforce;
  return force;
}



__host__ void ComputeForce (Force *force, float *Dens, float x, float y, float rsmoothing,
  float mass, int dimfxy, float a, float rh)
{
  float *globalforce;
  int k;

  globalforce = force->GlobalForce;
  float valor;

  for (k = 0; k < dimfxy; k++) {
    gpuErrchk(hipMemset(fxi_d, 0, NRAD*NSEC*sizeof(float)));
    gpuErrchk(hipMemset(fxo_d, 0, NRAD*NSEC*sizeof(float)));
    gpuErrchk(hipMemset(fyi_d, 0, NRAD*NSEC*sizeof(float)));
    gpuErrchk(hipMemset(fyo_d, 0, NRAD*NSEC*sizeof(float)));

    ComputeForceKernel<<<dimGrid2, dimBlock2>>>(CellAbscissa_d, CellOrdinate_d, Surf_d, Dens_d, x, y, rsmoothing,
      NSEC, NRAD, a, Rmed_d, dimfxy, rh, fxi_d, fxo_d, fyi_d, fyo_d, k);
    gpuErrchk(hipDeviceSynchronize());

    globalforce[k]            = DeviceReduce(fxi_d, NRAD*NSEC);
    globalforce[k + dimfxy]   = DeviceReduce(fxo_d, NRAD*NSEC);
    globalforce[k + 2*dimfxy] = DeviceReduce(fyi_d, NRAD*NSEC);
    globalforce[k + 3*dimfxy] = DeviceReduce(fyo_d, NRAD*NSEC);
  }

  force->fx_inner = globalforce[0];
  force->fx_ex_inner = globalforce[dimfxy-1];
  force->fx_outer = globalforce[dimfxy];
  force->fx_ex_outer = globalforce[2*dimfxy-1];
  force->fy_inner = globalforce[2*dimfxy];
  force->fy_ex_inner = globalforce[3*dimfxy-1];
  force->fy_outer = globalforce[3*dimfxy];
  force->fy_ex_outer = globalforce[4*dimfxy-1];
  force->GlobalForce = globalforce;
}

__host__ float Compute_smoothing(float r)
{
  float smooth;
  smooth = THICKNESSSMOOTHING * AspectRatioHost(r) * powf(r, 1.0+FLARINGINDEX);
  return smooth;
}

__host__ void FreeForce (Force *force)
{
  free (force->GlobalForce);
}
