#include "hip/hip_runtime.h"
#include "Main.cuh"

extern string OUTPUTDIR;

extern double ROCHESMOOTHING, THICKNESSSMOOTHING, FLARINGINDEX;

extern double *CellAbscissa, *CellOrdinate, *Surf, *forcesxi, *forcesyi, *forcesxo;
extern double *forcesyo;

extern double *Dens_d, *CellAbscissa_d, *CellOrdinate_d, *Surf_d;
extern double *fxi_d, *fxo_d, *fyi_d, *fyo_d;

extern double *Rmed, *Rmed_d;

extern int RocheSmoothing, size_grid, NRAD, NSEC, SelfGravity;

extern dim3 dimGrid2, dimBlock2;

__host__ void UpdateLog (Force *force, PlanetarySystem *sys, double *Dens, double *Energy, int TimeStep,
  double PhysicalTime, int dimfxy)
{
  FILE *out;
  double x, y, r, m, vx, vy, smoothing, a, rh;
  double *globalforce;
  char filename[500];
  char filename2[500];
  int i, nb;
  nb=sys->nb;
  string input;
  input = OUTPUTDIR +"tqwk";
  strncpy(filename, input.c_str(), sizeof(filename));
  filename[sizeof(filename)-1]=0;
  for (i = 0; i < nb; i++){
    x = sys->x[i];
    y = sys->y[i];
    vx = sys->vx[i];
    vy = sys->vy[i];
    r = sqrt(x*x+y*y);
    m = sys->mass[i];
    a = sqrt(x*x+y*y);
    rh = pow(m/3., 1./3.)*a+1e-15;

    if (RocheSmoothing) smoothing = r*pow(m/3.,1./3.)*ROCHESMOOTHING;
    else smoothing = Compute_smoothing(r);

    ComputeForce (force, Dens, x, y, smoothing, m, dimfxy, a, rh);

    globalforce = force->GlobalForce;
    sprintf (filename2, "%s%d.dat", filename,i);
    out = fopen(filename2, "a");
    if (out == NULL){
      fprintf(stderr, "Can't open %s\n",filename2 );
      fprintf(stderr, "Aborted.\n");
    }

    fprintf(out, "%d\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\n", TimeStep, \
      x*force->fy_inner-y*force->fx_inner, \
      x*force->fy_outer-y*force->fx_outer, \
      x*force->fy_ex_inner-y*force->fx_ex_inner, \
      x*force->fy_ex_outer-y*force->fx_ex_outer, \
      vx*force->fx_inner+vy*force->fy_inner , \
      vx*force->fx_outer+vy*force->fy_outer , \
      vx*force->fx_ex_inner+vy*force->fy_ex_inner , \
      vx*force->fx_ex_outer+vy*force->fy_ex_outer , PhysicalTime);
    fclose (out);

    if (!SelfGravity){
      for (int k = 0; k < dimfxy; k++) {
        sprintf( filename2, "%s%d_%d.dat", filename, i, k);
        out = fopen(filename2, "a");
        if (out == NULL){
          fprintf(stderr, "Can't open %s\n", filename2);
          fprintf(stderr, "Aborted.\n");
        }
        fprintf(out, "%d\t%.18g\t%.18g\t%.18g\t%.18g\t%.18g\n", TimeStep, \
          x*globalforce[2*dimfxy+k]-y*globalforce[k], \
          x*globalforce[3*dimfxy+k]-y*globalforce[dimfxy+k], \
          vx*globalforce[k]+vy*globalforce[2*dimfxy+k], \
          vx*globalforce[dimfxy+k]+vy*globalforce[3*dimfxy+k], PhysicalTime);
        fclose (out);
      }
    }
  }
}



__host__ Force *AllocateForce (int dimfxy)
{
  Force *force;
  double *globalforce;
  force = (Force *)malloc(sizeof(Force));
  globalforce = (double *)malloc(sizeof(double)*4*dimfxy);
  for (int i = 0; i < 4*dimfxy; i++)
    globalforce[i] = 0.0;
  force->GlobalForce = globalforce;
  return force;
}



__host__ void ComputeForce (Force *force, double *Dens, double x, double y, double rsmoothing,
  double mass, int dimfxy, double a, double rh)
{
  double *globalforce;
  int k;

  globalforce = force->GlobalForce;

  gpuErrchk(hipMemset(fxi_d, 0, NRAD*NSEC*sizeof(double)));
  gpuErrchk(hipMemset(fxo_d, 0, NRAD*NSEC*sizeof(double)));
  gpuErrchk(hipMemset(fyi_d, 0, NRAD*NSEC*sizeof(double)));
  gpuErrchk(hipMemset(fyo_d, 0, NRAD*NSEC*sizeof(double)));

  for (k = 0; k < dimfxy; k++) {
    ComputeForceKernel<<<dimGrid2, dimBlock2>>>(CellAbscissa_d, CellOrdinate_d, Surf_d, Dens_d, x, y, rsmoothing,
      NSEC, NRAD, a, Rmed_d, dimfxy, rh, fxi_d, fxo_d, fyi_d, fyo_d, k);
    gpuErrchk(hipDeviceSynchronize());

    globalforce[k]            = DeviceReduce(fxi_d, NRAD*NSEC);
    globalforce[k + dimfxy]   = DeviceReduce(fxo_d, NRAD*NSEC);
    globalforce[k + 2*dimfxy] = DeviceReduce(fyi_d, NRAD*NSEC);
    globalforce[k + 3*dimfxy] = DeviceReduce(fyo_d, NRAD*NSEC);
  }

  force->fx_inner = globalforce[0];
  force->fx_ex_inner = globalforce[dimfxy-1];
  force->fx_outer = globalforce[dimfxy];
  force->fx_ex_outer = globalforce[2*dimfxy-1];
  force->fy_inner = globalforce[2*dimfxy];
  force->fy_ex_inner = globalforce[3*dimfxy-1];
  force->fy_outer = globalforce[3*dimfxy];
  force->fy_ex_outer = globalforce[4*dimfxy-1];
  force->GlobalForce = globalforce;
}

__host__ double Compute_smoothing(double r)
{
  double smooth;
  smooth = THICKNESSSMOOTHING * AspectRatioHost(r) * pow(r, 1.0+FLARINGINDEX);
  return smooth;
}

__host__ void FreeForce (Force *force)
{
  free (force->GlobalForce);
}
