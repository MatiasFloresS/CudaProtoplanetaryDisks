#include "hip/hip_runtime.h"
#include "Main.cuh"

static double k1[100], k2[100], k3[100], k4[100], k5[100], k6[100];
static double Dist[100];

extern int Indirect_Term;

__host__ void RungeKutta (double *q0, float timestep, double *PlanetMasses, double *q1, int nb, int *feelothers)
{
  for (int i = 0; i < 4*nb; i++){
    k1[i] = k2[i] = k3[i] = k4[i] = k5[i] = k6[i] = 0;
  }

  DerivMotionRK5 (q0, PlanetMasses, k1, nb, timestep, feelothers);
  TranslatePlanetRK5 (q0, 0.2, 0.0, 0.0, 0.0, 0.0, q1, nb);
  DerivMotionRK5 (q1, PlanetMasses, k2, nb, timestep, feelothers);
  TranslatePlanetRK5 (q0, 0.075, 0.225, 0.0, 0.0, 0.0, q1, nb);
  DerivMotionRK5 (q1, PlanetMasses, k3, nb, timestep, feelothers);
  TranslatePlanetRK5 (q0, 0.3, -0.9, 1.2, 0.0, 0.0, q1, nb);
  DerivMotionRK5 (q1, PlanetMasses, k4, nb, timestep, feelothers);
  TranslatePlanetRK5 (q0, -11.0/54.0, 2.5, -70.0/27.0, 35.0/27.0, 0.0, q1, nb);
  DerivMotionRK5 (q1, PlanetMasses, k5, nb, timestep, feelothers);
  TranslatePlanetRK5 (q0, 1631.0/55296.0, 175.0/512.0, 575.0/13824.0, 44275.0/110592.0, 253.0/4096.0, q1, nb);
  DerivMotionRK5 (q1, PlanetMasses, k6, nb, timestep, feelothers);
  for (int i = 0; i < 4*nb; i++){
    q1[i] = q0[i]+37.0/378.0*k1[i]+250.0/621.0*k3[i]+125.0/594.0*k4[i]+512.0/1771.0*k6[i];
  }
}

__host__ void DerivMotionRK5 (double *q_init, double *PlanetMasses, double *deriv, int nb, float dt, int *feelothers)
{
  double *x, *y, *vx, *vy, dist;
  double *derivx, *derivy, *derivvx, *derivvy;
  int i,j;
  x = q_init;
  y = x+nb;
  vx = y+nb;
  vy = vx+nb;
  derivx = deriv;
  derivy = derivx+nb;
  derivvx = derivy+nb;
  derivvy = derivvx+nb;

  for (i = 0; i < nb; i++)
    Dist[i] = sqrt(x[i]*x[i]+y[i]*y[i]);

  for (i = 0; i < nb; i++){
    derivx[i] = vx[i];
    derivy[i] = vy[i];
    derivvx[i] = -G*1.0/Dist[i]/Dist[i]/Dist[i]*x[i];
    derivvy[i] = -G*1.0/Dist[i]/Dist[i]/Dist[i]*y[i];
    // printf("derivx %g\n", derivx[i]);
    // printf("derivy %g\n", derivy[i]);
    // printf("derivvx %g\n", derivvx[i]);
    // printf("derivvy %g\n", derivvy[i]);
    for (j = 0; j < nb; j++){
      if (Indirect_Term){
        derivvx[i] -= G*PlanetMasses[j]/Dist[j]/Dist[j]/Dist[j]*x[j];
        derivvy[i] -= G*PlanetMasses[j]/Dist[j]/Dist[j]/Dist[j]*y[j];
      }
      if ((j != i) && (feelothers[i] == YES)){
        dist = (x[i]-x[j])*(x[i]-x[j])+(y[i]-y[j])*(y[i]-y[j]);
        dist = sqrt(dist);
        derivvx[i] += G*PlanetMasses[j]/Dist[j]/Dist[j]/Dist[j]*(x[j]-x[i]);
        derivvy[i] += G*PlanetMasses[j]/Dist[j]/Dist[j]/Dist[j]*(y[j]-y[i]);
      }
    }
  }
  for (i = 0; i < 4*nb; i++)
    deriv[i] *= dt;
}

__host__ void TranslatePlanetRK5 (double *qold, double c1, double c2, double c3, double c4, double c5, double *qnew, int nb)
{
  int i;
  for (i = 0; i < 4*nb; i++){
    qnew[i] = qold[i]+c1*k1[i]+c2*k2[i]+c3*k3[i]+c4*k4[i]+c5*k5[i];
  }
}
