#include "Main.cuh"

extern float *Dens_d;
extern float ScalingFactor;
extern int NRAD, NSEC, size_grid;

extern dim3 dimGrid2, dimBlock2;

__host__ void MultiplyPolarGridbyConstant (float *Dens)
{
  MultiplyPolarGridbyConstantKernel<<<dimGrid2, dimBlock2>>>(Dens_d, NRAD, NSEC, ScalingFactor);
  gpuErrchk(hipDeviceSynchronize());
}
