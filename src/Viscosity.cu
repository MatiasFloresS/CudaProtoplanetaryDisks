#include "hip/hip_runtime.h"
#include "Main.cuh"

extern int NSEC, size_grid, NRAD;

extern float TRANSITIONWIDTH, TRANSITIONRADIUS, TRANSITIONRATIO, ASPECTRATIO, LAMBDADOUBLING, *SoundSpeed_d, \
*SoundSpeed, VISCOSITY, ViscosityAlpha, *Rmed, CAVITYRATIO, CAVITYRADIUS, CAVITYWIDTH, *GLOBAL_bufarray,     \
ALPHAVISCOSITY, *Vrad_d, *Vtheta_d, *invdiffRsup_d, *Rinf_d, *invdiffRmed_d, *Dens_d, *viscosity_array_d,    \
*invRinf_d, *Rsup, *invRmed, *VthetaInt_d, *VradInt_d, *viscosity_array, *Rsup_d, *invRmed_d, *VradInt,  \
*VthetaInt, *Rmed_d, *invdiffRsup_d;

float PhysicalTime =0.0, PhysicalTimeInitial= 0.0, *DivergenceVelocity, *DRP, *DRR, *DPP, *TAURR, *TAURP,    \
*TAUPP, *DivergenceVelocity_d, *DRP_d, *DRR_d, *DPP_d, *TAURR_d, *TAURP_d, *TAUPP_d;

extern dim3 dimGrid2, dimBlock2;

__host__ void UpdateVelocitiesWithViscosity(float *VradInt, float *VthetaInt, float *Dens, float DeltaT)
{

  UpdateVelocitiesKernel<<<dimGrid2, dimBlock2>>>(VthetaInt_d, VradInt_d, invRmed_d, Rmed_d, Rsup_d, Rinf_d,
    invdiffRmed_d, invdiffRsup_d,  Dens_d, invRinf_d, TAURR_d, TAURP_d, TAUPP_d, DeltaT, NRAD, NSEC);
    gpuErrchk(hipDeviceSynchronize());

}

__host__ float AspectRatio(float r)
{
  float aspectratio, rmin, rmax, scale;
  aspectratio = ASPECTRATIO;
  rmin = TRANSITIONRADIUS-TRANSITIONWIDTH*ASPECTRATIO;
  rmax = TRANSITIONRADIUS+TRANSITIONWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) aspectratio *= TRANSITIONRATIO;
  if ((r >= rmin) && (r <= rmax))
  {
    aspectratio *= exp((rmax-r)/(rmax-rmin)*log(TRANSITIONRATIO));
  }
  return aspectratio;
}

__host__ float FViscosity(float r)
{
  float viscosity, rmin, rmax, scale;
  int i = 0;
  viscosity = VISCOSITY;
  if (ViscosityAlpha)
  {
    while (Rmed[i] < r) i++;
    viscosity = ALPHAVISCOSITY*GLOBAL_bufarray[i] * GLOBAL_bufarray[i] * pow(r, 1.5);
  }
  rmin = CAVITYRADIUS-CAVITYWIDTH*ASPECTRATIO;
  rmax = CAVITYRADIUS+CAVITYWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) viscosity *= CAVITYRATIO;
  if ((r >= rmin) && (r <= rmax)) viscosity *= exp((rmax-r)/(rmax-rmin)*log(CAVITYRATIO));
  return viscosity;
}

__host__ void ComputeViscousTerms (float *Vrad, float *Vtheta, float *Dens, int option)
{

  if (ViscosityAlpha)
  {
    gpuErrchk(hipMemcpy(SoundSpeed, SoundSpeed_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
    Make1Dprofile (1);
  }

  for (int i = 0; i < NRAD; i++) viscosity_array[i] = FViscosity(Rmed[i]);
  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  if (option == 1)
  {

    ViscousTermsKernel<<<dimGrid2, dimBlock2>>>(Vrad_d, Vtheta_d, DRR_d, DPP_d, DivergenceVelocity_d, DRP_d, invdiffRsup_d,
      invRmed_d, Rsup_d, Rinf_d, invdiffRmed_d, NRAD, NSEC, TAURR_d, TAUPP_d, Dens_d, viscosity_array_d,
      TAURP_d, invRinf_d);
  }
  else
  {
    ViscousTermsKernel<<<dimGrid2, dimBlock2>>>(VradInt_d, VthetaInt_d, DRR_d, DPP_d, DivergenceVelocity_d, DRP_d, invdiffRsup_d,
      invRmed_d, Rsup_d, Rinf_d, invdiffRmed_d, NRAD, NSEC, TAURR_d, TAUPP_d, Dens_d, viscosity_array_d,
      TAURP_d, invRinf_d);
  }

  gpuErrchk(hipDeviceSynchronize());
}

__host__ void InitViscosity ()
{
  DivergenceVelocity  = (float *)malloc(size_grid*sizeof(float));
  DRR                 = (float *)malloc(size_grid*sizeof(float));
  DRP                 = (float *)malloc(size_grid*sizeof(float));
  DPP                 = (float *)malloc(size_grid*sizeof(float));
  TAURR               = (float *)malloc(size_grid*sizeof(float));
  TAURP               = (float *)malloc(size_grid*sizeof(float));
  TAUPP               = (float *)malloc(size_grid*sizeof(float));
  InitViscosityDevice ();
}

__host__ void InitViscosityDevice ()
{
  gpuErrchk(hipMalloc((void**)&DivergenceVelocity_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DRR_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DRP_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DPP_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAURR_d,              size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAURP_d,              size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAUPP_d,              size_grid*sizeof(float)));
}
