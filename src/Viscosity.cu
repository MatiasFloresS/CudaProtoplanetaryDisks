#include "hip/hip_runtime.h"
#include "Main.cuh"

extern int NSEC, size_grid, nsec2pot, nrad2pot, blocksize, NRAD;

extern float TRANSITIONWIDTH, TRANSITIONRADIUS, TRANSITIONRATIO, ASPECTRATIO, LAMBDADOUBLING, *SoundSpeed_d, \
*SoundSpeed, VISCOSITY, ViscosityAlpha, *Rmed, CAVITYRATIO, CAVITYRADIUS, CAVITYWIDTH, *GLOBAL_bufarray, \
ALPHAVISCOSITY, *vrad_d, *vtheta_d, *Drr_d, *Dpp_d, *divergence_d, *Drp_d, *invdiffRsup_d, *Rinf_d, *Dpp, \
*invdiffRmed_d, *Trr_d, *Tpp_d, *dens_d, *viscosity_array_d, *Trp_d, *divergence, *Drr, *Drp, *Trr, *Trp, \
*Tpp, *invRinf_d, *Rsup, *invRmed, *vthetaint_d, *vradint_d, *viscosity_array, *Rsup_d, *invRmed_d, \
dphi, invdphi, onethird, *vradint, *vthetaint, *Rmed_d, *invdiffRsup_d;

float PhysicalTime =0.0, PhysicalTimeInitial= 0.0;

extern dim3 dimGrid2, dimBlock2;

__host__ void UpdateVelocitiesWithViscosity(float *vradint, float *vthetaint, float *dens, float DeltaT)
{

  UpdateVelocitiesKernel<<<dimGrid2, dimBlock2>>>(vthetaint_d, vradint_d, invRmed_d, Rmed_d, Rsup_d, Rinf_d,
    invdiffRmed_d, invdiffRsup_d,  dens_d, invRinf_d, Trr_d, Trp_d, Tpp_d, DeltaT, NRAD, NSEC);
    gpuErrchk(hipDeviceSynchronize());

}

__host__ float AspectRatio(float r)
{
  float aspectratio, rmin, rmax, scale;
  aspectratio = ASPECTRATIO;
  rmin = TRANSITIONRADIUS-TRANSITIONWIDTH*ASPECTRATIO;
  rmax = TRANSITIONRADIUS+TRANSITIONWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) aspectratio *= TRANSITIONRATIO;
  if ((r >= rmin) && (r <= rmax))
  {
    aspectratio *= exp((rmax-r)/(rmax-rmin)*log(TRANSITIONRATIO));
  }
  return aspectratio;
}

__host__ float FViscosity(float r)
{
  float viscosity, rmin, rmax, scale;
  int i = 0;
  viscosity = VISCOSITY;
  if (ViscosityAlpha)
  {
    while (Rmed[i] < r) i++;
    viscosity = ALPHAVISCOSITY*GLOBAL_bufarray[i] * GLOBAL_bufarray[i] * pow(r, 1.5);
  }
  rmin = CAVITYRADIUS-CAVITYWIDTH*ASPECTRATIO;
  rmax = CAVITYRADIUS+CAVITYWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) viscosity *= CAVITYRATIO;
  if ((r >= rmin) && (r <= rmax)) viscosity *= exp((rmax-r)/(rmax-rmin)*log(CAVITYRATIO));
  return viscosity;
}

__host__ void ComputeViscousTerms (float *vradial, float *vazimutal, float *dens, int option)
{

  if (ViscosityAlpha)
  {
    gpuErrchk(hipMemcpy(SoundSpeed, SoundSpeed_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
    Make1Dprofile (SoundSpeed);
  }

  for (int i = 0; i < NRAD; i++) viscosity_array[i] = FViscosity(Rmed[i]);
  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  if (option == 1)
  {

    ViscousTermsKernel<<<dimGrid2, dimBlock2>>>(vrad_d, vtheta_d, Drr_d, Dpp_d, divergence_d, Drp_d, invdiffRsup_d,
      invdphi, invRmed_d, Rsup_d, Rinf_d, invdiffRmed_d, NRAD, NSEC, Trr_d, Tpp_d, dens_d, viscosity_array_d,
      onethird, Trp_d, invRinf_d);
  }
  else
  {
    ViscousTermsKernel<<<dimGrid2, dimBlock2>>>(vradint_d, vthetaint_d, Drr_d, Dpp_d, divergence_d, Drp_d, invdiffRsup_d,
      invdphi, invRmed_d, Rsup_d, Rinf_d, invdiffRmed_d, NRAD, NSEC, Trr_d, Tpp_d, dens_d, viscosity_array_d,
      onethird, Trp_d, invRinf_d);
  }

  gpuErrchk(hipDeviceSynchronize());
}
