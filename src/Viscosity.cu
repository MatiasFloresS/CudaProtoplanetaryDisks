#include "hip/hip_runtime.h"
#include "Main.cuh"

extern int NSEC, size_grid, nsec2pot, nrad2pot, blocksize, NRAD;

extern float TRANSITIONWIDTH, TRANSITIONRADIUS, TRANSITIONRATIO, ASPECTRATIO, LAMBDADOUBLING, *SoundSpeed_d, \
*SoundSpeed, VISCOSITY, ViscosityAlpha, *Rmed, CAVITYRATIO, CAVITYRADIUS, CAVITYWIDTH, *GLOBAL_bufarray, \
ALPHAVISCOSITY, *vrad_d, *vtheta_d, *invdiffRsup_d, *Rinf_d, *invdiffRmed_d, *dens_d, *viscosity_array_d,  \
*invRinf_d, *Rsup, *invRmed, *vthetaint_d, *VradInt_d, *viscosity_array, *Rsup_d, *invRmed_d, \
dphi, invdphi, onethird, *VradInt, *vthetaint, *Rmed_d, *invdiffRsup_d;

float PhysicalTime =0.0, PhysicalTimeInitial= 0.0, *DivergenceVelocity, *DRP, *DRR, *DPP, *TAURR, *TAURP, *TAUPP, \
*DivergenceVelocity_d, *DRP_d, *DRR_d, *DPP_d, *TAURR_d, *TAURP_d, *TAUPP_d;

extern dim3 dimGrid2, dimBlock2;

__host__ void UpdateVelocitiesWithViscosity(float *VradInt, float *vthetaint, float *dens, float DeltaT)
{

  UpdateVelocitiesKernel<<<dimGrid2, dimBlock2>>>(vthetaint_d, VradInt_d, invRmed_d, Rmed_d, Rsup_d, Rinf_d,
    invdiffRmed_d, invdiffRsup_d,  dens_d, invRinf_d, TAURR_d, TAURP_d, TAUPP_d, DeltaT, NRAD, NSEC);
    gpuErrchk(hipDeviceSynchronize());

}

__host__ float AspectRatio(float r)
{
  float aspectratio, rmin, rmax, scale;
  aspectratio = ASPECTRATIO;
  rmin = TRANSITIONRADIUS-TRANSITIONWIDTH*ASPECTRATIO;
  rmax = TRANSITIONRADIUS+TRANSITIONWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) aspectratio *= TRANSITIONRATIO;
  if ((r >= rmin) && (r <= rmax))
  {
    aspectratio *= exp((rmax-r)/(rmax-rmin)*log(TRANSITIONRATIO));
  }
  return aspectratio;
}

__host__ float FViscosity(float r)
{
  float viscosity, rmin, rmax, scale;
  int i = 0;
  viscosity = VISCOSITY;
  if (ViscosityAlpha)
  {
    while (Rmed[i] < r) i++;
    viscosity = ALPHAVISCOSITY*GLOBAL_bufarray[i] * GLOBAL_bufarray[i] * pow(r, 1.5);
  }
  rmin = CAVITYRADIUS-CAVITYWIDTH*ASPECTRATIO;
  rmax = CAVITYRADIUS+CAVITYWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) viscosity *= CAVITYRATIO;
  if ((r >= rmin) && (r <= rmax)) viscosity *= exp((rmax-r)/(rmax-rmin)*log(CAVITYRATIO));
  return viscosity;
}

__host__ void ComputeViscousTerms (float *vradial, float *vazimutal, float *dens, int option)
{

  if (ViscosityAlpha)
  {
    gpuErrchk(hipMemcpy(SoundSpeed, SoundSpeed_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
    Make1Dprofile (SoundSpeed);
  }

  for (int i = 0; i < NRAD; i++) viscosity_array[i] = FViscosity(Rmed[i]);
  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  if (option == 1)
  {

    ViscousTermsKernel<<<dimGrid2, dimBlock2>>>(vrad_d, vtheta_d, DRR_d, DPP_d, DivergenceVelocity_d, DRP_d, invdiffRsup_d,
      invdphi, invRmed_d, Rsup_d, Rinf_d, invdiffRmed_d, NRAD, NSEC, TAURR_d, TAUPP_d, dens_d, viscosity_array_d,
      onethird, TAURP_d, invRinf_d);
  }
  else
  {
    ViscousTermsKernel<<<dimGrid2, dimBlock2>>>(VradInt_d, vthetaint_d, DRR_d, DPP_d, DivergenceVelocity_d, DRP_d, invdiffRsup_d,
      invdphi, invRmed_d, Rsup_d, Rinf_d, invdiffRmed_d, NRAD, NSEC, TAURR_d, TAUPP_d, dens_d, viscosity_array_d,
      onethird, TAURP_d, invRinf_d);
  }

  gpuErrchk(hipDeviceSynchronize());
}

__host__ void InitViscosity ()
{
  DivergenceVelocity  = (float *)malloc(size_grid*sizeof(float));
  DRR                 = (float *)malloc(size_grid*sizeof(float));
  DRP                 = (float *)malloc(size_grid*sizeof(float));
  DPP                 = (float *)malloc(size_grid*sizeof(float));
  TAURR               = (float *)malloc(size_grid*sizeof(float));
  TAURP               = (float *)malloc(size_grid*sizeof(float));
  TAUPP               = (float *)malloc(size_grid*sizeof(float));
  InitViscosityDevice ();
}

__host__ void InitViscosityDevice ()
{
  gpuErrchk(hipMalloc((void**)&DivergenceVelocity_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DRR_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DRP_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DPP_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAURR_d,              size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAURP_d,              size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAUPP_d,              size_grid*sizeof(float)));
}
