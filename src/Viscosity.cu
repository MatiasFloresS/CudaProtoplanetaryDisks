#include "hip/hip_runtime.h"
#include "Main.cuh"

extern int NSEC, size_grid, NRAD;

extern float TRANSITIONWIDTH, TRANSITIONRADIUS, TRANSITIONRATIO, ASPECTRATIO, LAMBDADOUBLING;
extern float VISCOSITY, ViscosityAlpha, CAVITYRATIO, CAVITYRADIUS, CAVITYWIDTH, ALPHAVISCOSITY;

extern float *SoundSpeed_d, *Vrad_d, *Vtheta_d,  *Dens_d;
extern float *viscosity_array_d,  *VthetaInt_d, *VradInt_d;
extern float *Vradial_d, *Vazimutal_d;

extern float *SoundSpeed,  *GLOBAL_bufarray, *Rsup,  *viscosity_array, *VradInt;
extern float *VthetaInt;

extern double *invdiffRmed_d, *Rinf_d, *invRinf_d, *invRmed_d, *Rmed_d, *invRmed, *Rmed;
extern double *invdiffRsup_d, *Rsup_d;

float *DivergenceVelocity, *DRP, *DRR, *DPP, *TAURR, *TAURP, *TAUPP;
float *DivergenceVelocity_d, *DRP_d, *DRR_d, *DPP_d, *TAURR_d, *TAURP_d, *TAUPP_d;
float PhysicalTime =0.0, PhysicalTimeInitial= 0.0;

extern dim3 dimGrid2, dimBlock2;

__host__ void UpdateVelocitiesWithViscosity(float *VradInt, float *VthetaInt, float *Dens, float DeltaT)
{
  UpdateVelocitiesKernel<<<dimGrid2, dimBlock2>>>(VthetaInt_d, VradInt_d, invRmed_d, Rmed_d, Rsup_d, Rinf_d,
    invdiffRmed_d, invdiffRsup_d,  Dens_d, invRinf_d, TAURR_d, TAURP_d, TAUPP_d, DeltaT, NRAD, NSEC);
    gpuErrchk(hipDeviceSynchronize());
}


__host__ float FViscosity(double r)
{
  float viscosity, rmin, rmax, scale;
  int i = 0;
  viscosity = VISCOSITY;
  if (ViscosityAlpha){
    while (Rmed[i] < r) i++;
    viscosity = ALPHAVISCOSITY*GLOBAL_bufarray[i] * GLOBAL_bufarray[i] * pow(r, 1.5);
  }
  rmin = CAVITYRADIUS-CAVITYWIDTH*ASPECTRATIO;
  rmax = CAVITYRADIUS+CAVITYWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) viscosity *= CAVITYRATIO;
  if ((r >= rmin) && (r <= rmax)) viscosity *= exp((rmax-r)/(rmax-rmin)*log(CAVITYRATIO));
  return viscosity;
}

__host__ void ComputeViscousTerms (float *Vradial, float *Vazimutal, float *Dens)
{

  if (ViscosityAlpha){
    gpuErrchk(hipMemcpy(SoundSpeed, SoundSpeed_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
    Make1Dprofile (1);
  }

  for (int i = 0; i < NRAD; i++) viscosity_array[i] = FViscosity(Rmed[i]);
  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  ViscousTermsKernel<<<dimGrid2, dimBlock2>>>(Vradial_d, Vazimutal_d, DRR_d, DPP_d, DivergenceVelocity_d,
    DRP_d, invdiffRsup_d, invRmed_d, Rsup_d, Rinf_d, invdiffRmed_d, NRAD, NSEC, TAURR_d, TAUPP_d, Dens_d,
    viscosity_array_d, TAURP_d, invRinf_d);
  gpuErrchk(hipDeviceSynchronize());


  gpuErrchk(hipMemcpy(DRR, DRR_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));


  FILE *f;
  f = fopen("DRR.txt","w");
  for (int i = 0; i < (NRAD+1)*NSEC; i++) {
    fprintf(f, "%.10f\n", DRR[i]);
  }

  fclose(f);



}

__host__ void InitViscosity ()
{
  DivergenceVelocity  = (float *)malloc(size_grid*sizeof(float));
  DRR                 = (float *)malloc(size_grid*sizeof(float));
  DRP                 = (float *)malloc(size_grid*sizeof(float));
  DPP                 = (float *)malloc(size_grid*sizeof(float));
  TAURR               = (float *)malloc(size_grid*sizeof(float));
  TAURP               = (float *)malloc(size_grid*sizeof(float));
  TAUPP               = (float *)malloc(size_grid*sizeof(float));
  InitViscosityDevice ();
}

__host__ void InitViscosityDevice ()
{
  gpuErrchk(hipMalloc((void**)&DivergenceVelocity_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DRR_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DRP_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DPP_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAURR_d,              size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAURP_d,              size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAUPP_d,              size_grid*sizeof(float)));
}


__host__ float AspectRatioHost(float r)
{
  float aspectratio, rmin, rmax, scale;
  aspectratio = ASPECTRATIO;
  rmin = TRANSITIONRADIUS-TRANSITIONWIDTH*ASPECTRATIO;
  rmax = TRANSITIONRADIUS+TRANSITIONWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) aspectratio *= TRANSITIONRATIO;
  if ((r >= rmin) && (r <= rmax)){
    aspectratio *= expf((rmax-r)/(rmax-rmin)*logf(TRANSITIONRATIO));
  }
  return aspectratio;
}
