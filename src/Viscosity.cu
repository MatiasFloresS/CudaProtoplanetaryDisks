#include "hip/hip_runtime.h"
#include "Main.cuh"

extern int NSEC, size_grid, NRAD;

extern float TRANSITIONWIDTH, TRANSITIONRADIUS, TRANSITIONRATIO, ASPECTRATIO, LAMBDADOUBLING;
extern float VISCOSITY, CAVITYRATIO, CAVITYRADIUS, CAVITYWIDTH, ALPHAVISCOSITY;
extern float ViscosityAlpha;

extern float *SoundSpeed_d, *SoundSpeed;
extern float *viscosity_array_d;
extern float *GLOBAL_bufarray,  *viscosity_array;
extern float *invdiffRmed_d, *Rinf_d, *invRinf_d, *invRmed_d, *Rmed_d, *invRmed, *Rmed;
extern float *invdiffRsup_d, *Rsup_d, *Rsup;


extern float *Dens_d, *Vrad_d, *Vtheta_d, *Vradial_d, *Vazimutal_d, *VradInt, *VthetaInt_d, *VradInt_d, *VthetaInt;


float *DivergenceVelocity, *DRP, *DRR, *DPP, *TAURR, *TAURP, *TAUPP;
float *DivergenceVelocity_d, *DRP_d, *DRR_d, *DPP_d, *TAURR_d, *TAURP_d, *TAUPP_d;
float PhysicalTime =0.0, PhysicalTimeInitial= 0.0;

extern dim3 dimGrid2, dimBlock2;

__host__ void UpdateVelocitiesWithViscosity(float *VradInt, float *VthetaInt, float *Dens, float DeltaT)
{
  UpdateVelocitiesKernel<<<dimGrid2, dimBlock2>>>(VthetaInt_d, VradInt_d, invRmed_d, Rmed_d, Rsup_d, Rinf_d,
    invdiffRmed_d, invdiffRsup_d,  Dens_d, invRinf_d, TAURR_d, TAURP_d, TAUPP_d, DeltaT, NRAD, NSEC);
    gpuErrchk(hipDeviceSynchronize());
}


__host__ float FViscosity(float r)
{
  float viscosity, rmin, rmax, scale;
  int i = 0;
  viscosity = VISCOSITY;
  if (ViscosityAlpha){
    while (Rmed[i] < r) i++;
    viscosity = ALPHAVISCOSITY*GLOBAL_bufarray[i] * GLOBAL_bufarray[i] * powf(r, 1.5);
  }
  rmin = CAVITYRADIUS-CAVITYWIDTH*ASPECTRATIO;
  rmax = CAVITYRADIUS+CAVITYWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) viscosity *= CAVITYRATIO;
  if ((r >= rmin) && (r <= rmax)) viscosity *= exp((rmax-r)/(rmax-rmin)*log(CAVITYRATIO));
  return viscosity;
}

__host__ void ComputeViscousTerms (float *Vradial_d, float *Vazimutal_d, float *Dens)
{

  if (ViscosityAlpha){
    gpuErrchk(hipMemcpy(SoundSpeed, SoundSpeed_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
    Make1Dprofile (1);
  }

  for (int i = 0; i < NRAD; i++) viscosity_array[i] = FViscosity(Rmed[i]);
  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  ViscousTermsKernel<<<dimGrid2, dimBlock2>>>(Vradial_d, Vazimutal_d, DRR_d, DPP_d, DivergenceVelocity_d,
    DRP_d, invdiffRsup_d, invRmed_d, Rsup_d, Rinf_d, invdiffRmed_d, NRAD, NSEC, TAURR_d, TAUPP_d, Dens_d,
    TAURP_d, invRinf_d, Rmed_d, viscosity_array_d);
  gpuErrchk(hipDeviceSynchronize());
}

__host__ void InitViscosity ()
{
  DivergenceVelocity  = (float *)malloc(size_grid*sizeof(float));
  DRR                 = (float *)malloc(size_grid*sizeof(float));
  DRP                 = (float *)malloc(size_grid*sizeof(float));
  DPP                 = (float *)malloc(size_grid*sizeof(float));
  TAURR               = (float *)malloc(size_grid*sizeof(float));
  TAURP               = (float *)malloc(size_grid*sizeof(float));
  TAUPP               = (float *)malloc(size_grid*sizeof(float));
  InitViscosityDevice ();
}

__host__ void InitViscosityDevice ()
{
  gpuErrchk(hipMalloc((void**)&DivergenceVelocity_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DRR_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DRP_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DPP_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAURR_d,              size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAURP_d,              size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAUPP_d,              size_grid*sizeof(float)));
}


__host__ float AspectRatioHost(float r)
{
  float aspectratio, rmin, rmax, scale;
  aspectratio = ASPECTRATIO;
  rmin = TRANSITIONRADIUS-TRANSITIONWIDTH*ASPECTRATIO;
  rmax = TRANSITIONRADIUS+TRANSITIONWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) aspectratio *= TRANSITIONRATIO;
  if ((r >= rmin) && (r <= rmax)){
    aspectratio *= expf((rmax-r)/(rmax-rmin)*logf(TRANSITIONRATIO));
  }
  return aspectratio;
}
